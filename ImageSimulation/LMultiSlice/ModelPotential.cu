#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(ModelFragmented* modelFragmented, size_t nx, size_t ny, double dpa, double radius) {
	this->modelFragmented = modelFragmented;
	this->nx = nx;
	this->ny = ny;
	this->nz = modelFragmented->getNumberSlices();
	this->dpa = dpa;
	this->radius = radius;
}

ModelPotential::~ModelPotential(void) {
	if(this->modelFragmented != nullptr) this->modelFragmented = nullptr;
}

int ModelPotential::calculatePotentialGrid(Image *result) {
	const size_t nChannels = result->nChannels;
	const size_t numberSlices = modelFragmented->getNumberSlices();
	const size_t nAtoms = modelFragmented->getModelSource()->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = modelFragmented->getModelSource()->getA();
	const double b = modelFragmented->getModelSource()->getB();
	const double c = modelFragmented->getModelSource()->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / numberSlices;
	
<<<<<<< HEAD
	int *atomCountInSlice = modelFragmented->atomCountInSlice;
	int *atomId = modelFragmented->atomId;
	float (*xyz)[3] = modelFragmented->xyz;
	
	const size_t MAX_THREADS = 32;
	dim3 threads(MAX_THREADS, MAX_THREADS, 1);														//������ ����������
	dim3 grid(result->width / MAX_THREADS, result->height / MAX_THREADS, result->thickness );		//������� ����������� ����� ����� ������� ��� �����������

	hipError_t err = hipSuccess;
	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	calculateProjectedPotential<<<grid, threads>>>(atomCountInSlice, atomId, xyz, nAtoms, a, b, c, dx, dy, dz, (double*) (result->imageData), nChannels, nx, ny, numberSlices, dk);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		system("pause");
        exit(EXIT_FAILURE);
    }
	
	std::cout << std::endl << "Kernel time: " << time << "ms." << std::endl << std::endl;

	atomCountInSlice = nullptr;
	atomId = nullptr;
	xyz = nullptr;
=======

	int *atomId;
	float (*xyz)[3];
	float *val;

	hipMallocManaged(&atomId, nAtoms * sizeof(int));
	hipMallocManaged(&xyz, nAtoms * 3 * sizeof(float));
	hipMallocManaged(&val, sizeof(float));
	
	for(size_t kz = 0; kz < nz; kz++) {
		
		Slice *currentSlice = modelFragmented->getSliceByZ((kz + 1) * dz);
		for(size_t i = 0; i < currentSlice->size(); i++) {
			atomId[i] = modelFragmented->getModelSource()->getNumberByName((*currentSlice)[i].element.Atom);
			xyz[i][0] = (*currentSlice)[i].element.xsCoordinate.x;
			xyz[i][1] = (*currentSlice)[i].element.xsCoordinate.y;
			xyz[i][2] = (*currentSlice)[i].element.xsCoordinate.z;
		}

		const size_t MAX_THREADS = 16;
		dim3 threads(MAX_THREADS, MAX_THREADS, 1);										//������ ����������
		dim3 grid(result->width / MAX_THREADS, result->height / MAX_THREADS, 1);		//������� ����������� ����� ����� ������� ��� �����������

		calculateProjectedPotentialSlide<<<grid, threads>>>(atomId, xyz, nAtoms, a, b, c, dx, dy, dz, (double*) (result->imageData), nChannels, nx, ny, dk);
		hipDeviceSynchronize();

	} // z
>>>>>>> parent of d306cea... all slides of model calculate on GPU

	return 0;
}

int ModelPotential::savePotential(Image *image) {

	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
