#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"


ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double dpa, double radius) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
	this->radius = radius;
	
	this->potential = (double*) malloc(nx * ny * nz * sizeof(double));
	memset(this->potential, 0, nx * ny * nz * sizeof(double));
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
	if(this->potential != nullptr) { free(this->potential); }
}

int ModelPotential::calculatePotentialGrid() {
	const size_t nAtoms = model->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = model->getA();
	const double b = model->getB();
	const double c = model->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / this->nz;
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	double *potentialSlice;
	hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double));
	memset(potentialSlice, 0, nx * ny * sizeof(double));
	CUERR

	//////////////////////////////////////////////////////////////////////////////////////////////////////

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY, 1);										// ������ ����������
	dim3 grid(this->nx / BLOCKSIZEX / UNROLLX, this->ny / BLOCKSIZEY, 1 );		// ������� ����������� ����� ����� ������� ��� �����������


	AModel::Cortege *pAtoms = model->getTableCell();
	std::sort(pAtoms, pAtoms + nAtoms);
	
	int		atominfoid_host[ATOMS_IN_CONST_MEMORY];
	float	atominfoxy_host[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * ATOMS_IN_CONST_MEMORY];
	
	int j = 0;
	for(size_t kz = 0; kz * dz < c; kz++) {
		for(size_t i = 0; i < nAtoms; i++) {
			if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c && pAtoms[i].element.xsCoordinate.z * c < (kz + 1) * dz ) {
 				atominfoid_host[j] = model->getNumberByName(pAtoms[i].element.Atom) - 1;
 				atominfoxy_host[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * j + 0] = pAtoms[i].element.xsCoordinate.x;
 				atominfoxy_host[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * j + 1] = pAtoms[i].element.xsCoordinate.y;
				j = j + 1;
			}

			if(j == ATOMS_IN_CONST_MEMORY || i == nAtoms - 1) {
 				checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(atominfoid), atominfoid_host, ATOMS_IN_CONST_MEMORY * sizeof(int), 0, hipMemcpyHostToDevice));
 				checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(atominfoxy), atominfoxy_host, ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * ATOMS_IN_CONST_MEMORY * sizeof(float), 0, hipMemcpyHostToDevice));
				
 				calculatePotentialGridGPU<<<grid, threads>>>(j, a, b, c, dx, dy, dz, potentialSlice, radius, dk);
 				hipDeviceSynchronize();
				CUERR

				//std::cout << "Slice: " << kz << std::endl << "Atoms: "<< j << " Current atom: " << i << std::endl;
				j = 0;				
			}
		}

		// Device to Host
		memcpy(potential + nx * ny * kz, potentialSlice, nx * ny * sizeof(double));
		memset(potentialSlice, 0, nx * ny * sizeof(double));
	}

	pAtoms = nullptr;
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	std::cout << std::endl << "Kernel time calculating potential grid: " << time << "ms." << std::endl << std::endl;
	
	return 0;
}

// __global__ void calculatePotentialGridGPU(int nAtoms, double a, double b, double c, double dx, double dy, double dz, double *image, double r, double dk) {
// 	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
// 	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
// 	const int iz = blockDim.z * blockIdx.z + threadIdx.z;
// 	const int LINESIZE = (gridDim.x * blockDim.x);
// 	const int SLIDESIZE = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
// 		
// 	int l;
// 	
// 	for(l = 0; l < nAtoms; l++) {
// 		float dX = fabsf(atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 0] * a - (ix * dx));
// 		float dY = fabsf(atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 1] * b - (iy * dy));
// 		
// 		dX = ( dX >= a / 2.0 ) ? dX - a : dX;
// 		dY = ( dY >= b / 2.0 ) ? dY - b : dY;
// 
// 		float dR = sqrtf(dX * dX + dY * dY);	
// 		if(dR > r) continue;
// 		dR = (dR < 1.0e-10) ? 1.0e-10 : dR;
// 
// 		image[ SLIDESIZE * iz + LINESIZE * iy + ix ] += calculateProjectedPotential(atominfoid[l], dR);
// 	}
// }

__global__ void calculatePotentialGridGPU(int nAtoms, double a, double b, double c, double dx, double dy, double *potential, double r, double dk) {
	const int ix = blockDim.x * blockIdx.x * UNROLLX + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = UNROLLX * gridDim.x * blockDim.x;

	int l;
	
	double imageval1 = 0.0;
	double imageval2 = 0.0;
	double imageval3 = 0.0;
	double imageval4 = 0.0;
	double imageval5 = 0.0;
	double imageval6 = 0.0;
	double imageval7 = 0.0;
	double imageval8 = 0.0;

	for(l = 0; l < nAtoms; l++) {
		float dY = fabsf(atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 1] * b - (iy * dy));
		dY = ( dY >= b / 2.0 ) ? dY - b : dY;
		dY = dY * dY;

		float x = atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 0] * a;

// 		float dX1 = ix * dx - x;
// 		float dX2 = dX1 + gridspacing_u;
// 		float dX3 = dX2 + gridspacing_u;
// 		float dX4 = dX3 + gridspacing_u;
// 		float dX5 = dX4 + gridspacing_u;
// 		float dX6 = dX5 + gridspacing_u;
// 		float dX7 = dX6 + gridspacing_u;
// 		float dX8 = dX7 + gridspacing_u;

		float dX1 = fabsf(x - (ix + 0 * blockDim.x) * dx);
		float dX2 = fabsf(x - (ix + 1 * blockDim.x) * dx);
 		float dX3 = fabsf(x - (ix + 2 * blockDim.x) * dx);
		float dX4 = fabsf(x - (ix + 3 * blockDim.x) * dx);
 		float dX5 = fabsf(x - (ix + 4 * blockDim.x) * dx);
 		float dX6 = fabsf(x - (ix + 5 * blockDim.x) * dx);
 		float dX7 = fabsf(x - (ix + 6 * blockDim.x) * dx);
 		float dX8 = fabsf(x - (ix + 7 * blockDim.x) * dx);

		dX1 = ( dX1 >= a / 2.0 ) ? dX1 - a : dX1;
		dX2 = ( dX2 >= a / 2.0 ) ? dX2 - a : dX2;
		dX3 = ( dX3 >= a / 2.0 ) ? dX3 - a : dX3;
		dX4 = ( dX4 >= a / 2.0 ) ? dX4 - a : dX4;
		dX5 = ( dX5 >= a / 2.0 ) ? dX5 - a : dX5;
		dX6 = ( dX6 >= a / 2.0 ) ? dX6 - a : dX6;
		dX7 = ( dX7 >= a / 2.0 ) ? dX7 - a : dX7;
		dX8 = ( dX8 >= a / 2.0 ) ? dX8 - a : dX8;
		
		float dR1 = sqrtf(dX1 * dX1 + dY);
		float dR2 = sqrtf(dX2 * dX2 + dY);
		float dR3 = sqrtf(dX3 * dX3 + dY);
		float dR4 = sqrtf(dX4 * dX4 + dY);
		float dR5 = sqrtf(dX5 * dX5 + dY);
		float dR6 = sqrtf(dX6 * dX6 + dY);
		float dR7 = sqrtf(dX7 * dX7 + dY);
		float dR8 = sqrtf(dX8 * dX8 + dY);

		
		int atomid = atominfoid[l];
		
		if(dR1 < r) {
			dR1 = (dR1 < 1.0e-10) ? 1.0e-10 : dR1;
			imageval1 += calculateProjectedPotential(atomid, dR1);
		}
		if(dR2 < r) {
			dR2 = (dR2 < 1.0e-10) ? 1.0e-10 : dR2;
			imageval2 += calculateProjectedPotential(atomid, dR2);
		}
		if(dR3 < r) {
			dR3 = (dR3 < 1.0e-10) ? 1.0e-10 : dR3;
			imageval3 += calculateProjectedPotential(atomid, dR3);
		}
		if(dR4 < r) {
			dR4 = (dR4 < 1.0e-10) ? 1.0e-10 : dR4;
			imageval4 += calculateProjectedPotential(atomid, dR4);
		}
		if(dR5 < r) {
			dR5 = (dR5 < 1.0e-10) ? 1.0e-10 : dR5;
			imageval5 += calculateProjectedPotential(atomid, dR5);
		}
		if(dR6 < r) {
			dR6 = (dR6 < 1.0e-10) ? 1.0e-10 : dR6;
			imageval6 += calculateProjectedPotential(atomid, dR6);
		}
		if(dR7 < r) {
			dR7 = (dR7 < 1.0e-10) ? 1.0e-10 : dR7;
			imageval7 += calculateProjectedPotential(atomid, dR7);
		}
		if(dR8 < r) {
			dR8 = (dR8 < 1.0e-10) ? 1.0e-10 : dR8;
			imageval8 += calculateProjectedPotential(atomid, dR8);
		}
	}

	potential[ LINESIZE * iy + ix					] = potential[ LINESIZE * iy + ix					] + imageval1;
  	potential[ LINESIZE * iy + ix + 1 * blockDim.x	] = potential[ LINESIZE * iy + ix + 1 * blockDim.x	] + imageval2; 
  	potential[ LINESIZE * iy + ix + 2 * blockDim.x	] = potential[ LINESIZE * iy + ix + 2 * blockDim.x	] + imageval3;
  	potential[ LINESIZE * iy + ix + 3 * blockDim.x	] = potential[ LINESIZE * iy + ix + 3 * blockDim.x	] + imageval4;
  	potential[ LINESIZE * iy + ix + 4 * blockDim.x	] = potential[ LINESIZE * iy + ix + 4 * blockDim.x	] + imageval5;
  	potential[ LINESIZE * iy + ix + 5 * blockDim.x	] = potential[ LINESIZE * iy + ix + 5 * blockDim.x	] + imageval6;
  	potential[ LINESIZE * iy + ix + 6 * blockDim.x	] = potential[ LINESIZE * iy + ix + 6 * blockDim.x	] + imageval7;
  	potential[ LINESIZE * iy + ix + 7 * blockDim.x	] = potential[ LINESIZE * iy + ix + 7 * blockDim.x	] + imageval8;

}

__device__ double	calculateProjectedPotential(int numberAtom, double r) {
	double sumf;
	double sums;
 	double dR1;

	sumf = 0.0;
	sums = 0.0;
	dR1 = 6.2831853071796 * r; // 2 * PI * r

 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 	}
	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
	sums *= 150.36539697148; // 2 * PI * PI * a0 * e

	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;

	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	
	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}

int	ModelPotential::savePotential(const char* filename) {
	Image *image = new Image(nx, ny, nz, sizeof(double), 1);
	char filenamept[256];
	strcpy(filenamept, filename);
	strcat(filenamept, "_pt");
	memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
	image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
	delete image;

	return 0;
}

AModel::Model* ModelPotential::getModel() {
	return model;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}

