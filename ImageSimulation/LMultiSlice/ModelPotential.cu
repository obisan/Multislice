#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"


ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double dpa, double radius) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
	this->radius = radius;
	
	this->potential = (double*) malloc(nx * ny * nz * sizeof(double));
	memset(this->potential, 0, nx * ny * nz * sizeof(double));
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
	if(this->potential != nullptr) { free(this->potential); }
}

int ModelPotential::calculatePotentialGrid() {
	const size_t nAtoms = model->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = model->getA();
	const double b = model->getB();
	const double c = model->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / this->nz;
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	double *potentialSlice;
	hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double));
	memset(potentialSlice, 0, nx * ny * sizeof(double));
	CUERR

	//////////////////////////////////////////////////////////////////////////////////////////////////////

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY, 1);									// ������ ����������
	dim3 grid(this->nx / BLOCKSIZEX, this->ny / BLOCKSIZEY, 1 );		// ������� ����������� ����� ����� ������� ��� �����������


	AModel::Cortege *pAtoms = model->getTableCell();
	std::sort(pAtoms, pAtoms + nAtoms);
	
	float bindimx = 15; // angstrem
	float bindimy = 15; // angstrem

	int	binx = ceil(a / bindimx);
	int	biny = ceil(b / bindimy);

	std::vector<atom> *bins = new std::vector<atom> [biny * binx];
	
	int *bins_num;
	hipMallocManaged(&(bins_num), binx * biny * sizeof(int));
	
	std::vector<atom> slice;

	for(size_t kz = 0; kz < nz; kz++) {
		for(size_t i = 0; i < nAtoms; i++) {
			if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c && pAtoms[i].element.xsCoordinate.z * c <= (kz + 1) * dz ) {
				atom buff;
 				buff.id = i + 1;
				buff.num = model->getNumberByName(pAtoms[i].element.Atom) - 1;
				buff.x = pAtoms[i].element.xsCoordinate.x;
				buff.y = pAtoms[i].element.xsCoordinate.y;

				slice.push_back(buff);
			}
		}

		for(size_t iy = 0; iy < biny; iy++) {
			for(size_t jx = 0; jx < binx; jx++) {
				for(auto t : slice) {
					if( t.x * a >= jx * bindimx && t.x * a < (jx + 1) * bindimx )
						if( t.y * b >= iy * bindimy && t.y * b < (iy + 1) * bindimy )
							bins[binx * iy + jx].push_back(t);
				}
			}
		}

		int summ_atoms_in_bins = 0;
		for(size_t i = 0; i < binx * biny; i++) {
			summ_atoms_in_bins += bins[i].size();
		}

		atom *bins_d;
		hipMallocManaged(&(bins_d), slice.size() * sizeof(atom));

		for(size_t i = 0, k = 0; i < binx * biny; i++) {
			bins_num[i] = bins[i].size();
			for(auto s : bins[i]) {
				bins_d[k] = s;
				k++;
			}
			bins[i].clear();
		}

		// make number to offset
		for(int i = 0; i < binx * biny - 1; i++) {
			bins_num[i+1] += bins_num[i];
		}
		bins_num[0] = 0;

		calculatePotentialGridGPU<<<grid, threads>>>(potentialSlice, bins_num, bins_d, a, b, c, dx, dy, binx, biny, bindimx, bindimy, radius);
		hipDeviceSynchronize();

		hipMemcpy(potential + nx * ny * kz, potentialSlice, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
		//memset(potentialSlice, 0, nx * ny * sizeof(double));

		slice.clear();

		hipFree(bins_d);
	}

	pAtoms = nullptr;
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	std::cout << std::endl << "Kernel time calculating potential grid: " << time << "ms." << std::endl << std::endl;
	
	return 0;
}

__global__ void calculatePotentialGridGPU(double *potential, int *bin_num, atom *bin_d, double a, double b, double c, double dx, double dy, double binx, double biny, double bindimx, double bindimy, double radius) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;

	int i,j,kb;
	i = j = kb = 0;

	int bins[64] = {0};
	
	float coordx = ix * dx;
	float coordy = iy * dy;

	for(i = 0; i < biny && kb < 64; i++) {
		float dY1 = fabsf(coordy - (i + 0) * bindimy);
		float dY2 = fabsf(coordy - (i + 1) * bindimy);
		
		dY1 = ( dY1 >= b / 2.0 ) ? fabsf(dY1 - b) : dY1;
		dY2 = ( dY2 >= b / 2.0 ) ? fabsf(dY2 - b) : dY2;

		if(dY1 <= radius + 0.866 * bindimy || dY2 <= radius + 0.866 * bindimy) { // 6.928 = 4 * sqrt(3) // 0.866 = sqrt(3)/2
			for(j = 0; j < binx && kb < 64; j++) {
				float dX1 = fabsf(coordx - (j + 0) * bindimx);
				float dX2 = fabsf(coordx - (j + 1) * bindimx);
			
				dX1 = ( dX1 >= a / 2.0 ) ? fabsf(dX1 - a) : dX1;
				dX2 = ( dX2 >= a / 2.0 ) ? fabsf(dX2 - a) : dX2;

				if(dX1 <= radius + 0.866 * bindimx || dX2 <= radius + 0.866 * bindimx) { // 6.928 = 4 * sqrt(3) // 0.866 = sqrt(3)/2
					bins[kb] = binx * i + j;
					kb++;
				}				
			}
		}
	}

	float imageval = 0.0f;

	for(i = 0; i < kb - 1; i++) {
		int n = bin_num[bins[i]+1] - bin_num[bins[i]];
		int offset = bin_num[bins[i]];
		for(j = 0; j < n; j++) {
			float x = fabsf(bin_d[offset + j].x * a - ix * dx);
			float y = fabsf(bin_d[offset + j].y * b - iy * dy);

			x = ( x >= a / 2.0 ) ? x - a : x;
			y = ( y >= b / 2.0 ) ? y - b : y;

			float r = sqrtf(x * x + y * y);

			imageval += calculateProjectedPotential(bin_d[offset + j].num, r);
		}
	}

	potential[ LINESIZE * iy + ix ] = imageval; 
	
}

__device__ double	calculateProjectedPotential(int numberAtom, double r) {
	double sumf;
	double sums;
 	double dR1;

	sumf = 0.0;
	sums = 0.0;
	dR1 = 6.2831853071796 * r; // 2 * PI * r

 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 	}
	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
	sums *= 150.36539697148; // 2 * PI * PI * a0 * e

	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;
		
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	
	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}

int	ModelPotential::savePotential(const char* filename) {
	Image *image = new Image(nx, ny, nz, sizeof(double), 1);
	char filenamept[256];
	strcpy(filenamept, filename);
	strcat(filenamept, "_pt");
	memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
	image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
	delete image;

	return 0;
}

AModel::Model* ModelPotential::getModel() {
	return model;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}

