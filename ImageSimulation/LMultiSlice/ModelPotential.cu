#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(ModelFragmented* modelFragmented, size_t nx, size_t ny, double dpa, double radius) {
	this->modelFragmented = modelFragmented;
	this->nx = nx;
	this->ny = ny;
	this->nz = modelFragmented->getNumberSlices();
	this->dpa = dpa;
	this->radius = radius;
}

ModelPotential::~ModelPotential(void) {
	if(this->modelFragmented != nullptr) this->modelFragmented = nullptr;
}

int ModelPotential::calculatePotentialGrid(Image *result) {
	const size_t nChannels = result->nChannels;
	const size_t numberSlices = modelFragmented->getNumberSlices();
	const size_t nAtoms = modelFragmented->getModelSource()->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = modelFragmented->getModelSource()->getA();
	const double b = modelFragmented->getModelSource()->getB();
	const double c = modelFragmented->getModelSource()->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / numberSlices;
	
	int		*atomId		= modelFragmented->atomId;
	float	(*xyz)[3]	= modelFragmented->xyz;
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	printf("\nDetected %d CUDA accelerators:\n", deviceCount);
	int dev;
	for (dev=0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n", 
			   dev, deviceProp.name, 
			   deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
			   deviceProp.major, deviceProp.minor);
	  }

	int cudadev = 0;
	printf("  Single-threaded single-GPU test run.\n");
	printf("  Opening CUDA device %d...\n\n", cudadev);
	hipSetDevice(cudadev);
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	const size_t MAX_THREADS = 16;
	dim3 threads(MAX_THREADS, MAX_THREADS, 1);														//������ ����������
	dim3 grid(result->width / MAX_THREADS, result->height / MAX_THREADS, result->thickness );		//������� ����������� ����� ����� ������� ��� �����������

	CUERR

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	calculateProjectedPotential<<<grid, threads>>>(atomId, xyz, nAtoms, a, b, c, dx, dy, dz, (double*) (result->imageData), nChannels, nx, ny, numberSlices, radius, dk);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
		
	CUERR
	
	std::cout << std::endl << "Kernel time: " << time << "ms." << std::endl << std::endl;

	atomId = nullptr;
	xyz = nullptr;

	return 0;
}

int ModelPotential::savePotential(Image *image) {

	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
