#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(ModelFragmented* modelFragmented, size_t nx, size_t ny, double dpa, double radius) {
	this->modelFragmented = modelFragmented;
	this->nx = nx;
	this->ny = ny;
	this->nz = modelFragmented->getNumberSlices();
	this->dpa = dpa;
	this->radius = radius;
}

ModelPotential::~ModelPotential(void) {
	if(this->modelFragmented != nullptr) this->modelFragmented = nullptr;
}

int ModelPotential::calculatePotentialGrid(Image *result) {
	const size_t nChannels = result->nChannels;
	const size_t numberSlices = modelFragmented->getNumberSlices();
	const size_t nAtoms = modelFragmented->getModelSource()->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = modelFragmented->getModelSource()->getA();
	const double b = modelFragmented->getModelSource()->getB();
	const double c = modelFragmented->getModelSource()->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / numberSlices;
	

	int *atomId;
	float (*xyz)[3];
	float *val;

	hipMallocManaged(&atomId, nAtoms * sizeof(int));
	hipMallocManaged(&xyz, nAtoms * 3 * sizeof(float));
	hipMallocManaged(&val, sizeof(float));
	
	for(size_t kz = 0; kz < nz; kz++) {
		
		Slice *currentSlice = modelFragmented->getSliceByZ((kz + 1) * dz);
		for(size_t i = 0; i < currentSlice->size(); i++) {
			atomId[i] = modelFragmented->getModelSource()->getNumberByName((*currentSlice)[i].element.Atom);
			xyz[i][0] = (*currentSlice)[i].element.xsCoordinate.x;
			xyz[i][1] = (*currentSlice)[i].element.xsCoordinate.y;
			xyz[i][2] = (*currentSlice)[i].element.xsCoordinate.z;
		}

 		for(size_t iy = 0; iy < ny; iy++) {
  			double *pResult = result->getPointer<double>(0, iy);
  			for(size_t jx = 0; jx < nx; jx++) {
				for(size_t l = 0; l < nAtoms; l++) {
					double dX = fabs(xyz[l][0] * a - (jx * dx));
  					double dY = fabs(xyz[l][1] * b - (iy * dy));
  					double dZ = fabs(xyz[l][2] * c - (0 * dz));
  					
  					if(dZ >= dz) continue;
  
  					if( dX >= a / 2.0 ) dX = dX - a;
  					if( dY >= b / 2.0 ) dY = dY - b;
  					
  					double dR = sqrt(dX * dX + dY * dY) * dk;
  					int m = atomId[l] - 1;
  					double calculateProjectedAtomicPotential;
  
  					if( dR < 1.0e-10 ) dR = 1.0e-10;
  
  					//pResult[nChannels * jx + 0] += calculateProjectedPotential(m, dR);
					//calculateProjectedPotential(m, dR, val);
					calculateProjectedPotentialGPU<<<1, 1>>> (m, dR, val);
					hipDeviceSynchronize();
					
					float hval;
					memcpy(&hval, val, sizeof(float));

					pResult[nChannels * jx + 0] += hval;
  					pResult[nChannels * jx + 1] = 0;
				} // atom
  			} // x
  		} // y

	} // z

	return 0;
}

int ModelPotential::savePotential(Image *image) {

	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
