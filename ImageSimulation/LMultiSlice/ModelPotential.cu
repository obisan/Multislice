#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(ModelFragmented* modelFragmented, size_t nx, size_t ny, double dpa, double radius) {
	this->modelFragmented = modelFragmented;
	this->nx = nx;
	this->ny = ny;
	this->nz = modelFragmented->getNumberSlices();
	this->dpa = dpa;
	this->radius = radius;
}

ModelPotential::~ModelPotential(void) {
	if(this->modelFragmented != nullptr) this->modelFragmented = nullptr;
}

int ModelPotential::calculatePotentialGrid(Image *result) {
	const size_t nChannels = result->nChannels;
	const size_t numberSlices = modelFragmented->getNumberSlices();
	const size_t nAtoms = modelFragmented->getModelSource()->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = modelFragmented->getModelSource()->getA();
	const double b = modelFragmented->getModelSource()->getB();
	const double c = modelFragmented->getModelSource()->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / numberSlices;
	

	int *atomId;
	float (*xyz)[3];
	float *val;

	hipMallocManaged(&atomId, nAtoms * sizeof(int));
	hipMallocManaged(&xyz, nAtoms * 3 * sizeof(float));
	hipMallocManaged(&val, sizeof(float));
	
	for(size_t kz = 0; kz < nz; kz++) {
		
		Slice *currentSlice = modelFragmented->getSliceByZ((kz + 1) * dz);
		for(size_t i = 0; i < currentSlice->size(); i++) {
			atomId[i] = modelFragmented->getModelSource()->getNumberByName((*currentSlice)[i].element.Atom);
			xyz[i][0] = (*currentSlice)[i].element.xsCoordinate.x;
			xyz[i][1] = (*currentSlice)[i].element.xsCoordinate.y;
			xyz[i][2] = (*currentSlice)[i].element.xsCoordinate.z;
		}

		const size_t MAX_THREADS = 16;
		dim3 threads(MAX_THREADS, MAX_THREADS, 1);										//������ ����������
		dim3 grid(result->width / MAX_THREADS, result->height / MAX_THREADS, 1);		//������� ����������� ����� ����� ������� ��� �����������

		calculateProjectedPotentialSlide<<<grid, threads>>>(atomId, xyz, nAtoms, a, b, c, dx, dy, dz, (double*) (result->imageData), nChannels, nx, ny, dk);
		hipDeviceSynchronize();

	} // z

	return 0;
}

int ModelPotential::savePotential(Image *image) {

	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
