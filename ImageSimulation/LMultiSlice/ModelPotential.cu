#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double dpa, double radius) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
	this->radius = radius;
	
	hipMallocManaged(&(this->potential), nx * ny * nz * sizeof(double));

	hipError_t err;
	if ((err = hipGetLastError()) != hipSuccess)
		printf("CUDA error: %s, file %s, line %d\n", hipGetErrorString(err), __FILE__,  __LINE__);
	
	memset(this->potential, 0, nx * ny * nz * sizeof(double));
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
}

int ModelPotential::calculatePotentialGrid() {
	const size_t nAtoms = model->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = model->getA();
	const double b = model->getB();
	const double c = model->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / this->nz;
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	printf("\nDetected %d CUDA accelerators:\n", deviceCount);
	int dev;
	for (dev=0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n", 
			   dev, deviceProp.name, 
			   deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
			   deviceProp.major, deviceProp.minor);
	  }

	int cudadev = 0;
	printf("  Single-threaded single-GPU test run.\n");
	printf("  Opening CUDA device %d...\n\n", cudadev);
	hipSetDevice(cudadev);
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	const size_t MAX_THREADS = 16;
	dim3 threads(MAX_THREADS, MAX_THREADS, 1);							// ������ ����������
	dim3 grid(this->nx / MAX_THREADS, this->ny / MAX_THREADS, 1 );		// ������� ����������� ����� ����� ������� ��� �����������

	AModel::Cortege *pAtoms = model->getTableCell();
	std::sort(pAtoms, pAtoms + nAtoms);
	
	thrust::device_vector<int1>		atomIDDevice;
	thrust::device_vector<float3>	atomXYZDevice;

	int1 buffiD;
	float3 buffXYZ;
				
	for(size_t kz = 0; kz * dz < c; kz++) {
		for(size_t i = 0; i < nAtoms; i++) {
			if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c && pAtoms[i].element.xsCoordinate.z * c < (kz + 1) * dz ) {
				buffiD.x = model->getNumberByName(pAtoms[i].element.Atom) - 1;
				buffXYZ.x = pAtoms[i].element.xsCoordinate.x;
				buffXYZ.y = pAtoms[i].element.xsCoordinate.y;
				buffXYZ.z = pAtoms[i].element.xsCoordinate.z;

				atomIDDevice.push_back(buffiD);
				atomXYZDevice.push_back(buffXYZ);
			}
		}
		int1	*patomID	= thrust::raw_pointer_cast(&atomIDDevice[0]);
		float3	*patomXYZ	= thrust::raw_pointer_cast(&atomXYZDevice[0]);

		calculateProjectedPotential<<<grid, threads>>>(atomIDDevice.size(), patomID, patomXYZ, a, b, c, dx, dy, dz, potential + nx * ny * kz, radius, dk);
		hipDeviceSynchronize();

		atomIDDevice.clear();
		atomXYZDevice.clear();
	}

	pAtoms = nullptr;
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	CUERR
	
	std::cout << std::endl << "Kernel time: " << time << "ms." << std::endl << std::endl;
	
	return 0;
}

int	ModelPotential::savePotential(const char* filename) {
	Image *image = new Image(nx, ny, nz, sizeof(double), 1);
	memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
	image->saveMRC(filename, model, nx, ny, nz, mrc_FLOAT);
	delete image;

	return 0;
}

AModel::Model* ModelPotential::getModel() {
	return model;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
