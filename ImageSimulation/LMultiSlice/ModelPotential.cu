#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double dpa, double radius) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
	this->radius = radius;
	
	hipMallocManaged(&(this->potential), nx * ny * nz * sizeof(double));

	hipError_t err;
	if ((err = hipGetLastError()) != hipSuccess)
		printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);
	
	memset(this->potential, 0, nx * ny * nz * sizeof(double));
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
}

int ModelPotential::calculatePotentialGrid() {
	const size_t nAtoms = model->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = model->getA();
	const double b = model->getB();
	const double c = model->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / this->nz;
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	printf("\nDetected %d CUDA accelerators:\n", deviceCount);
	int dev;
	for (dev=0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n", 
			   dev, deviceProp.name, 
			   deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
			   deviceProp.major, deviceProp.minor);
	  }

	int cudadev = 0;
	printf("  Single-threaded single-GPU test run.\n");
	printf("  Opening CUDA device %d...\n\n", cudadev);
	hipSetDevice(cudadev);
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	int1	*atomId;
	float3	*atomXYZ;
	
	hipMallocManaged(&atomId,	nAtoms * sizeof(int3));
	hipMallocManaged(&atomXYZ,	nAtoms * sizeof(float3));
	
	CUERR

	AModel::Cortege *pAtoms = model->getTableCell();
	std::sort(pAtoms, pAtoms + nAtoms);
	for(size_t i = 0; i < nAtoms; i++) {
		atomId[i].x  = model->getNumberByName(pAtoms[i].element.Atom) - 1;
		atomXYZ[i].x = pAtoms[i].element.xsCoordinate.x;
		atomXYZ[i].y = pAtoms[i].element.xsCoordinate.y;
		atomXYZ[i].z = pAtoms[i].element.xsCoordinate.z;
	}
	pAtoms = nullptr;

	const size_t MAX_THREADS = 16;
	dim3 threads(MAX_THREADS, MAX_THREADS, 1);							// ������ ����������
	dim3 grid(this->nx / MAX_THREADS, this->ny / MAX_THREADS, 1 );		// ������� ����������� ����� ����� ������� ��� �����������

	CUERR

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	for(size_t kz = 0; kz * dz < c; kz++) {
		calculateProjectedPotential<<<grid, threads>>>(atomId, atomXYZ, nAtoms, a, b, c, dx, dy, dz, potential + nx * ny * kz, nx, ny, nz, radius, dk);
		hipDeviceSynchronize();
	}
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	CUERR
	
	std::cout << std::endl << "Kernel time: " << time << "ms." << std::endl << std::endl;

	hipFree(atomId);
	hipFree(atomXYZ);

	atomId = nullptr;
	atomXYZ = nullptr;
	
	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
