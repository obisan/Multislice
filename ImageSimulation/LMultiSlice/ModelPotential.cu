#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"


ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double dpa, double radius) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
	this->radius = radius;
	
	this->potential = (double*) malloc(nx * ny * nz * sizeof(double));
	memset(this->potential, 0, nx * ny * nz * sizeof(double));
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
	if(this->potential != nullptr) { free(this->potential); }
}

int ModelPotential::calculatePotentialGrid() {
	const size_t nAtoms = model->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = model->getA();
	const double b = model->getB();
	const double c = model->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / this->nz;
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	double *potentialSlice;
	hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double));
	memset(potentialSlice, 0, nx * ny * sizeof(double));
	CUERR

	//////////////////////////////////////////////////////////////////////////////////////////////////////

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	dim3 threads(BLOCKSIZEX, BLOCKSIZEY, 1);									// ������ ����������
	//dim3 grid(this->nx / BLOCKSIZEX / UNROLLX, this->ny / BLOCKSIZEY, 1 );		// ������� ����������� ����� ����� ������� ��� �����������
	dim3 grid(this->nx / BLOCKSIZEX, this->ny / BLOCKSIZEY, 1 );		// ������� ����������� ����� ����� ������� ��� �����������


	AModel::Cortege *pAtoms = model->getTableCell();
	std::sort(pAtoms, pAtoms + nAtoms);
	
	float bindimx = 15; // angstrem
	float bindimy = 15; // angstrem

	int	binx = ceil(a / bindimx);
	int	biny = ceil(b / bindimy);

	std::vector<atom> *bins = new std::vector<atom> [biny * binx];
	
	int *bins_num;
	hipMallocManaged(&(bins_num), binx * biny * sizeof(int));
	
	std::vector<atom> slice;

	for(size_t kz = 0; kz * dz < c; kz++) {
		for(size_t i = 0; i < nAtoms; i++) {
			if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c && pAtoms[i].element.xsCoordinate.z * c < (kz + 1) * dz ) {
				atom buff;
 				buff.id = i;
				buff.num = model->getNumberByName(pAtoms[i].element.Atom) - 1;
				buff.x = pAtoms[i].element.xsCoordinate.x;
				buff.y = pAtoms[i].element.xsCoordinate.y;

				slice.push_back(buff);
			}
		}

		for(size_t iy = 0; iy < biny; iy++) {
			for(size_t jx = 0; jx < binx; jx++) {
				for(auto t : slice) {
					if( t.x * a >= jx * bindimx && t.x * a < (jx + 1) * bindimx )
						if( t.y * b >= iy * bindimy && t.y * b < (iy + 1) * bindimy )
							bins[binx * iy + jx].push_back(t);
				}
			}
		}

		int summ_atoms_in_bins = 0;
		for(size_t i = 0; i < binx * biny; i++) {
			summ_atoms_in_bins += bins[i].size();
		}

		std::cout << "atoms in slice / bins: " << slice.size() << " / " << summ_atoms_in_bins << std::endl;
		
		atom *bins_d;
		hipMallocManaged(&(bins_d), slice.size() * sizeof(atom));

		for(size_t i = 0, k = 0; i < binx * biny; i++) {
			bins_num[i] = bins[i].size();
			for(auto s : bins[i]) {
				bins_d[k] = s;
				k++;
			}
			bins[i].clear();
		}

		// make number to offset
		for(int i = 0; i < binx * biny - 1; i++) {
			bins_num[i+1] += bins_num[i];
		}
		bins_num[0] = 0;

		calculatePotentialGridGPU<<<grid, threads>>>(potentialSlice, bins_num, bins_d, a, b, c, dx, dy, binx, biny, bindimx, bindimy, radius);
		hipDeviceSynchronize();

		hipMemcpy(potential + nx * ny * kz, potentialSlice, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
		memset(potentialSlice, 0, nx * ny * sizeof(double));

		slice.clear();

		hipFree(bins_d);
	}

	pAtoms = nullptr;
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	std::cout << std::endl << "Kernel time calculating potential grid: " << time << "ms." << std::endl << std::endl;
	
	return 0;
}

__global__ void calculatePotentialGridGPU(double *potential, int *bin_num, atom *bin_d, double a, double b, double c, double dx, double dy, double binx, double biny, double bindimx, double bindimy, double radius) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;

	int i,j,kb;
	i = j = kb = 0;

	int bins[32] = {0};
	
	float coordx = ix * dx;
	float coordy = iy * dy;

	for(i = 0; i < biny; i++) {
		float dY1 = fabsf(coordy - i * bindimy);
		float dY2 = fabsf(coordy - (i + 1) * bindimy);
		
		dY1 = ( dY1 >= b / 2.0 ) ? fabsf(dY1 - b) : dY1;
		dY2 = ( dY2 >= b / 2.0 ) ? fabsf(dY2 - b) : dY2;

		if(dY1 < bindimy || dY2 < bindimy) {
			for(j = 0; j < binx; j++) {
				float dX1 = fabsf(coordx - j * bindimx);
				float dX2 = fabsf(coordx - (j + 1) * bindimx);
			
				dX1 = ( dX1 >= a / 2.0 ) ? fabsf(dX1 - a) : dX1;
				dX2 = ( dX2 >= a / 2.0 ) ? fabsf(dX2 - a) : dX2;

				if(dX1 < bindimx || dX2 < bindimx) {
					bins[kb] = binx * i + j;
					kb++;
				}				
			}
		}
	}

	float imageval = 0.0f;

	for(i = 0; i < kb - 1; i++) {
		int n = bin_num[bins[i]+1] - bin_num[bins[i]];
		int offset = bin_num[bins[i]];
		for(j = 0; j < n; j++) {
			float x = fabsf(bin_d[offset + j].x * a - ix * dx);
			float y = fabsf(bin_d[offset + j].y * b - iy * dy);

			x = ( x >= a / 2.0 ) ? x - a : x;
			y = ( y >= b / 2.0 ) ? y - b : y;

			float r = sqrtf(x * x + y * y);

			imageval += calculateProjectedPotential(bin_d[offset + j].num, r);
		}
	}

	potential[ LINESIZE * iy + ix ] = potential[ LINESIZE * iy + ix ] + imageval; 
	
}

// __global__ void calculatePotentialGridGPU(int nAtoms, double a, double b, double c, double dx, double dy, double *potential, double r, double dk) {
// 	const int ix = blockDim.x * blockIdx.x * UNROLLX + threadIdx.x;
// 	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
// 	const int LINESIZE = UNROLLX * gridDim.x * blockDim.x;
// 
// 	int l;
// 	
// 	double imageval1 = 0.0;
// 	double imageval2 = 0.0;
// 	double imageval3 = 0.0;
// 	double imageval4 = 0.0;
// 	double imageval5 = 0.0;
// 	double imageval6 = 0.0;
// 	double imageval7 = 0.0;
// 	double imageval8 = 0.0;
// 
// 	for(l = 0; l < nAtoms; l++) {
// 		float dY = fabsf(atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 1] * b - (iy * dy));
// 		dY = ( dY >= b / 2.0 ) ? dY - b : dY;
// 		dY = dY * dY;
// 
// 		float x = atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 0] * a;
// 
// // 		float dX1 = ix * dx - x;
// // 		float dX2 = dX1 + gridspacing_u;
// // 		float dX3 = dX2 + gridspacing_u;
// // 		float dX4 = dX3 + gridspacing_u;
// // 		float dX5 = dX4 + gridspacing_u;
// // 		float dX6 = dX5 + gridspacing_u;
// // 		float dX7 = dX6 + gridspacing_u;
// // 		float dX8 = dX7 + gridspacing_u;
// 
// 		float dX1 = fabsf(x - (ix + 0 * blockDim.x) * dx);
// 		float dX2 = fabsf(x - (ix + 1 * blockDim.x) * dx);
//  		float dX3 = fabsf(x - (ix + 2 * blockDim.x) * dx);
// 		float dX4 = fabsf(x - (ix + 3 * blockDim.x) * dx);
//  		float dX5 = fabsf(x - (ix + 4 * blockDim.x) * dx);
//  		float dX6 = fabsf(x - (ix + 5 * blockDim.x) * dx);
//  		float dX7 = fabsf(x - (ix + 6 * blockDim.x) * dx);
//  		float dX8 = fabsf(x - (ix + 7 * blockDim.x) * dx);
// 
// 		dX1 = ( dX1 >= a / 2.0 ) ? dX1 - a : dX1;
// 		dX2 = ( dX2 >= a / 2.0 ) ? dX2 - a : dX2;
// 		dX3 = ( dX3 >= a / 2.0 ) ? dX3 - a : dX3;
// 		dX4 = ( dX4 >= a / 2.0 ) ? dX4 - a : dX4;
// 		dX5 = ( dX5 >= a / 2.0 ) ? dX5 - a : dX5;
// 		dX6 = ( dX6 >= a / 2.0 ) ? dX6 - a : dX6;
// 		dX7 = ( dX7 >= a / 2.0 ) ? dX7 - a : dX7;
// 		dX8 = ( dX8 >= a / 2.0 ) ? dX8 - a : dX8;
// 		
// 		float dR1 = sqrtf(dX1 * dX1 + dY);
// 		float dR2 = sqrtf(dX2 * dX2 + dY);
// 		float dR3 = sqrtf(dX3 * dX3 + dY);
// 		float dR4 = sqrtf(dX4 * dX4 + dY);
// 		float dR5 = sqrtf(dX5 * dX5 + dY);
// 		float dR6 = sqrtf(dX6 * dX6 + dY);
// 		float dR7 = sqrtf(dX7 * dX7 + dY);
// 		float dR8 = sqrtf(dX8 * dX8 + dY);
// 
// 		
// 		int atomid = atominfoid[l];
// 		
// 		if(dR1 < r) {
// 			dR1 = (dR1 < 1.0e-10) ? 1.0e-10 : dR1;
// 			imageval1 += calculateProjectedPotential(atomid, dR1);
// 		}
// 		if(dR2 < r) {
// 			dR2 = (dR2 < 1.0e-10) ? 1.0e-10 : dR2;
// 			imageval2 += calculateProjectedPotential(atomid, dR2);
// 		}
// 		if(dR3 < r) {
// 			dR3 = (dR3 < 1.0e-10) ? 1.0e-10 : dR3;
// 			imageval3 += calculateProjectedPotential(atomid, dR3);
// 		}
// 		if(dR4 < r) {
// 			dR4 = (dR4 < 1.0e-10) ? 1.0e-10 : dR4;
// 			imageval4 += calculateProjectedPotential(atomid, dR4);
// 		}
// 		if(dR5 < r) {
// 			dR5 = (dR5 < 1.0e-10) ? 1.0e-10 : dR5;
// 			imageval5 += calculateProjectedPotential(atomid, dR5);
// 		}
// 		if(dR6 < r) {
// 			dR6 = (dR6 < 1.0e-10) ? 1.0e-10 : dR6;
// 			imageval6 += calculateProjectedPotential(atomid, dR6);
// 		}
// 		if(dR7 < r) {
// 			dR7 = (dR7 < 1.0e-10) ? 1.0e-10 : dR7;
// 			imageval7 += calculateProjectedPotential(atomid, dR7);
// 		}
// 		if(dR8 < r) {
// 			dR8 = (dR8 < 1.0e-10) ? 1.0e-10 : dR8;
// 			imageval8 += calculateProjectedPotential(atomid, dR8);
// 		}
// 	}
// 
// 	potential[ LINESIZE * iy + ix					] = potential[ LINESIZE * iy + ix					] + imageval1;
//   	potential[ LINESIZE * iy + ix + 1 * blockDim.x	] = potential[ LINESIZE * iy + ix + 1 * blockDim.x	] + imageval2; 
//   	potential[ LINESIZE * iy + ix + 2 * blockDim.x	] = potential[ LINESIZE * iy + ix + 2 * blockDim.x	] + imageval3;
//   	potential[ LINESIZE * iy + ix + 3 * blockDim.x	] = potential[ LINESIZE * iy + ix + 3 * blockDim.x	] + imageval4;
//   	potential[ LINESIZE * iy + ix + 4 * blockDim.x	] = potential[ LINESIZE * iy + ix + 4 * blockDim.x	] + imageval5;
//   	potential[ LINESIZE * iy + ix + 5 * blockDim.x	] = potential[ LINESIZE * iy + ix + 5 * blockDim.x	] + imageval6;
//   	potential[ LINESIZE * iy + ix + 6 * blockDim.x	] = potential[ LINESIZE * iy + ix + 6 * blockDim.x	] + imageval7;
//   	potential[ LINESIZE * iy + ix + 7 * blockDim.x	] = potential[ LINESIZE * iy + ix + 7 * blockDim.x	] + imageval8;
// 
// }

__device__ double	calculateProjectedPotential(int numberAtom, double r) {
	double sumf;
	double sums;
 	double dR1;

	sumf = 0.0;
	sums = 0.0;
	dR1 = 6.2831853071796 * r; // 2 * PI * r

 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 	}
	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
	sums *= 150.36539697148; // 2 * PI * PI * a0 * e

	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;

	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	
	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}

int	ModelPotential::savePotential(const char* filename) {
	Image *image = new Image(nx, ny, nz, sizeof(double), 1);
	char filenamept[256];
	strcpy(filenamept, filename);
	strcat(filenamept, "_pt");
	memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
	image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
	delete image;

	return 0;
}

AModel::Model* ModelPotential::getModel() {
	return model;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}

