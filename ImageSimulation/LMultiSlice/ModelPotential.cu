#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double dpa, double radius) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
	this->radius = radius;
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
}

int ModelPotential::calculatePotentialGrid(Image *result) {
	const size_t nChannels = result->nChannels;
	const size_t nAtoms = model->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = model->getA();
	const double b = model->getB();
	const double c = model->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / this->nz;
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	printf("\nDetected %d CUDA accelerators:\n", deviceCount);
	int dev;
	for (dev=0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n", 
			   dev, deviceProp.name, 
			   deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
			   deviceProp.major, deviceProp.minor);
	  }

	int cudadev = 0;
	printf("  Single-threaded single-GPU test run.\n");
	printf("  Opening CUDA device %d...\n\n", cudadev);
	hipSetDevice(cudadev);
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	int		*atomId;
	float	*atomR;
	int		*atomsinpixel;
	
	hipMallocManaged(&atomId,			result->width * result->height * 50 * sizeof(int));
	hipMallocManaged(&atomR,			result->width * result->height * 50 * sizeof(float));
	hipMallocManaged(&atomsinpixel,	result->width * result->height * sizeof(int));

	CUERR

	memset(atomId,			1,	result->width * result->height * 50 * sizeof(int));
	memset(atomR,			0,	result->width * result->height * 50 * sizeof(float));
	memset(atomsinpixel,	0,	result->width * result->height * sizeof(int));

	const size_t MAX_THREADS = 16;
	dim3 threads(MAX_THREADS, MAX_THREADS, 1);														//������ ����������
	dim3 grid(result->width / MAX_THREADS, result->height / MAX_THREADS, 1 );		//������� ����������� ����� ����� ������� ��� �����������

	CUERR

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	AModel::Cortege *pAtoms = model->getTableCell();
	for(size_t kz = 0; kz * dz < c; kz++) {

		for(size_t ky = 0; ky * dy < b; ky++) {
			for(size_t kx = 0; kx * dx < a; kx++) {

				size_t batomsinpixel = 0;
				for(size_t i = 0; i < nAtoms && batomsinpixel < 50; i++) {
					AModel::XYZ ppAtoms = pAtoms[i].element.xsCoordinate;
					if( (ppAtoms.z * c > kz * dz) && (ppAtoms.z * c <= (kz + 1) * dz) ) {
						double dX = fabs(ppAtoms.x * a - (kx * dx));
						double dY = fabs(ppAtoms.y * b - (ky * dy));
						 		
						dX = ( dX >= a / 2.0 ) ? dX - a : dX;
						dY = ( dY >= b / 2.0 ) ? dY - b : dY;
						 
						double dR = sqrt(dX * dX + dY * dY);
						 		
						if(dR > radius) continue;
						
						atomId	[50 * nx * ky + 50 * kx + batomsinpixel] = model->getNumberByName(pAtoms[i].element.Atom) - 1;
						atomR	[50 * nx * ky + 50 * kx + batomsinpixel] = ( dR < 1.0e-10 ) ? 1.0e-10 : dR;
						atomsinpixel[nx * ky + kx] = ++batomsinpixel;
					}
				}		
			}
		}

		
		calculateProjectedPotential<<<grid, threads>>>(atomsinpixel, atomId, atomR, a, b, c, dx, dy, dz, /*(double*) (result->imageData)*/ result->getPointer<double>(kz, 0), nChannels, nx, ny, nz, radius, dk);
		hipDeviceSynchronize();
	}
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	CUERR
	
	std::cout << std::endl << "Kernel time: " << time << "ms." << std::endl << std::endl;

	hipFree(atomsinpixel);
	hipFree(atomId);
	hipFree(atomR);

	atomId = nullptr;
	atomR = nullptr;
	atomsinpixel = nullptr;

	return 0;
}

int ModelPotential::savePotential(Image *image) {

	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
