#include "stdafx.h"
#include "ModelPotential.h"
#include "kernel.cuh"

ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(ModelFragmented* modelFragmented, size_t nx, size_t ny, double dpa, double radius) {
	this->modelFragmented = modelFragmented;
	this->nx = nx;
	this->ny = ny;
	this->nz = modelFragmented->getNumberSlices();
	this->dpa = dpa;
	this->radius = radius;
}

ModelPotential::~ModelPotential(void) {
	if(this->modelFragmented != nullptr) this->modelFragmented = nullptr;
}

int ModelPotential::calculatePotentialGrid(Image *result) {
	const size_t nChannels = result->nChannels;
	const size_t numberSlices = modelFragmented->getNumberSlices();
	const size_t nAtoms = modelFragmented->getModelSource()->getNumberAtoms();
	const double dk = 1.0 / dpa;
	const double a = modelFragmented->getModelSource()->getA();
	const double b = modelFragmented->getModelSource()->getB();
	const double c = modelFragmented->getModelSource()->getC();
	const double dx = a / this->nx;
	const double dy = b / this->ny;
	const double dz = c / numberSlices;
	
	int *atomCount = modelFragmented->atomCount;
	int *atomId = modelFragmented->atomId;
	float (*xyz)[3] = modelFragmented->xyz;
	
	const size_t MAX_THREADS = 32;
	dim3 threads(MAX_THREADS, MAX_THREADS, 1);														//������ ����������
	dim3 grid(result->width / MAX_THREADS, result->height / MAX_THREADS, result->thickness );		//������� ����������� ����� ����� ������� ��� �����������

	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	calculateProjectedPotential<<<grid, threads>>>(atomCount, atomId, xyz, nAtoms, a, b, c, dx, dy, dz, (double*) (result->imageData), nChannels, nx, ny, numberSlices, dk);
	//hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	std::cout << "Kernel time: " << time << "ms." << std::endl;

	atomCount = nullptr;
	atomId = nullptr;
	xyz = nullptr;

	return 0;
}

int ModelPotential::savePotential(Image *image) {

	return 0;
}

size_t ModelPotential::getNx() {
	return nx;
}

size_t ModelPotential::getNy() {
	return ny;
}

size_t ModelPotential::getNz() {
	return nz;
}
