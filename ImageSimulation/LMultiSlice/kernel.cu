#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "kernel.cuh"


#define ATOMS_PER_PIXEL_FOR_KERNEL 1024
#define APP ATOMS_PER_PIXEL_FOR_KERNEL

__global__ void calculateProjectedPotential(unsigned int nAtoms, int1 *atomId, float3 *atomXYZ, double a, double b, double c, double dx, double dy, double dz, double *image, double r, double dk) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int iz = blockDim.z * blockIdx.z + threadIdx.z;
	const int LINESIZE = (gridDim.x * blockDim.x);
	const int SLIDESIZE = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
	
	int1	atom[APP];
	float1	atomdist[APP];
	
	int k = 0;
	for(int l = 0; l < nAtoms && k < APP; l++) {
		double dX = fabs(atomXYZ[l].x * a - (ix * dx));
		double dY = fabs(atomXYZ[l].y * b - (iy * dy));
		
		dX = ( dX >= a / 2.0 ) ? dX - a : dX;
		dY = ( dY >= b / 2.0 ) ? dY - b : dY;

		double dR = sqrt(dX * dX + dY * dY);
		
		if(dR > r) continue;

		atom[k].x = atomId[l].x;
		atomdist[k].x = ( dR < 1.0e-10 ) ? 1.0e-10 : dR;
		
		k++;
	}

	__syncthreads();

	for(int l = 0; l < k; l++) 
		image[ SLIDESIZE * iz + LINESIZE * iy + ix ] += calculateProjectedPotential(atom[l].x, atomdist[l].x);

}

__global__ void phaseObject(double *potential, cusp::complex<double>* wave, unsigned int nx, unsigned int ny, double sigma) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;

	/// t(x, y) = exp(sigma * potential(x, y))		
	double fi_re = cos(sigma * potential[ LINESIZE * iy + ix ] / 1000.0); // k - eV
	double fi_im = sin(sigma * potential[ LINESIZE * iy + ix ] / 1000.0);

	cusp::complex<double> fi(fi_re, fi_im);
	cusp::complex<double> fi2(wave[LINESIZE * iy + ix].x, wave[LINESIZE * iy + ix].y);

	/// [ t(x, y) * phi(x, y) ]
	wave[LINESIZE * iy + ix].x = (fi * fi2).real();
	wave[LINESIZE * iy + ix].y = (fi * fi2).imag();
}

__global__ void	normalize(cusp::complex<double> *pfftw, unsigned int n) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;

	if(ix < n * n) {
		pfftw[ix].x = pfftw[ix].x / n;
		pfftw[ix].y = pfftw[ix].y / n;
	}
}

__global__ void	rearrangement(cusp::complex<double> *pfftw) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE2 = gridDim.x * blockDim.x;
	const int LINESIZE = 2 * LINESIZE2;

	// 4 - 2
	swap(pfftw[iy * LINESIZE + ix].x, pfftw[(iy + LINESIZE2) * LINESIZE + LINESIZE2 + ix].x);
	swap(pfftw[iy * LINESIZE + ix].y, pfftw[(iy + LINESIZE2) * LINESIZE + LINESIZE2 + ix].y);

	// 1 - 3
	swap(pfftw[((LINESIZE2 - 1 - iy) + LINESIZE2) * LINESIZE + ix].x, pfftw[(LINESIZE2 - 1 - iy) * LINESIZE + LINESIZE2 + ix].x);
	swap(pfftw[((LINESIZE2 - 1 - iy) + LINESIZE2) * LINESIZE + ix].y, pfftw[(LINESIZE2 - 1 - iy) * LINESIZE + LINESIZE2 + ix].y);
}	

__global__ void objectLens(cusp::complex<double> *wave, double lambda, double Cs, double aperture, double defocus, double imageSizeAngstrems) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;
	
	double u1 = fabs(LINESIZE / 2.0 - iy) / imageSizeAngstrems;
	double u2 = fabs(LINESIZE / 2.0 - ix) / imageSizeAngstrems;
	double k = u1 * u1 + u2 * u2;
	double alpha = getAlpha(k, lambda, Cs, defocus);
	double Es =  getEs(k, lambda, Cs, aperture, defocus);
	cusp::complex<double> w1(Es * cos(alpha), Es * sin(alpha));
	cusp::complex<double> w2(wave[iy * LINESIZE + ix].x, wave[iy * LINESIZE + ix].y);
	wave[iy * LINESIZE + ix].x = (w1 * w2).real();
	wave[iy * LINESIZE + ix].y = (w1 * w2).imag();
}

__global__ void propagate(cusp::complex<double> *wave_0, cusp::complex<double> *wave_1, double lambda, double dZ, double imageSizeAngstrems) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;
	
	double u1 = fabs(LINESIZE / 2.0 - iy) / imageSizeAngstrems;
	double u2 = fabs(LINESIZE / 2.0 - ix) / imageSizeAngstrems;
	double k = u1 * u1 + u2 * u2;
	
	cusp::complex<double> w1(cos(- M_PI * lambda * k * k * dZ), sin(- M_PI * lambda * k * k * dZ));
	//cusp::complex<double> w1( 1.0 / (lambda * dZ) * cos(M_PI / (lambda * dZ) * k), 1.0 / (lambda * dZ) * sin(M_PI / (lambda * dZ) * k));
	//cusp::complex<double> w1( cos(-dZ / (4.0 * M_PI) * lambda * k), sin(-dZ / (4.0 * M_PI) * lambda * k));
 	cusp::complex<double> w2(wave_0[iy * LINESIZE + ix].x, wave_0[iy * LINESIZE + ix].y);
	//wave_1[iy * LINESIZE + ix].x = (w1 * w2).real();
	//wave_1[iy * LINESIZE + ix].y = (w1 * w2).imag();
	wave_1[iy * LINESIZE + ix] = (w1 * w2);
}

__device__ double	calculateProjectedPotential(int numberAtom, double r) {
	double sumf = 0, sums = 0;
 	double dR1 = 6.2831853071796 * r; // 2 * PI * r

 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 	}
	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
	sums *= 150.36539697148; // 2 * PI * PI * a0 * e

	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;
	double k0a[] = { -0.57721566, 0.42278420, 0.23069756,
		0.03488590, 0.00262698, 0.00010750, 0.00000740};
	
	double k0b[] = { 1.25331414, -0.07832358, 0.02189568,
		-0.01062446, 0.00587872, -0.00251540, 0.00053208};
	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	double i0a[] = { 1.0, 3.5156229, 3.0899424, 1.2067492,
		0.2659732, 0.0360768, 0.0045813 };
 	double i0b[] = { 0.39894228, 0.01328592, 0.00225319,
 		-0.00157565, 0.00916281, -0.02057706, 0.02635537,
 		-0.01647633, 0.00392377};

	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}

__device__ void		swap(double& a, double& b) {
		double buffer = a;
		a = b;
		b = buffer;
}

__device__ double	getAlpha(double k, double lambda, double Cs, double defocus) {
	return M_PI * lambda * k * k * (0.5 * Cs * lambda * lambda * k * k - defocus * 10);
}

__device__ double	getEs(double k, double lambda, double Cs, double aperture, double defocus) {
	return exp(- pow(M_PI * aperture / (lambda * 1000), 2) * pow(Cs * pow(lambda, 3) * k * k * k + defocus * lambda * k, 2));
}