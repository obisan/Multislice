#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "kernel.cuh"

__global__ void phaseObject(double *potential, cusp::complex<double>* wave, unsigned int nx, unsigned int ny, double sigma) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;

	/// t(x, y) = exp(sigma * potential(x, y))		
	double fi_re = cos(sigma * potential[ LINESIZE * iy + ix ] / 1000.0); // k - eV
	double fi_im = sin(sigma * potential[ LINESIZE * iy + ix ] / 1000.0);

	////////////////////////////////////////////
	cusp::complex<double> fi(fi_re, fi_im);
	
	/// [ t(x, y) * phi(x, y) ]
	wave[LINESIZE * iy + ix] = fi * wave[LINESIZE * iy + ix];
}

__global__ void nulling(cusp::complex<double> *pfftw) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	pfftw[ix].x = pfftw[ix].y = 0.0;
}

__global__ void	normalize(cusp::complex<double> *pfftw, unsigned int n) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;

	if(ix < n * n) {
		pfftw[ix].x = pfftw[ix].x / n;
		pfftw[ix].y = pfftw[ix].y / n;
	}
}

__global__ void	rearrangement(cusp::complex<double> *pfftw) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE2 = gridDim.x * blockDim.x;
	const int LINESIZE = 2 * LINESIZE2;

	// 4 - 2
	swap(pfftw[iy * LINESIZE + ix].x, pfftw[(iy + LINESIZE2) * LINESIZE + LINESIZE2 + ix].x);
	swap(pfftw[iy * LINESIZE + ix].y, pfftw[(iy + LINESIZE2) * LINESIZE + LINESIZE2 + ix].y);

	// 1 - 3
	swap(pfftw[((LINESIZE2 - 1 - iy) + LINESIZE2) * LINESIZE + ix].x, pfftw[(LINESIZE2 - 1 - iy) * LINESIZE + LINESIZE2 + ix].x);
	swap(pfftw[((LINESIZE2 - 1 - iy) + LINESIZE2) * LINESIZE + ix].y, pfftw[(LINESIZE2 - 1 - iy) * LINESIZE + LINESIZE2 + ix].y);
}	

__global__ void objectLens(cusp::complex<double> *wave, double lambda, double Cs, double aperture, double defocus, double imageSizeAngstrems) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;
	
	double u1 = fabs(LINESIZE / 2.0 - iy) / imageSizeAngstrems;
	double u2 = fabs(LINESIZE / 2.0 - ix) / imageSizeAngstrems;
	double k = u1 * u1 + u2 * u2;

	double alpha = getAlpha(k, lambda, Cs, defocus);
	double Es =  getEs(k, lambda, Cs, aperture, defocus);
	cusp::complex<double> w1(Es * cos(alpha), Es * sin(alpha));
	cusp::complex<double> w2(wave[iy * LINESIZE + ix].x, wave[iy * LINESIZE + ix].y);
	wave[iy * LINESIZE + ix].x = (w1 * w2).real();
	wave[iy * LINESIZE + ix].y = (w1 * w2).imag();
}

__global__ void propagate(cusp::complex<double> *wave_in, cusp::complex<double> *wave_out, double lambda, double dZ, double imageSizeAngstrems) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;
	
	double u1 = fabs(LINESIZE / 2.0 - iy) / imageSizeAngstrems;
	double u2 = fabs(LINESIZE / 2.0 - ix) / imageSizeAngstrems;
	double k = u1 * u1 + u2 * u2;
	
	//cusp::complex<double> w1( 1.0 / (lambda * dZ) * sin(M_PI / (lambda * dZ) * k), - 1.0 / (lambda * dZ) * cos(M_PI / (lambda * dZ) * k)); // p-mini after sanya
	cusp::complex<double> w1( cos(M_PI * lambda * k * k * dZ), sin(M_PI * lambda * k * k * dZ)); // p-big
	wave_out[iy * LINESIZE + ix] = w1 * wave_in[iy * LINESIZE + ix];
}

__device__ void		swap(double& a, double& b) {
		double buffer = a;
		a = b;
		b = buffer;
}

__device__ double	getAlpha(double k, double lambda, double Cs, double defocus) {
	return M_PI * lambda * k * k * (0.5 * Cs * lambda * lambda * k * k - defocus * 10);
}

__device__ double	getEs(double k, double lambda, double Cs, double aperture, double defocus) {
	return exp(- pow(M_PI * aperture / (lambda * 1000), 2) * pow(Cs * pow(lambda, 3) * k * k * k + defocus * lambda * k, 2));
}