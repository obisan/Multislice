#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "kernel.cuh"

__global__ void calculateProjectedPotential(int *sliceId, int *atomId, float (*xyz)[3], unsigned int nAtoms, double a, double b, double c, double dx, double dy, double dz, double *image, unsigned int nChannels, unsigned int nx, unsigned int ny, unsigned int nz, double dk) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int iz = blockDim.z * blockIdx.z + threadIdx.z;
	
	for(int l = 0; l < nAtoms; l++) {
		if(sliceId[l] == iz) {
			double dX = fabs(xyz[l][0] * a - (ix * dx));
			double dY = fabs(xyz[l][1] * b - (iy * dy));
  
			if( dX >= a / 2.0 ) dX = dX - a;
			if( dY >= b / 2.0 ) dY = dY - b;

			double dR = sqrt(dX * dX + dY * dY) * dk;

			int m = atomId[l] - 1;

			if( dR < 1.0e-10 ) dR = 1.0e-10;
			
			image[ nChannels * ((gridDim.x * blockDim.x) * (gridDim.y * blockDim.y) * iz + (gridDim.x * blockDim.x) * iy + ix) + 0 ] += calculateProjectedPotential(m, dR);
			image[ nChannels * ((gridDim.x * blockDim.x) * (gridDim.y * blockDim.y) * iz + (gridDim.x * blockDim.x) * iy + ix) + 1 ] = 0;
		}
	}

	__syncthreads();
}

__global__ void calculateProjectedPotentialSlide(int *atomId, float (*xyz)[3], unsigned int nAtoms, double a, double b, double c, double dx, double dy, double dz, double *image, unsigned int nChannels, unsigned int nx, unsigned int ny, double dk) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;

	for(int l = 0; l < nAtoms; l++) {
		double dX = fabs(xyz[l][0] * a - (ix * dx));
		double dY = fabs(xyz[l][1] * b - (iy * dy));
		double dZ = fabs(xyz[l][2] * c - (0 * dz));

		if(dZ >= dz) continue;
  
		if( dX >= a / 2.0 ) dX = dX - a;
		if( dY >= b / 2.0 ) dY = dY - b;

		double dR = sqrt(dX * dX + dY * dY) * dk;

		int m = atomId[l] - 1;

		if( dR < 1.0e-10 ) dR = 1.0e-10;

		image[ nChannels * (gridDim.x * blockDim.x * iy + ix) + 0 ] += calculateProjectedPotential(m, dR);
		image[ nChannels * (gridDim.x * blockDim.x * iy + ix) + 1 ] = 0;
	}

	__syncthreads();
}

__device__ double calculateProjectedPotential(int numberAtom, double r) {
	double sumf = 0, sums = 0;
 	double dR1 = 6.2831853071796 * r; // 2 * PI * r
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));
 	}
 				
 	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
 		
 	sums *= 150.36539697148; // 2 * PI * PI * a0 * e
 	
	return (sumf + sums);
}

__device__ double calculatePotential(int numberAtom, double r) {
 		double sumf = 0, sums = 0;
 		double dR1 = 6.2831853071796 * r; // 2 * PI * r
 		for(int k = 0; k < 3; k++) {
 			int Offs = (numberAtom) * 12 + k * 2;
 			sumf += FParamsDevice[Offs + 0] / r * exp(- dR1 * sqrt(FParamsDevice[Offs + 1]));
 		}				
 		sumf *= 150.365396971475; // 4 * PI * PI *a0 * e
 		
 		for(int k = 0; k < 3; k++) {
 			int Offs = (numberAtom) * 12 + k * 2;
 			sums += FParamsDevice[Offs + 6] * pow(FParamsDevice[Offs + 7], -3.0 / 2.0) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 		}
 		sums *= 266.5157269050303; // 2 * PI * PI * a0 * e
 		return (sumf + sums);
 	}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;
	double k0a[] = { -0.57721566, 0.42278420, 0.23069756,
		0.03488590, 0.00262698, 0.00010750, 0.00000740};
	
	double k0b[] = { 1.25331414, -0.07832358, 0.02189568,
		-0.01062446, 0.00587872, -0.00251540, 0.00053208};
	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	double i0a[] = { 1.0, 3.5156229, 3.0899424, 1.2067492,
		0.2659732, 0.0360768, 0.0045813 };
 	double i0b[] = { 0.39894228, 0.01328592, 0.00225319,
 		-0.00157565, 0.00916281, -0.02057706, 0.02635537,
 		-0.01647633, 0.00392377};

	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}