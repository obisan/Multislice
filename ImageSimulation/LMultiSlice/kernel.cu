#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "kernel.cuh"
#include "cusp.cuh"

#define ATOMS_PER_PIXEL_FOR_KERNEL 1024
#define APP ATOMS_PER_PIXEL_FOR_KERNEL

__global__ void calculateProjectedPotential(int1 *atomId, float3 *atomXYZ, unsigned int nAtoms, double a, double b, double c, double dx, double dy, double dz, double *image, unsigned int nx, unsigned int ny, unsigned int nz, double r, double dk) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int iz = blockDim.z * blockIdx.z + threadIdx.z;
	const int LINESIZE = (gridDim.x * blockDim.x);
	const int SLIDESIZE = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
	
	int1 atom[APP];
	float1 atomdist[APP];
	
	int k = 0;
	for(int l = 0; l < nAtoms && k < APP; l++) {
		double dX = fabs(atomXYZ[l].x * a - (ix * dx));
		double dY = fabs(atomXYZ[l].y * b - (iy * dy));
		
		dX = ( dX >= a / 2.0 ) ? dX - a : dX;
		dY = ( dY >= b / 2.0 ) ? dY - b : dY;

		double dR = sqrt(dX * dX + dY * dY);
		
		if(dR > r) continue;

		atom[k].x = atomId[l].x;
		atomdist[k].x = ( dR < 1.0e-10 ) ? 1.0e-10 : dR;
		
		k++;
	}

	__syncthreads();

	for(int l = 0; l < k; l++) 
		image[ SLIDESIZE * iz + LINESIZE * iy + ix ] += calculateProjectedPotential(atom[l].x, atomdist[l].x);

}

__global__ void phaseObject(double *potential, fftw_complex* pfftw, unsigned int nx, unsigned int ny, double sigma) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;

	/// T(x, y) = exp(sigma * p(x, y))
	double fi_re = cos(sigma * potential[ LINESIZE * iy + ix ] / 1000.0); // k - eV
	double fi_im = sin(sigma * potential[ LINESIZE * iy + ix ] / 1000.0);

	cusp::complex<double> fi(fi_re, fi_im);
	cusp::complex<double> fi2(pfftw[LINESIZE * iy + ix][0], pfftw[LINESIZE * iy + ix][1]);

	/// [ T(x, y) * phi(x, y) ]
	pfftw[LINESIZE * iy + ix][0] = (fi * fi2).real();
	pfftw[LINESIZE * iy + ix][1] = (fi * fi2).imag();
}

__global__ void	normalize(fftw_complex *pfftw, unsigned int n) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;

	if(ix < n * n) {
		pfftw[ix][0] = pfftw[ix][0] / n;
		pfftw[ix][1] = pfftw[ix][1] / n;
	}
}

__global__ void	rearrangement(fftw_complex *pfftw) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE2 = gridDim.x * blockDim.x;
	const int LINESIZE = 2 * LINESIZE2;

	// 4 - 2
	swap(pfftw[iy * LINESIZE + ix][0], pfftw[(iy + LINESIZE2) * LINESIZE + LINESIZE2 + ix][0]);
	swap(pfftw[iy * LINESIZE + ix][1], pfftw[(iy + LINESIZE2) * LINESIZE + LINESIZE2 + ix][1]);

	// 1 - 3
	swap(pfftw[((LINESIZE2 - 1 - iy) + LINESIZE2) * LINESIZE + ix][0], pfftw[(LINESIZE2 - 1 - iy) * LINESIZE + LINESIZE2 + ix][0]);
	swap(pfftw[((LINESIZE2 - 1 - iy) + LINESIZE2) * LINESIZE + ix][1], pfftw[(LINESIZE2 - 1 - iy) * LINESIZE + LINESIZE2 + ix][1]);
}	

__global__ void objectLens(fftw_complex *pfftw, double lambda, double Cs, double aperture, double defocus, double imageSizeAngstrems) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int LINESIZE = gridDim.x * blockDim.x;

	double u1 = fabs(LINESIZE / 2.0 - iy) / imageSizeAngstrems;
	double u2 = fabs(LINESIZE / 2.0 - ix) / imageSizeAngstrems;
	double k = u1 * u1 + u2 * u2;
	double alpha = getAlpha(k, lambda, Cs, defocus);
	double Es =  getEs(k, lambda, Cs, aperture, defocus);
	cusp::complex<double> w1(Es * cos(alpha), Es * sin(alpha));
	cusp::complex<double> w2(pfftw[iy * LINESIZE + ix][0], pfftw[iy * LINESIZE + ix][1]);
	pfftw[iy * LINESIZE + ix][0] = (w1 * w2).real();
	pfftw[iy * LINESIZE + ix][1] = (w1 * w2).imag();

}

__device__ double calculateProjectedPotential(int numberAtom, double r) {
	double sumf = 0, sums = 0;
 	double dR1 = 6.2831853071796 * r; // 2 * PI * r

 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 	}
	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
	sums *= 150.36539697148; // 2 * PI * PI * a0 * e

	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;
	double k0a[] = { -0.57721566, 0.42278420, 0.23069756,
		0.03488590, 0.00262698, 0.00010750, 0.00000740};
	
	double k0b[] = { 1.25331414, -0.07832358, 0.02189568,
		-0.01062446, 0.00587872, -0.00251540, 0.00053208};
	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	double i0a[] = { 1.0, 3.5156229, 3.0899424, 1.2067492,
		0.2659732, 0.0360768, 0.0045813 };
 	double i0b[] = { 0.39894228, 0.01328592, 0.00225319,
 		-0.00157565, 0.00916281, -0.02057706, 0.02635537,
 		-0.01647633, 0.00392377};

	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}

__device__ void		swap(double& a, double& b) {
		double buffer = a;
		a = b;
		b = buffer;
}

__device__ double	getAlpha(double k, double lambda, double Cs, double defocus) {
	return M_PI * lambda * k * k * (0.5 * Cs * lambda * lambda * k * k - defocus * 10);
}

__device__ double	getEs(double k, double lambda, double Cs, double aperture, double defocus) {
	return exp(- pow(M_PI * aperture / (lambda * 1000), 2) * pow(Cs * pow(lambda, 3) * k * k * k + defocus * lambda * k, 2));
}