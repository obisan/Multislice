#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "kernel.cuh"
#include "cusp.cuh"

#define ATOMS_PER_PIXEL_FOR_KERNEL 1024
#define APP ATOMS_PER_PIXEL_FOR_KERNEL

__global__ void calculateProjectedPotential(int1 *atomId, float3 *atomXYZ, unsigned int nAtoms, double a, double b, double c, double dx, double dy, double dz, double *image, unsigned int nx, unsigned int ny, unsigned int nz, double r, double dk) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int iz = blockDim.z * blockIdx.z + threadIdx.z;
	const int LINESIZE = (gridDim.x * blockDim.x);
	const int SLIDESIZE = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
	
	int1 atom[APP];
	float1 atomdist[APP];
	
	int k = 0;
	for(int l = 0; l < nAtoms && k < APP; l++) {
		double dX = fabs(atomXYZ[l].x * a - (ix * dx));
		double dY = fabs(atomXYZ[l].y * b - (iy * dy));
		
		dX = ( dX >= a / 2.0 ) ? dX - a : dX;
		dY = ( dY >= b / 2.0 ) ? dY - b : dY;

		double dR = sqrt(dX * dX + dY * dY);
		
		if(dR > r) continue;

		atom[k].x = atomId[l].x;
		atomdist[k].x = ( dR < 1.0e-10 ) ? 1.0e-10 : dR;
		
		k++;
	}

	__syncthreads();

	for(int l = 0; l < k; l++) 
		image[ SLIDESIZE * iz + LINESIZE * iy + ix ] += calculateProjectedPotential(atom[l].x, atomdist[l].x);

}

__device__ double calculateProjectedPotential(int numberAtom, double r) {
	double sumf = 0, sums = 0;
 	double dR1 = 6.2831853071796 * r; // 2 * PI * r

 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 	}
	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
	sums *= 150.36539697148; // 2 * PI * PI * a0 * e

	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;
	double k0a[] = { -0.57721566, 0.42278420, 0.23069756,
		0.03488590, 0.00262698, 0.00010750, 0.00000740};
	
	double k0b[] = { 1.25331414, -0.07832358, 0.02189568,
		-0.01062446, 0.00587872, -0.00251540, 0.00053208};
	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	double i0a[] = { 1.0, 3.5156229, 3.0899424, 1.2067492,
		0.2659732, 0.0360768, 0.0045813 };
 	double i0b[] = { 0.39894228, 0.01328592, 0.00225319,
 		-0.00157565, 0.00916281, -0.02057706, 0.02635537,
 		-0.01647633, 0.00392377};

	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}