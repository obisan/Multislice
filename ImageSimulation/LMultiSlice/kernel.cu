#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "kernel.cuh"

__global__ void calculateProjectedPotential(int *atomId, float (*xyz)[3], unsigned int nAtoms, double a, double b, double c, double dx, double dy, double dz, double *image, unsigned int nChannels, unsigned int nx, unsigned int ny, unsigned int nz, double r, double dk) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;
	const int iz = blockDim.z * blockIdx.z + threadIdx.z;
	const int lineSize = (gridDim.x * blockDim.x);
	const int slideSize = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
	
	int l;
	for(l = 0; l < nAtoms; l++) {
		double dX = fabs(xyz[l][0] * a - (ix * dx));
		double dY = fabs(xyz[l][1] * b - (iy * dy));
		double dZ = fabs(xyz[l][2] * b - (iz * dz));
		int m = atomId[l];

		dX = ( dX >= a / 2.0 ) ? dX - a : dX;
		dY = ( dY >= b / 2.0 ) ? dY - b : dY;
	
		double dR = sqrt(dX * dX + dY * dY);
		
		if(dZ > dz) continue;
		if(dR > r) continue;

		dR *= dk;
		dR = ( dR < 1.0e-10 ) ? 1.0e-10 : dR;
				
		image[ nChannels * (slideSize * iz + lineSize * iy + ix) + 0 ] += calculateProjectedPotential(m, dR);
		image[ nChannels * (slideSize * iz + lineSize * iy + ix) + 1 ] = 0;
	}
	
}


__device__ double calculateProjectedPotential(int numberAtom, double r) {
	double sumf = 0, sums = 0;
 	double dR1 = 6.2831853071796 * r; // 2 * PI * r
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));
 	}
 				
 	sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
 	for(int k = 0; k < 3; k++) {
 		int Offs = (numberAtom) * 12 + k * 2;
 		sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 	}
 		
 	sums *= 150.36539697148; // 2 * PI * PI * a0 * e
 	
	return (sumf + sums);
}

__device__ double	bessk0( double x ) {
	double ax, x2, sum;
	double k0a[] = { -0.57721566, 0.42278420, 0.23069756,
		0.03488590, 0.00262698, 0.00010750, 0.00000740};
	
	double k0b[] = { 1.25331414, -0.07832358, 0.02189568,
		-0.01062446, 0.00587872, -0.00251540, 0.00053208};
	
	ax = fabs( x );
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = ax / 2.0;
		x2 = x2 * x2;
		sum = k0a[6];
		for( int i = 5; i >= 0; i--) sum = sum * x2 + k0a[i];
		sum = -log(ax / 2.0) * bessi0( x ) + sum;
	} else if( ax > 2.0 ) {
		x2 = 2.0/ax;
		sum = k0b[6];
		for( int i=5; i>=0; i--) sum = sum*x2 + k0b[i];
		sum = exp( -ax ) * sum / sqrt( ax );
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double x ) {
 	double ax, sum, t;
 	double i0a[] = { 1.0, 3.5156229, 3.0899424, 1.2067492,
		0.2659732, 0.0360768, 0.0045813 };
 	double i0b[] = { 0.39894228, 0.01328592, 0.00225319,
 		-0.00157565, 0.00916281, -0.02057706, 0.02635537,
 		-0.01647633, 0.00392377};

	ax = fabs( x );
	if( ax <= 3.75 ) {
		t = x / 3.75;
		t = t * t;
		sum = i0a[6];
		for( int  i = 5; i >= 0; i--) sum = sum * t + i0a[i]; 
	} else {
		t = 3.75 / ax;
		sum = i0b[8];
		for( int i = 7; i >= 0; i--) sum = sum * t + i0b[i];
		sum = exp( ax ) * sum / sqrt( ax );
	}
	return( sum );
}