#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelSimulated.h"
#include "kernel.cuh"

ModelSimulated::ModelSimulated(void) {
	
}

ModelSimulated::~ModelSimulated(void) {
	if(this->modelPotential != nullptr) this->modelPotential = nullptr;
}

ModelSimulated::ModelSimulated(ModelPotential* modelPotential, size_t nx, size_t ny, size_t nz, double dpa) {
	this->modelPotential = modelPotential;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
}

int ModelSimulated::imageCalculation(Image *result, Microscope *microscope) {
	fftw_complex *pfftw_in  = nullptr; 
 	fftw_complex *pfftw_out = nullptr; 
 	hipMallocManaged(&(pfftw_in),  nx * ny * sizeof(fftw_complex));
 	hipMallocManaged(&(pfftw_out), nx * ny * sizeof(fftw_complex));

	for(size_t i = 0; i < nx * ny; i++) {
		pfftw_in[i][0] = 1.0;	
		pfftw_in[i][1] = 0.0;
	}
	
	double *potential = modelPotential->potential;
	double dz = this->modelPotential->getModel()->getC() / nx;
	for(size_t kz = 0; kz < nz; kz++) {	
		
		////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// t(x, y) = exp(sigma * potential(x, y))
		/// [ t(x, y) * phi(x, y) ]
		////////////////////////////////////////////////////////////////////////////////////////////////////////		
		const unsigned int MAX_THREADS_PHASE_OBJECT = 16;
		dim3 threads_phase(MAX_THREADS_PHASE_OBJECT, MAX_THREADS_PHASE_OBJECT, 1);							// ������ ��������
		dim3 grid_phase( (int) nx / MAX_THREADS_PHASE_OBJECT, (int) ny / MAX_THREADS_PHASE_OBJECT, 1 );		// ������� ��������� ����� ����� ������� ��� �����������

		phaseObject<<<grid_phase, threads_phase>>>(potential + nx * ny * kz, (cusp::complex<double>*) pfftw_in, nx, ny, microscope->getSigma());
		hipDeviceSynchronize();

		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		///// PHI(k) = FT [ phi(x, y) ]
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_forward = fftw_plan_dft_2d( (int) nx, (int) ny, pfftw_in, pfftw_out, FFTW_FORWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_forward);
		fftw_destroy_plan(fftw_forward);

		const unsigned int MAX_THREADS_NORMALIZE = 1024;
		dim3 threads_normalize(MAX_THREADS_NORMALIZE, 1, 1);
		dim3 grid_normalize( (int) nx * ny / MAX_THREADS_NORMALIZE, 1, 1 );

		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) pfftw_out, nx);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////
		/// Rearrangement 
		///	4 3  to 2 1 
		/// 1 2     3 4
		/////////////////////////////////////////////////////////////////////////////////////////////////
		const unsigned int MAX_THREADS_REARRANGEMENT = 16;
		dim3 threads_rearrangement(MAX_THREADS_REARRANGEMENT, MAX_THREADS_REARRANGEMENT, 1);									// ������ ��������
		dim3 grid_rearrangement( (int) nx / 2 / MAX_THREADS_REARRANGEMENT, (int) ny / 2 / MAX_THREADS_REARRANGEMENT, 1 );		// ������� ��������� ����� ����� ������� ��� �����������

		rearrangement<<<grid_rearrangement, threads_rearrangement>>>((cusp::complex<double>*) pfftw_out);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// p(x, y, dZ) * [ t(x, y) * phi(x, y) ]
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		const unsigned int MAX_THREADS_PROPAGATION = 16;
		dim3 threads_propagation(MAX_THREADS_PROPAGATION, MAX_THREADS_PROPAGATION, 1);							// ������ ��������
		dim3 grid_propagation( (int) nx / MAX_THREADS_PROPAGATION, (int) ny / MAX_THREADS_PROPAGATION, 1 );		// ������� ��������� ����� ����� ������� ��� �����������
		
		propagate<<<grid_propagation, threads_propagation>>>((cusp::complex<double>*) pfftw_out, microscope->getLambda(), kz * dz, nx / dpa );
		hipDeviceSynchronize();

		// 	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	 	/// H(k) * PHI(k)
	 	/////////////////////////////////////////////////////////////////////////////////////////////////////////
// 	 	const double imageSizeAngstrems = nx / dpa; // ������ ����� ����������� � ����������
// 	 	const double lambda = microscope->getLambda();
// 	 	const double Cs = microscope->getCs();
// 	 	const double aperture = microscope->getAperture();
// 	 	const double defocus = microscope->getDefocus();
// 	 	//double Z = 0;
// 	 
// 	 	const unsigned int MAX_THREADS_OBJECT_LENS = 16;
// 	 	dim3 threads_object_lens(MAX_THREADS_OBJECT_LENS, MAX_THREADS_OBJECT_LENS, 1);							// ������ ��������
// 	 	dim3 grid_object_lens( (int) nx / MAX_THREADS_OBJECT_LENS, (int) ny / MAX_THREADS_OBJECT_LENS, 1 );		// ������� ��������� ����� ����� ������� ��� �����������
// 	 
// 	 	objectLens<<<grid_object_lens, threads_object_lens>>>((cusp::complex<double>*) pfftw_out, lambda, Cs, aperture, defocus, imageSizeAngstrems);
// 	 	hipDeviceSynchronize();
	 	

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// phi(x, y) = FT^(-1) { PHI(k) }
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_backward = fftw_plan_dft_2d((int) nx, (int) ny, pfftw_out, pfftw_in, FFTW_BACKWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_backward);
		fftw_destroy_plan(fftw_backward);
		
		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) pfftw_in, nx);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////

		Image::copyFFTtoImage<double>(result, pfftw_in, kz);

		/// !!!!!!!!!!!!!!!!!!!!
		//Z = 10;
	}



	hipFree(pfftw_in);
	hipFree(pfftw_out);
	return 0;
}