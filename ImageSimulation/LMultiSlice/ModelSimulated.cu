#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelSimulated.h"
#include "kernel.cuh"

ModelSimulated::ModelSimulated(void) {
	
}

ModelSimulated::~ModelSimulated(void) {
	if(this->modelPotential != nullptr) this->modelPotential = nullptr;
}

ModelSimulated::ModelSimulated(ModelPotential* modelPotential, size_t nx, size_t ny, size_t nz, double dpa) {
	this->modelPotential = modelPotential;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
}

int ModelSimulated::imageCalculation(Image *result, Microscope *microscope) {
	fftw_complex *wave_in; 
 	fftw_complex *wave_out; 
 	hipMallocManaged(&(wave_in),  (nz + 1) * nx * ny * sizeof(fftw_complex));
 	hipMallocManaged(&(wave_out), (nz + 1) * nx * ny * sizeof(fftw_complex));

	for(size_t i = 0; i < nx * ny; i++) {
		wave_in[i][0] = 1.0;	
		wave_in[i][1] = 0.0;
	}
	
	double *potential = modelPotential->potential;
	double dz = this->modelPotential->getModel()->getC() / nx;
	for(size_t kz = 0; kz < nz; kz++) {	
		
		////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// t(x, y) = exp(sigma * potential(x, y))
		/// [ t(x, y) * phi(x, y) ]
		////////////////////////////////////////////////////////////////////////////////////////////////////////		
		const unsigned int MAX_THREADS_PHASE_OBJECT = 16;
		dim3 threads_phase(MAX_THREADS_PHASE_OBJECT, MAX_THREADS_PHASE_OBJECT, 1);							// ������ ��������
		dim3 grid_phase( (int) nx / MAX_THREADS_PHASE_OBJECT, (int) ny / MAX_THREADS_PHASE_OBJECT, 1 );		// ������� ��������� ����� ����� ������� ��� �����������

		phaseObject<<<grid_phase, threads_phase>>>(potential + nx * ny * kz, (cusp::complex<double>*) wave_in + nx * ny * kz, nx, ny, microscope->getSigma());
		hipDeviceSynchronize();

		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		///// PHI(k) = FFT [ t_n(x, y) * phi_n(x, y) ]
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_forward = fftw_plan_dft_2d( (int) nx, (int) ny, wave_in + nx * ny * kz, wave_out + nx * ny * kz, FFTW_FORWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_forward);
		fftw_destroy_plan(fftw_forward);

		const unsigned int MAX_THREADS_NORMALIZE = 1024;
		dim3 threads_normalize(MAX_THREADS_NORMALIZE, 1, 1);
		dim3 grid_normalize( (int) nx * ny / MAX_THREADS_NORMALIZE, 1, 1 );

		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_out + nx * ny * kz, nx);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////
		/// Rearrangement 
		///	4 3  to 2 1 
		/// 1 2     3 4
		/////////////////////////////////////////////////////////////////////////////////////////////////
		const unsigned int MAX_THREADS_REARRANGEMENT = 16;
		dim3 threads_rearrangement(MAX_THREADS_REARRANGEMENT, MAX_THREADS_REARRANGEMENT, 1);									// ������ ��������
		dim3 grid_rearrangement( (int) nx / 2 / MAX_THREADS_REARRANGEMENT, (int) ny / 2 / MAX_THREADS_REARRANGEMENT, 1 );		// ������� ��������� ����� ����� ������� ��� �����������

		rearrangement<<<grid_rearrangement, threads_rearrangement>>>((cusp::complex<double>*) wave_out + nx * ny * kz);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// p_n(x, y, dZ) * FFT [ t_n(x, y) * phi_n(x, y) ]
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		const unsigned int MAX_THREADS_PROPAGATION = 16;
		dim3 threads_propagation(MAX_THREADS_PROPAGATION, MAX_THREADS_PROPAGATION, 1);							// ������ ��������
		dim3 grid_propagation( (int) nx / MAX_THREADS_PROPAGATION, (int) ny / MAX_THREADS_PROPAGATION, 1 );		// ������� ��������� ����� ����� ������� ��� �����������
		
		propagate<<<grid_propagation, threads_propagation>>>((cusp::complex<double>*) wave_out + nx * ny * kz, (cusp::complex<double>*) wave_out + nx * ny * (kz + 1), microscope->getLambda(), (kz + 1) * dz, nx / dpa );
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// phi_n+1(x, y) = FFT^(-1) { p_n(x, y, dZ) * [ t_n(x, y) * phi_n(x, y) ] }
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_backward_propagate = fftw_plan_dft_2d((int) nx, (int) ny, wave_out + nx * ny * (kz + 1), wave_in  + nx * ny * (kz + 1), FFTW_BACKWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_backward_propagate);
		fftw_destroy_plan(fftw_backward_propagate);
		
		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_in + nx * ny * (kz + 1), nx);
		hipDeviceSynchronize();

 		/////////////////////////////////////////////////////////////////////////////////////////////////////////
	 	/// H(k) * PHI(k)
	 	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	 	const double imageSizeAngstrems = nx / dpa; // ������ ����� ����������� � ����������
	 	const double lambda = microscope->getLambda();
	 	const double Cs = microscope->getCs();
	 	const double aperture = microscope->getAperture();
	 	const double defocus = microscope->getDefocus();
	 	//double Z = 0;
	 
	 	const unsigned int MAX_THREADS_OBJECT_LENS = 16;
	 	dim3 threads_object_lens(MAX_THREADS_OBJECT_LENS, MAX_THREADS_OBJECT_LENS, 1);							// ������ ��������
	 	dim3 grid_object_lens( (int) nx / MAX_THREADS_OBJECT_LENS, (int) ny / MAX_THREADS_OBJECT_LENS, 1 );		// ������� ��������� ����� ����� ������� ��� �����������
	 
	 	objectLens<<<grid_object_lens, threads_object_lens>>>((cusp::complex<double>*) wave_out + nx * ny * kz, lambda, Cs, aperture, defocus, imageSizeAngstrems);
	 	hipDeviceSynchronize();
 
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// phi(x, y) = FFT^(-1) { PHI(k) }
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_backward = fftw_plan_dft_2d((int) nx, (int) ny, wave_out + nx * ny * kz, wave_in + nx * ny * kz, FFTW_BACKWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_backward);
		fftw_destroy_plan(fftw_backward);
		
		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_in + nx * ny * kz, nx);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////

		Image::copyFFTtoImage<double>(result, wave_in + nx * ny * kz, kz);

		/// !!!!!!!!!!!!!!!!!!!!
		//Z = 10;
	}
		
	hipFree(wave_in);
	hipFree(wave_out);
	return 0;
}