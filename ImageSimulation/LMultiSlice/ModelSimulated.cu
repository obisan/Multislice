#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelSimulated.h"
#include "kernel.cuh"

ModelSimulated::ModelSimulated(void) {
	
}

ModelSimulated::~ModelSimulated(void) {
	if(this->modelPotential != nullptr) this->modelPotential = nullptr;
}

ModelSimulated::ModelSimulated(ModelPotential* modelPotential, size_t nx, size_t ny, size_t nz, double dpa) {
	this->modelPotential = modelPotential;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->dpa = dpa;
}

int ModelSimulated::imageCalculation(Image *result, Microscope *microscope) {
	const double keV = microscope->getKeV();
	double *potential = modelPotential->potential;
	
	fftw_complex *pfftw_in  = nullptr; 
 	fftw_complex *pfftw_out = nullptr; 
 	hipMallocManaged(&(pfftw_in),  nx * ny * sizeof(fftw_complex));
 	hipMallocManaged(&(pfftw_out), nx * ny * sizeof(fftw_complex));

	for(size_t i = 0; i < nx * ny; i++) {
		pfftw_in[i][0] = 1.0;	
		pfftw_in[i][1] = 0.0;
	}
	
	for(size_t kz = 0; kz < nz; kz++) {	
		for(size_t iy = 0; iy < ny; iy++) {
			for(size_t jx = 0; jx < nx; jx++) {
				////////////////////////////////////////////////////////////////////////////////////////////////////////
				/// T(x, y) = exp(sigma * p(x, y))
				////////////////////////////////////////////////////////////////////////////////////////////////////////
				double fi_re = cos(microscope->getSigma() * potential[ nx * ny * kz + nx * iy + jx ] / 1000.0); // k - eV
				double fi_im = sin(microscope->getSigma() * potential[ nx * ny * kz + nx * iy + jx ] / 1000.0);
				
				std::complex<double> fi(fi_re, fi_im);
				std::complex<double> fi2(pfftw_in[nx * iy + jx][0], pfftw_in[nx * iy + jx][1]);

				/////////////////////////////////////////////////////////////////////////////////////////////////////////
				/// [ T(x, y) * phi(x, y) ]
				/////////////////////////////////////////////////////////////////////////////////////////////////////////
				pfftw_in[nx * iy + jx][0]	= (fi * fi2).real();
				pfftw_in[nx * iy + jx][1]	= (fi * fi2).imag();
			}
		}
		
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		///// PHI(k) = FT [ phi(x, y) ]
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_forward = fftw_plan_dft_2d( (int) nx, (int) ny, pfftw_in, pfftw_out, FFTW_FORWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_forward);
		fftw_destroy_plan(fftw_forward);

		for(size_t i = 0; i < nx * ny; i++) {
			pfftw_out[i][0] /= (double) nx;
			pfftw_out[i][1] /= (double) nx;
		}

		/////////////////////////////////////////////////////////////////////////////////////////////////
		/// Rearrangement 
		///	4 3  to 2 1 
		/// 1 2     3 4
		/////////////////////////////////////////////////////////////////////////////////////////////////
		const size_t nx2 = nx / 2;
		const size_t ny2 = ny / 2;
		for(size_t iy = 0; iy < ny2; iy++) {
			for(size_t jx = 0; jx < nx2; jx++) {
				// 4 - 2
				swap<double>(pfftw_out[iy * nx + jx][0], pfftw_out[(iy + ny2) * nx + nx2 + jx][0]);
				swap<double>(pfftw_out[iy * nx + jx][1], pfftw_out[(iy + ny2) * nx + nx2 + jx][1]);

				// 1 - 3
				swap<double>(pfftw_out[((ny2 - 1 - iy) + ny2) * nx + jx][0], pfftw_out[(ny2 - 1 - iy) * nx + nx2 + jx][0]);
				swap<double>(pfftw_out[((ny2 - 1 - iy) + ny2) * nx + jx][1], pfftw_out[(ny2 - 1 - iy) * nx + nx2 + jx][1]);
			}
		}

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// H(k) * PHI(k)
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		const double dImgSize = nx / dpa; // ������ ����� ����������� � ����������
		//double Z = 0;
		
		for(size_t iy = 0; iy < ny; iy++) {
			for(size_t jx = 0; jx < nx; jx++) {
				///////////////////////////////////////////////////////////////////////////////////
				double u1 = fabs(ny / 2.0 - iy) / dImgSize;
				double u2 = fabs(nx / 2.0 - jx) / dImgSize;
				double k = u1 * u1 + u2 * u2;
				double alpha = microscope->alpha(k);
				double Es = microscope->Es(k);
				std::complex<double> w1(Es * cos(alpha), Es * sin(alpha));
				std::complex<double> w2(pfftw_out[iy * nx + jx][0], pfftw_out[iy * nx + jx][1]);
				pfftw_out[iy * nx + jx][0] = (w1 * w2).real();
				pfftw_out[iy * nx + jx][1] = (w1 * w2).imag();
			}
		}

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// phi(x, y) = FT^(-1) { PHI(k) }
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_backward = fftw_plan_dft_2d((int) nx, (int) ny, pfftw_out, pfftw_in, FFTW_BACKWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_backward);
		fftw_destroy_plan(fftw_backward);
		
		for(size_t i = 0; i < nx * ny; i++) {
			pfftw_in[i][0] /= (double) nx;
			pfftw_in[i][1] /= (double) nx;
		}
		
		/// !!!!!!!!!!!!!!!!!!!!
		//Z = 10;
	}

	Image::copyFFTtoImage<double>(result, pfftw_in, 0);

	hipFree(pfftw_in);
	hipFree(pfftw_out);
	return 0;
}