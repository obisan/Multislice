#include "hip/hip_runtime.h"
﻿#include "stdafx.h"
#include "ModelSimulated.h"
#include "kernel.cuh"

ModelSimulated::ModelSimulated(void) {
	
}

ModelSimulated::~ModelSimulated(void) {
	if(this->model != nullptr) this->model = nullptr;
}

ModelSimulated::ModelSimulated(const char* potentialDirectory, AModel::Model* model, int nx, int ny, int countSlices, double dpa) {
	this->nx = nx;
	this->ny = ny;
	this->countSlices = countSlices;
	this->dpa = dpa;

	this->model = model;

	for(size_t i = 0; i < countSlices; i++) {
		char slicename[256];
		sprintf(slicename, "%s/slice%003u.slc", potentialDirectory, i);
		std::string slice(slicename);
		slices.push_back(slicename);
	}
}

int ModelSimulated::imageCalculation(Image *result, Microscope *microscope) {
	fftw_complex *wave_in; 
 	fftw_complex *wave_out; 
	fftw_complex *wave_next;
 	hipMallocManaged(&(wave_in),	nx * ny * sizeof(fftw_complex));
	CUERR
 	hipMallocManaged(&(wave_out),	nx * ny * sizeof(fftw_complex));
	CUERR
	hipMallocManaged(&(wave_next), nx * ny * sizeof(fftw_complex));
	CUERR

	double *potentialSlice;
	hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double));
	memset(potentialSlice, 0, nx * ny * sizeof(double));
	CUERR


	for(size_t i = 0; i < nx * ny; i++) {
		wave_in[i][0] = 1.0;	
		wave_in[i][1] = 0.0;
	}
		
	hipEvent_t start,stop;
	float time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	

	double dz = this->model->getC() / countSlices;
	for(size_t kz = 0; kz < countSlices; kz++) {	
		FILE *pFile;
		pFile = fopen(slices[kz].c_str(), "rb");
		fread(potentialSlice, sizeof(double), nx * ny, pFile);
		fclose(pFile);
		
		////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// t(x, y) = exp(sigma * potential(x, y))
		/// [ t(x, y) * phi(x, y) ]
		////////////////////////////////////////////////////////////////////////////////////////////////////////
		
		const unsigned int MAX_THREADS_PHASE_OBJECT = 16;
		dim3 threads_phase(MAX_THREADS_PHASE_OBJECT, MAX_THREADS_PHASE_OBJECT, 1);							// ðàçìåð êâàäðàòà
		dim3 grid_phase( (int) nx / MAX_THREADS_PHASE_OBJECT, (int) ny / MAX_THREADS_PHASE_OBJECT, 1 );		// ñêîëüêî êâàäðàòîâ íóæíî ÷òîáû ïîêðûòü âñå èçîáðàæåíèå

		phaseObject<<<grid_phase, threads_phase>>>(potentialSlice, (cusp::complex<double>*) wave_in, nx, ny, microscope->getSigma());
		hipDeviceSynchronize();

		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		///// PHI(k) = FFT [ t_n(x, y) * phi_n(x, y) ]
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_forward = fftw_plan_dft_2d( (int) nx, (int) ny, wave_in, wave_out, FFTW_FORWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_forward);
		fftw_destroy_plan(fftw_forward);

		const unsigned int MAX_THREADS_NORMALIZE = 1024;
		dim3 threads_normalize(MAX_THREADS_NORMALIZE, 1, 1);
		dim3 grid_normalize( (int) nx * ny / MAX_THREADS_NORMALIZE, 1, 1 );

		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_out, nx);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////
		/// Rearrangement 
		///	4 3  to 2 1 
		/// 1 2     3 4
		/////////////////////////////////////////////////////////////////////////////////////////////////
		const unsigned int MAX_THREADS_REARRANGEMENT = 16;
		dim3 threads_rearrangement(MAX_THREADS_REARRANGEMENT, MAX_THREADS_REARRANGEMENT, 1);									// ðàçìåð êâàäðàòà
		dim3 grid_rearrangement( (int) nx / 2 / MAX_THREADS_REARRANGEMENT, (int) ny / 2 / MAX_THREADS_REARRANGEMENT, 1 );		// ñêîëüêî êâàäðàòîâ íóæíî ÷òîáû ïîêðûòü âñå èçîáðàæåíèå

		rearrangement<<<grid_rearrangement, threads_rearrangement>>>((cusp::complex<double>*) wave_out);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// p_n(x, y, dZ) * FFT [ t_n(x, y) * phi_n(x, y) ]
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		const unsigned int MAX_THREADS_PROPAGATION = 16;
		dim3 threads_propagation(MAX_THREADS_PROPAGATION, MAX_THREADS_PROPAGATION, 1);							// ðàçìåð êâàäðàòà
		dim3 grid_propagation( (int) nx / MAX_THREADS_PROPAGATION, (int) ny / MAX_THREADS_PROPAGATION, 1 );		// ñêîëüêî êâàäðàòîâ íóæíî ÷òîáû ïîêðûòü âñå èçîáðàæåíèå
		
		propagate<<<grid_propagation, threads_propagation>>>((cusp::complex<double>*) wave_out, (cusp::complex<double>*) wave_next, microscope->getLambda(), (kz + 1) * dz, nx / dpa );
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		/// phi_n+1(x, y) = FFT^(-1) { p_n(x, y, dZ) * [ t_n(x, y) * phi_n(x, y) ] }
		/////////////////////////////////////////////////////////////////////////////////////////////////////////
		fftw_plan fftw_backward_propagate = fftw_plan_dft_2d((int) nx, (int) ny, wave_next, wave_in, FFTW_BACKWARD, FFTW_ESTIMATE);
		fftw_execute(fftw_backward_propagate);
		fftw_destroy_plan(fftw_backward_propagate);
		
		normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_in, nx);
		hipDeviceSynchronize();

		/////////////////////////////////////////////////////////////////////////////////////////////////////////

		//Image::copyFFTtoImage<double>(result, wave_in + nx * ny * kz, kz);

		/// !!!!!!!!!!!!!!!!!!!!
		//Z = 10;
	}
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	///// PHI(k) = FFT [ phi_exit(x, y) ]
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	fftw_plan fftw_forward = fftw_plan_dft_2d( (int) nx, (int) ny, wave_in, wave_out, FFTW_FORWARD, FFTW_ESTIMATE);
	fftw_execute(fftw_forward);
	fftw_destroy_plan(fftw_forward);

	const unsigned int MAX_THREADS_NORMALIZE = 1024;
	dim3 threads_normalize(MAX_THREADS_NORMALIZE, 1, 1);
	dim3 grid_normalize( (int) nx * ny / MAX_THREADS_NORMALIZE, 1, 1 );

	normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_out, nx);
	hipDeviceSynchronize();

	/////////////////////////////////////////////////////////////////////////////////////////////////
	/// Rearrangement 
	///	4 3  to 2 1 
	/// 1 2     3 4
	/////////////////////////////////////////////////////////////////////////////////////////////////
	const unsigned int MAX_THREADS_REARRANGEMENT = 16;
	dim3 threads_rearrangement(MAX_THREADS_REARRANGEMENT, MAX_THREADS_REARRANGEMENT, 1);									// размер квадрата
	dim3 grid_rearrangement( (int) nx / 2 / MAX_THREADS_REARRANGEMENT, (int) ny / 2 / MAX_THREADS_REARRANGEMENT, 1 );		// сколько квадратов нужно чтобы покрыть все изображение

	rearrangement<<<grid_rearrangement, threads_rearrangement>>>((cusp::complex<double>*) wave_out);
	hipDeviceSynchronize();

	///////////////////////////////////////////////////////////////////////////////////////////////////////
	/// H(k) * PHI(k)
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	const double imageSizeAngstrems = nx / dpa; // ðàçìåð âñåãî èçîáðàæåíèÿ â àíãñòðåìàõ
	const double lambda = microscope->getLambda();
	const double Cs = microscope->getCs();
	const double aperture = microscope->getAperture();
	const double defocus = microscope->getDefocus();
	//double Z = 0;
	 
	const unsigned int MAX_THREADS_OBJECT_LENS = 16;
	dim3 threads_object_lens(MAX_THREADS_OBJECT_LENS, MAX_THREADS_OBJECT_LENS, 1);							// ðàçìåð êâàäðàòà
	dim3 grid_object_lens( (int) nx / MAX_THREADS_OBJECT_LENS, (int) ny / MAX_THREADS_OBJECT_LENS, 1 );		// ñêîëüêî êâàäðàòîâ íóæíî ÷òîáû ïîêðûòü âñå èçîáðàæåíèå
	 
	objectLens<<<grid_object_lens, threads_object_lens>>>((cusp::complex<double>*) wave_out, lambda, Cs, aperture, defocus, imageSizeAngstrems);
	hipDeviceSynchronize();
 
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	/// phi(x, y) = FFT^(-1) { PHI(k) }
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	fftw_plan fftw_backward = fftw_plan_dft_2d((int) nx, (int) ny, wave_out, wave_in, FFTW_BACKWARD, FFTW_ESTIMATE);
	fftw_execute(fftw_backward);
	fftw_destroy_plan(fftw_backward);
		
	normalize<<<grid_normalize, threads_normalize>>>((cusp::complex<double>*) wave_in, nx);
	hipDeviceSynchronize();

	Image::copyFFTtoImage<double>(result, wave_in, 0);
	
	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	std::cout << std::endl << "Kernel time calculating electron image image: " << time << "ms." << std::endl << std::endl;
	

	hipFree(wave_in);
	hipFree(wave_out);
	hipFree(wave_next);
	


	return 0;
}