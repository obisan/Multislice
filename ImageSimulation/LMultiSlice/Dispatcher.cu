#include "stdafx.h"
#include "Dispatcher.h"
//#include "ModelPotential.h"
#include "ModelSimulated.h"

Dispatcher::Dispatcher(void) {

}

Dispatcher::~Dispatcher(void) {

}

bool Dispatcher::isFileExist(const char *fname) {
	_finddata_t data;
	intptr_t nFind = _findfirst(fname,&data);
	if (nFind != -1) {
		// ���� ����� �� �������, �� ���������� ������ ��������
		_findclose(nFind);
		return true;
	}
	return false;
}

bool Dispatcher::isDirectoryExist(const char* dirname) {
	struct stat statbuf;
	if(stat(dirname,&statbuf)) {
		return false;
	}
	return true;
}

int Dispatcher::parseCommand(const char* fileNameXML, Command& command) {
	xml_document doc;
	xml_parse_result result = doc.load_file(fileNameXML);

	if (result)
		std::cout << "XML [" << fileNameXML << "] parsed without errors, attr value: [" << doc.child("node").attribute("attr").value() << "]\n\n";
	else {
		std::cout << "XML [" << fileNameXML << "] parsed with errors, attr value: [" << doc.child("node").attribute("attr").value() << "]\n";
		std::cout << "Error description: " << result.description() << "\n";
		std::cout << "Error offset: " << result.offset << " (error at [..." << (fileNameXML + result.offset) << "]\n\n";
		return -1;
	}

	char buffer[256];

	strcpy(command.fileNameInput, doc.child("action").child("io").child("fileNameInput").child_value());
	if( strlen(this->command.fileNameInput) == 0 ) {
		std::cerr << "Empty \"file input\" field!" << std::endl;
		return -1;
	}

	strcpy(command.potentialDirectory, doc.child("action").child("io").child("fileNameOutput").child_value());
	if( strlen(command.fileNameOutput) == 0 ) {
		std::cerr << "Empty \"file output\" field!" << std::endl;
		return -1;
	}

	strcpy(command.fileNameOutput, doc.child("action").child("io").child("PotentialDirectory").child_value());
	if( strlen(command.potentialDirectory) == 0 ) {
		std::cerr << "Empty \"Potential Directory\" field!" << std::endl;
		return -1;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////////////
	
	strcpy(buffer, doc.child("action").child("image").child("nx").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"nx\" field!" << std::endl;
		return -1;
	} try {
		command.nx = atoi(buffer);
	} catch(...) {
		std::cerr << "Convert \"nx\" problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("image").child("ny").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"ny\" field!" << std::endl;
		return -1;
	} try {
		command.ny = atoi(buffer);
	} catch(...) {
		std::cerr << "Convert \"ny\" problems!" << std::endl;
		return -1;
	}
	
	strcpy(buffer, doc.child("action").child("image").child("dpa").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"dpa\" field!" << std::endl;
		return -1;
	} try {
		command.dpa = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert \"dpa\" problems!" << std::endl;
		return -1;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////////////

	strcpy(buffer, doc.child("action").child("slicing").child("radius").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"radius\" field!" << std::endl;
		return -1;
	} try {
		command.radius = (float) atof(buffer);

		if(command.radius > 20.0f) {
			std::cerr << "Radius is too big, it was changed on 20A!" << std::endl;
			command.bindim = 20.0f;
		}
	} catch(...) {
		std::cerr << "Convert \"radius\" problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("slicing").child("bindim").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"binsize\" field!" << std::endl;
		return -1;
	} try {
		command.bindim = atoi(buffer);

		if(command.bindim < 10.0f) {
			std::cerr << "Bin size too small, it was changed on 10A!" << std::endl;
			command.bindim = 10.0f;
		}
	} catch(...) {
		std::cerr << "Convert \"bindim\" problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("slicing").child("numberslices").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"numberslices\" field!" << std::endl;
		return -1;
	} try {
		command.numberSlices = atoi(buffer);
	} catch(...) {
		std::cerr << "Convert \"numberslices\" problems!" << std::endl;
		return -1;
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////

	strcpy(buffer, doc.child("action").child("microscope").child("aperture").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"aperture\" field!" << std::endl;
		return -1;
	} try {
		command.aperture = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert \"aperture\" problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("microscope").child("cs").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"cs\" field!" << std::endl;
		return -1;
	} try {
		command.cs = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert \"cs\" problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("microscope").child("defocus").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"defocus\" field!" << std::endl;
		return -1;
	} try {
		command.defocus = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert \"defocus\" problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("microscope").child("keV").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty \"keV\" field!" << std::endl;
		return -1;
	} try {
		command.keV = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert \"keV\" problems!" << std::endl;
		return -1;
	}


	return 0;
}

int Dispatcher::Run(const char* fileNameXML) {
	if(!isFileExist(fileNameXML)) {
		std::cerr << "XML File with name [" << fileNameXML << "] doesn't exist." << std::endl;
		return -1;
	} else {
		std::cout << "XML File with name [" << fileNameXML << "] exist." << std::endl;
	}
	
	if( parseCommand(fileNameXML, command) == -1) {
		return -1;
	}
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	printf("\nDetected %d CUDA accelerators:\n", deviceCount);
	int dev;
	for (dev=0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n", 
			dev, deviceProp.name, 
			deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
			deviceProp.major, deviceProp.minor);
	}

	int cudadev = 0;
	printf("  Single-threaded single-GPU test run.\n");
	printf("  Opening CUDA device %d...\n\n", cudadev);
	hipSetDevice(cudadev);
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////

	AModel::Model *model = getModelType(command.fileNameInput);
	
	if(!isDirectoryExist(command.potentialDirectory)) {
		if( model->read(command.fileNameInput) == -1 ) {
			std::cout << "Can not read file [" << command.fileNameInput << "] !!!" << std::endl;
			return -1;
		} else {
			std::cout << "Read file model [" << command.fileNameInput << "] successful." << std::endl;
		} 

		//////////////////////////////////////////////////////////////////////////
		// Calculating map potentials	//////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////
		PotentialBuilder::ModelPotential *modelPotential 
			= new PotentialBuilder::ModelPotential(model, command.nx, command.ny, command.numberSlices, command.radius, command.bindim, command.potentialDirectory);
		if(modelPotential->calculatePotentialGrid() == -1) 
			return -1;
		modelPotential->savePotentialStack(command.fileNameOutput, command.potentialDirectory);
		delete modelPotential;
	} else {
		if( model->readhead(command.fileNameInput) == -1 ) {
			std::cout << "Can not read file [" << command.fileNameInput << "] !!!" << std::endl;
			return -1;
		} else {
			std::cout << "Read file model [" << command.fileNameInput << "] successful." << std::endl;
		} 
	}

	//////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////
	
	std::cout << std::endl;
	std::cout << "Image size		= " << command.nx << "x" << command.ny << std::endl;
	std::cout << "Number of slices	= " << command.numberSlices << std::endl;
	std::cout << "Number of atoms	= " << model->getNumberAtoms() << std::endl;
	std::cout << "Dots per atom		= " << command.dpa << std::endl;
	

	ModelSimulated *modelSimulated = new ModelSimulated(command.potentialDirectory, model, command.nx, command.ny, command.numberSlices, command.dpa);
	Microscope *microscope = new Microscope(command.keV, command.cs, command.aperture, command.defocus);
	Image *result = new Image(command.nx, command.ny, 1, sizeof(double), 2);
	
	modelSimulated->imageCalculation(result, microscope);
	Image *result_module = result->getModule();
	result_module->saveMRC(command.fileNameOutput, model, command.nx, command.ny, 1, mrc_FLOAT);

	delete result_module;
	delete result;
	delete microscope;
	delete modelSimulated;

	delete model;

	//////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////
	hipDeviceReset();

	std::cout	<< "Calculation for [" << fileNameXML <<  "] finished successful." << std::endl << std::endl;

	return 0;
}