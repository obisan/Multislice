#include "stdafx.h"
#include "Dispatcher.h"
#include "ModelPotential.h"
#include "ModelSimulated.h"

Dispatcher::Dispatcher(void) {

}

Dispatcher::~Dispatcher(void) {

}

bool Dispatcher::CheckFileExist(const char *fname) {
	_finddata_t data;
	intptr_t nFind = _findfirst(fname,&data);
	if (nFind != -1) {
		// ���� ����� �� �������, �� ���������� ������ ��������
		_findclose(nFind);
		return true;
	}
	return false;
}

int Dispatcher::parseCommand(const char* fileNameXML, Command& command) {
	xml_document doc;
	xml_parse_result result = doc.load_file(fileNameXML);

	if (result)
		std::cout << "XML [" << fileNameXML << "] parsed without errors, attr value: [" << doc.child("node").attribute("attr").value() << "]\n\n";
	else {
		std::cout << "XML [" << fileNameXML << "] parsed with errors, attr value: [" << doc.child("node").attribute("attr").value() << "]\n";
		std::cout << "Error description: " << result.description() << "\n";
		std::cout << "Error offset: " << result.offset << " (error at [..." << (fileNameXML + result.offset) << "]\n\n";
		return -1;
	}

	char buffer[256];

	strcpy(command.fileNameInput, doc.child("action").child("io").child("fileNameInput").child_value());
	if( strlen(this->command.fileNameInput) == 0 ) {
		std::cerr << "Empty file input field!" << std::endl;
		return -1;
	}

	strcpy(command.fileNameOutput, doc.child("action").child("io").child("fileNameOutput").child_value());
	if( strlen(command.fileNameOutput) == 0 ) {
		std::cerr << "Empty file output field!" << std::endl;
		return -1;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////////////
	
	strcpy(buffer, doc.child("action").child("image").child("nx").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty nx field!" << std::endl;
		return -1;
	} try {
		command.nx = atoi(buffer);
	} catch(...) {
		std::cerr << "Convert nx problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("image").child("ny").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty ny field!" << std::endl;
		return -1;
	} try {
		command.ny = atoi(buffer);
	} catch(...) {
		std::cerr << "Convert ny problems!" << std::endl;
		return -1;
	}
	
	strcpy(buffer, doc.child("action").child("image").child("dpa").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty dpa field!" << std::endl;
		return -1;
	} try {
		command.dpa = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert dpa problems!" << std::endl;
		return -1;
	}

	////////////////////////////////////////////////////////////////////////////////////////////////////////////

	strcpy(buffer, doc.child("action").child("slicing").child("radiuc").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty radiuc field!" << std::endl;
		return -1;
	} try {
		command.radiuc = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert radiuc problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("slicing").child("numberslices").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty numberslices field!" << std::endl;
		return -1;
	} try {
		command.numberSlices = atoi(buffer);
	} catch(...) {
		std::cerr << "Convert numberslices problems!" << std::endl;
		return -1;
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////

	strcpy(buffer, doc.child("action").child("microscope").child("aperture").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty aperture field!" << std::endl;
		return -1;
	} try {
		command.aperture = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert aperture problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("microscope").child("cs").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty cs field!" << std::endl;
		return -1;
	} try {
		command.cs = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert cs problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("microscope").child("defocus").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty defocus field!" << std::endl;
		return -1;
	} try {
		command.defocus = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert defocus problems!" << std::endl;
		return -1;
	}

	strcpy(buffer, doc.child("action").child("microscope").child("keV").child_value() );
	if( strlen(buffer) == 0 ) {
		std::cerr << "Empty keV field!" << std::endl;
		return -1;
	} try {
		command.keV = (float) atof(buffer);
	} catch(...) {
		std::cerr << "Convert keV problems!" << std::endl;
		return -1;
	}


	return 0;
}

int Dispatcher::Run(const char* fileNameXML) {
	if(!CheckFileExist(fileNameXML)) {
		std::cerr << "XML File with name [" << fileNameXML << "] doesn't exist." << std::endl;
		return -1;
	} else {
		std::cout << "XML File with name [" << fileNameXML << "] exist." << std::endl;
	}
	
	if( parseCommand(fileNameXML, command) == -1) {
		return -1;
	}

	AModel::Model *model = getModelType(command.fileNameInput);
	if( model->read(command.fileNameInput) == -1 ) {
		std::cout << "Can not read file " << command.fileNameInput << "!!!" << std::endl;
		return -1;
	} else {
		std::cout << "Read file model [" << command.fileNameInput << "] successful." << std::endl;
	} 

	/************************************************************************/
	/* Calculating map potentials	*****************************************/
	/************************************************************************/
	std::cout << std::endl;
	std::cout << "Image size		= " << command.nx << "x" << command.ny << std::endl;
	std::cout << "Number of slices	= " << command.numberSlices << std::endl;
	std::cout << "Number of atoms	= " << model->getNumberAtoms() << std::endl;
	std::cout << "dots per atom		= " << command.dpa << std::endl;

	///////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	printf("\nDetected %d CUDA accelerators:\n", deviceCount);
	int dev;
	for (dev=0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n", 
			dev, deviceProp.name, 
			deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
			deviceProp.major, deviceProp.minor);
	}

	int cudadev = 0;
	printf("  Single-threaded single-GPU test run.\n");
	printf("  Opening CUDA device %d...\n\n", cudadev);
	hipSetDevice(cudadev);
	///////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////

	ModelPotential *modelPotential = new ModelPotential(model, command.nx, command.ny, command.numberSlices, command.dpa, command.radiuc);
	if(modelPotential->calculatePotentialGrid() == -1) 
		return -1;
	
	modelPotential->savePotential(command.fileNameOutput);

	ModelSimulated *modelSimulated = new ModelSimulated(modelPotential, command.nx, command.ny, command.numberSlices, command.dpa);
	Microscope *microscope = new Microscope(command.keV, command.cs, command.aperture, command.defocus);
	Image *result = new Image(command.nx, command.ny, 1, sizeof(double), 2);
	
	modelSimulated->imageCalculation(result, microscope);
	Image *result_module = result->getModule();
	result_module->saveMRC(command.fileNameOutput, model, command.nx, command.ny, 1, mrc_FLOAT);

	delete result_module;
	delete result;
	delete microscope;
	delete modelSimulated;

	delete modelPotential;
	
	delete model;

	/************************************************************************/
	/************************************************************************/
	/************************************************************************/
	hipDeviceReset();

	std::cout	<< "Calculation for [" << fileNameXML <<  "] finished successful." << std::endl << std::endl;

	return 0;
}