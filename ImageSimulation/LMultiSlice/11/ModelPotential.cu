#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"

struct __align__(16) atom {
	int id;
	int num;
	float x;
	float y;
};

__global__ void calculatePotentialGridGPU(double *potential, int *bins_offset, int *bins_num, atom *bins_d, unsigned short *bins_lattice);

__device__ double	bessk0( double x );
__device__ double	bessi0( double x );
__device__ void		swap2(double& a, double& b);


ModelPotential::ModelPotential(void) {

}

ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double radius, double bindim) {
	this->model = model;
	this->nx = nx;
	this->ny = ny;
	this->nz = nz;
	this->radius = radius;
	this->bindim = bindim;
	
	this->potential = (double*) malloc(nx * ny * nz * sizeof(double));
	memset(this->potential, 0, nx * ny * nz * sizeof(double));
}

ModelPotential::~ModelPotential(void) {
	if(this->model != nullptr) { model = nullptr; }
	if(this->potential != nullptr) { free(this->potential); }
}

int ModelPotential::calculatePotentialGrid() {
	const size_t nAtoms = model->getNumberAtoms();
	const double a_h = model->getA();
	const double b_h = model->getB();
	const double c_h = model->getC();
	const double dx = a_h / this->nx;
	const double dy = b_h / this->ny;
	const double dz = c_h / this->nz;
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(radius_d), &radius, sizeof(double)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dx_d), &dx, sizeof(double)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dy_d), &dy, sizeof(double)) );
	
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(a_d), &a_h, sizeof(double)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(b_d), &b_h, sizeof(double)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(c_d), &c_h, sizeof(double)) );

	double bindimx = this->bindim; // angstrem
	double bindimy = this->bindim; // angstrem
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(bindimx_d), &bindimx, sizeof(double)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(bindimy_d), &bindimy, sizeof(double)) );

	int	binx = ceil(a_h / bindimx); // dimensionless
	int	biny = ceil(b_h / bindimy); // dimensionless
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(binx_d), &binx, sizeof(int)) );
	checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(biny_d), &biny, sizeof(int)) );

	//////////////////////////////////////////////////////////////////////////////////////////////////////

	double *potentialSlice;
	checkCudaErrors( hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double)));
	memset(potentialSlice, 0, nx * ny * sizeof(double));
	CUERR
	
	int *bins_offset;
	checkCudaErrors( hipMallocManaged(&(bins_offset), (binx * biny + 1) * sizeof(int)));
	memset(bins_offset, 0, (binx * biny + 1) * sizeof(int));
	CUERR

	int *bins_num;
	checkCudaErrors( hipMallocManaged(&(bins_num), (nx * ny) * sizeof(int)));
	memset(bins_num, 0, (nx * ny) * sizeof(int));
	CUERR


	//////////////////////////////////////////////////////////////////////////////////////////////////////

	dim3 threads(BLOCKSIZEX, BLOCKSIZEY, 1);		
	dim3 grid(this->nx / BLOCKSIZEX, this->ny / BLOCKSIZEY, 1 );

	AModel::Cortege *pAtoms = model->getTableCell();
	std::sort(pAtoms, pAtoms + nAtoms);

	std::vector<atom> slice;
	std::vector<atom> *bins = new std::vector<atom> [biny * binx];
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	float time_kernel = 0.0f;
	float time_total = 0.0f;
	hipEvent_t start_total,stop_total;
	hipEventCreate(&start_total);
	hipEventCreate(&stop_total);
	hipEventRecord(start_total,0);
	
	//////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	clock_t time_lattice = clock();
	
	unsigned short *bins_lattice;
	checkCudaErrors( hipMallocManaged(&(bins_lattice), (nx * ny * MAX_BINS_PER_PX) * sizeof(unsigned short)));
	memset(bins_lattice, -1, (nx * ny * MAX_BINS_PER_PX) * sizeof(unsigned short));

	for(size_t iy = 0; iy < ny; iy++) {
		int coordbinstarty	= floor(iy * ((double) biny / ny) - radius / bindimy ) - 1;		// iy in px
		int coordbinendy	= ceil(iy * ((double) biny / ny) + radius / bindimy ) - 1;

		for(size_t ix = 0; ix < nx; ix++) {                                 // jx in px
			int coordbinstartx	= floor(ix * ((double) binx / nx) - radius / bindimx ) - 1;
			int coordbinendx	= ceil(ix * ((double) binx / nx) + radius / bindimx ) - 1;
			
			int k = 0;
			for(int i = coordbinstarty; i <= coordbinendy && k < MAX_BINS_PER_PX; i++) {
				int bincurr_y = (i + biny) % biny;
				for(int j = coordbinstartx; j <= coordbinendx && k < MAX_BINS_PER_PX; j++, k++) {
					int bincur_x = (j + binx) % binx;
					bins_lattice[ MAX_BINS_PER_PX * (nx * iy + ix) + k ] = binx * bincurr_y + bincur_x;
					bins_num[nx * iy + ix] = k + 1;
				}
			}
		}
	}

	std::cout << "calculate lattice-to-bin: " << (clock() - time_lattice) / CLOCKS_PER_SEC << "s." << std::endl;

	//////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////////////////////

	for(size_t kz = 0; kz < nz; kz++) {
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////	Divide on slices /////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		for(size_t i = 0; i < nAtoms; i++) {
			if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c_h && pAtoms[i].element.xsCoordinate.z * c_h <= (kz + 1) * dz ) {
				atom buff;
 				buff.id = i + 1;
				buff.num = model->getNumberByName(pAtoms[i].element.Atom) - 1;
				buff.x = pAtoms[i].element.xsCoordinate.x;
				buff.y = pAtoms[i].element.xsCoordinate.y;

				slice.push_back(buff);
			}
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////	Divide on bins	//////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		for(size_t iy = 0; iy < biny; iy++) {
			for(size_t jx = 0; jx < binx; jx++) {
				for(auto t : slice) {
					if( t.x * a_h >= jx * bindimx && t.x * a_h <= (jx + 1) * bindimx )
						if( t.y * b_h >= iy * bindimy && t.y * b_h <= (iy + 1) * bindimy )
							bins[binx * iy + jx].push_back(t);
				}
			}
		}

		atom *bins_d;
		checkCudaErrors( hipMallocManaged(&(bins_d), slice.size() * sizeof(atom)));

		bins_offset[0] = 0;
		for(size_t i = 0, k = 0; i < binx * biny; i++) {
			bins_offset[i + 1] = bins[i].size();
			for(auto s : bins[i]) {
				bins_d[k] = s;
				k++;
			}
			bins[i].clear();
		}

		// make number to offset
		for(int i = 2; i < binx * biny + 1; i++) {
			bins_offset[i] = bins_offset[i] + bins_offset[i - 1];
		}

		hipEvent_t start,stop;
		float ctime = 0.0f;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);

		calculatePotentialGridGPU<<<grid, threads>>>(potentialSlice, bins_offset, bins_num, bins_d, bins_lattice);
		checkCudaErrors( hipDeviceSynchronize() );

		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ctime, start, stop);
		time_kernel += ctime;

		std::cout << "slice: " << kz << std::endl << "calculated atoms: " << slice.size() << std::endl;

		hipMemcpy(potential + nx * ny * kz, potentialSlice, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
		
		slice.clear();	
		hipFree(bins_d);
	}

	hipFree(bins_num);
	hipFree(bins_lattice);
	hipFree(bins_offset);

	hipEventRecord(stop_total,0);
	hipEventSynchronize(stop_total);
	hipEventElapsedTime(&time_total, start_total, stop_total);

	pAtoms = nullptr;
	
	std::cout << std::endl;
	std::cout << "Kernel time calculating potential grid: " << time_kernel	<< "ms." << std::endl;
	std::cout << "Total  time calculating potential grid: " << time_total	<< "ms." << std::endl << std::endl;
	

	return 0;
}

__global__ void calculatePotentialGridGPU(double *potential, int *bins_offset, int *bins_num, atom *bins_d, unsigned short *bins_lattice) {
	const int ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
	const int iy = __umul24(blockDim.y, blockIdx.y) + threadIdx.y;
	const int is = __umul24(blockDim.x, threadIdx.y) + threadIdx.x;
	const int LINESIZE = __umul24(gridDim.x, blockDim.x);
	
	int i,j;
	
	double latticex = ix * dx_d; // lattice x
	double latticey = iy * dy_d; // lattice y
	
	const int numberBins = bins_num[LINESIZE * iy + ix];

	__shared__ double imageval[BLOCKSIZEX*BLOCKSIZEY];
	imageval[is] = 0.0;
	
	for(i = 0; i < numberBins; i++) {
		int ibin = bins_lattice[ MAX_BINS_PER_PX * (LINESIZE * iy + ix) + i];
		
		int n = bins_offset[ibin + 1] - bins_offset[ibin];
		int offset = bins_offset[ibin];

		for(j = 0; j < n; j++) {
			int numberAtom = bins_d[offset + j].num;
			double x = fabs(bins_d[offset + j].x * a_d - latticex);
			double y = fabs(bins_d[offset + j].y * b_d - latticey);

			x = ( x >= a_d / 2.0 ) ? x - a_d : x;
			y = ( y >= b_d / 2.0 ) ? y - b_d : y;

			double r = __dsqrt_rn(x * x + y * y);
			r = (r < 1e-20) ? 1e-20 : r;
			double dR1 = 6.2831853071796 * r; // 2 * PI * r;
		
			imageval[is] += ( 
					FParamsDevice[(numberAtom) * 12 + 0 * 2 + 0] * bessk0(dR1 * __dsqrt_rn(FParamsDevice[(numberAtom) * 12 + 0 * 2 + 1]))
				+	FParamsDevice[(numberAtom) * 12 + 1 * 2 + 0] * bessk0(dR1 * __dsqrt_rn(FParamsDevice[(numberAtom) * 12 + 1 * 2 + 1]))
				+	FParamsDevice[(numberAtom) * 12 + 2 * 2 + 0] * bessk0(dR1 * __dsqrt_rn(FParamsDevice[(numberAtom) * 12 + 2 * 2 + 1])) 
				) * 300.73079394295
				+ (
					(FParamsDevice[(numberAtom) * 12 + 0 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 0 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 0 * 2 + 7])
				+	(FParamsDevice[(numberAtom) * 12 + 1 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 1 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 1 * 2 + 7])
				+	(FParamsDevice[(numberAtom) * 12 + 2 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 2 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 2 * 2 + 7])
				) * 150.36539697148;

		}
	}

	__syncthreads();

	potential[ LINESIZE * iy + ix ] = imageval[is]; 
	
}

__device__ void		swap2(double& a, double& b) {
		double buffer = a;
		a = b;
		b = buffer;
}

__device__ double	bessk0( double ax ) {
	double x2;
	double sum;
		
	if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
		x2 = __ddiv_rn(ax, 2.0);
		x2 = __dmul_rd(x2, x2);
		sum = __fma_rn(k0a[6], x2, k0a[5]);
		sum = __fma_rn(sum, x2, k0a[4]);
		sum = __fma_rn(sum, x2, k0a[3]);
		sum = __fma_rn(sum, x2, k0a[2]);
		sum = __fma_rn(sum, x2, k0a[1]);
		sum = __fma_rn(sum, x2, k0a[0]);



		sum = -log(ax / 2.0) * bessi0( ax ) + sum;


	} else if( ax > 2.0 ) {
		x2 = __ddiv_rn(2.0, ax);
		sum = __fma_rn(k0b[6], x2, k0b[5]);
		sum = __fma_rn(sum, x2, k0b[4]);
		sum = __fma_rn(sum, x2, k0b[3]);
		sum = __fma_rn(sum, x2, k0b[2]);
		sum = __fma_rn(sum, x2, k0b[1]);
		sum = __fma_rn(sum, x2, k0b[0]);

		sum = exp( -ax ) * sum / __dsqrt_rn(ax);
	} else sum = 1.0e20;
	return ( sum );
}

__device__ double	bessi0( double ax ) {
 	double sum;
	double t;
 	
	if( ax <= 3.75 ) {
		t = __ddiv_rn(ax, 3.75);
		t = __dmul_rd(t, t);
		sum = __fma_rn(i0a[6], t, i0a[5]);
		sum = __fma_rn(sum, t, i0a[4]);
		sum = __fma_rn(sum, t, i0a[3]);
		sum = __fma_rn(sum, t, i0a[2]);
		sum = __fma_rn(sum, t, i0a[1]);
		sum = __fma_rn(sum, t, i0a[0]);
	} else {
		t = __ddiv_rn(3.75, ax);
		sum = __fma_rn(i0b[8], t, i0a[7]);
		sum = __fma_rn(sum, t, i0a[6]);
		sum = __fma_rn(sum, t, i0a[5]);
		sum = __fma_rn(sum, t, i0a[4]);
		sum = __fma_rn(sum, t, i0a[3]);
		sum = __fma_rn(sum, t, i0a[2]);
		sum = __fma_rn(sum, t, i0a[1]);
		sum = __fma_rn(sum, t, i0a[0]);

		sum = exp( ax ) * sum / __dsqrt_rn( ax );
	}
	return( sum );
}

int	ModelPotential::savePotential(const char* filename) {
	Image *image = new Image(nx, ny, nz, sizeof(double), 1);
	char filenamept[256];
	strcpy(filenamept, filename);
	strcat(filenamept, "_pt");
	memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
	image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
	delete image;

	return 0;
}

AModel::Model* ModelPotential::getModel() {
	return model;
}

