#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"

namespace PotentialBuilder {
	ModelPotential::ModelPotential(void) {

	}

	ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double radius, double bindim) {
		this->model = model;
		this->nx = nx;
		this->ny = ny;
		this->nz = nz;
		this->radius = radius;
		this->bindim = bindim;

		this->potential = (double*) malloc(nx * ny * nz * sizeof(double));
		memset(this->potential, 0, nx * ny * nz * sizeof(double));
	}

	ModelPotential::~ModelPotential(void) {
		if(this->model != nullptr) { model = nullptr; }
		if(this->potential != nullptr) { free(this->potential); }
	}

	int ModelPotential::calculatePotentialGrid() {
		const size_t nAtoms = model->getNumberAtoms();
		const double a = model->getA();
		const double b = model->getB();
		const double c = model->getC();
		const double dx = a / this->nx;
		const double dy = b / this->ny;
		const double dz = c / this->nz;
	
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		double *potentialSlice;
		hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double));
		memset(potentialSlice, 0, nx * ny * sizeof(double));
		CUERR

		//////////////////////////////////////////////////////////////////////////////////////////////////////

		hipEvent_t start_total,stop_total;
		float time_total = 0.0f;
		hipEventCreate(&start_total);
		hipEventCreate(&stop_total);
		hipEventRecord(start_total,0);
	
		dim3 threads(BLOCKSIZEX, BLOCKSIZEY, 1);										// ������ ����������
		dim3 grid(this->nx / BLOCKSIZEX / UNROLLX, this->ny / BLOCKSIZEY, 1 );		// ������� ����������� ����� ����� ������� ��� �����������


		AModel::Cortege *pAtoms = model->getTableCell();
		std::sort(pAtoms, pAtoms + nAtoms);
	
		int		atominfoid_host[ATOMS_IN_CONST_MEMORY];
		float	atominfoxy_host[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * ATOMS_IN_CONST_MEMORY];
	
		int j = 0;
		float time_kernel = 0.0f;
		for(size_t kz = 0; kz * dz < c; kz++) {
			for(size_t i = 0; i < nAtoms; i++) {
				if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c && pAtoms[i].element.xsCoordinate.z * c < (kz + 1) * dz ) {
 					atominfoid_host[j] = model->getNumberByName(pAtoms[i].element.Atom) - 1;
 					atominfoxy_host[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * j + 0] = pAtoms[i].element.xsCoordinate.x;
 					atominfoxy_host[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * j + 1] = pAtoms[i].element.xsCoordinate.y;
					j = j + 1;
				}

				if(j == ATOMS_IN_CONST_MEMORY || i == nAtoms - 1) {
 					checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(atominfoid), atominfoid_host, ATOMS_IN_CONST_MEMORY * sizeof(int), 0, hipMemcpyHostToDevice));
 					checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(atominfoxy), atominfoxy_host, ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * ATOMS_IN_CONST_MEMORY * sizeof(float), 0, hipMemcpyHostToDevice));
				
					hipEvent_t start_kernel,stop_kernel;
					float ctime = 0.0f;
					hipEventCreate(&start_kernel);
					hipEventCreate(&stop_kernel);
					hipEventRecord(start_kernel,0);

 					calculatePotentialGridGPU<<<grid, threads>>>(j, a, b, c, dx, dy, potentialSlice, radius);
 					checkCudaErrors( hipDeviceSynchronize() );
					
					hipEventRecord(stop_kernel,0);
					hipEventSynchronize(stop_kernel);
					hipEventElapsedTime(&ctime, start_kernel, stop_kernel);

					time_kernel += ctime;

					std::cout << "slice: " << kz << "calculated atoms: " << j << " current atom: " << i << std::endl;
					
					j = 0;				
				}
			}

			// Device to Host
			memcpy(potential + nx * ny * kz, potentialSlice, nx * ny * sizeof(double));
			memset(potentialSlice, 0, nx * ny * sizeof(double));
		}

		pAtoms = nullptr;
	
		hipEventRecord(stop_total,0);
		hipEventSynchronize(stop_total);
		hipEventElapsedTime(&time_total, start_total, stop_total);
	
		std::cout << std::endl;
		std::cout << "Kernel time calculating potential grid: " << time_kernel	<< "ms." << std::endl;
		std::cout << "Total  time calculating potential grid: " << time_total	<< "ms." << std::endl << std::endl;
	
		return 0;
	}

	__global__ void calculatePotentialGridGPU(int nAtoms, double a, double b, double c, double dx, double dy, double *potential, double r) {
		const int ix = blockDim.x * blockIdx.x * UNROLLX + threadIdx.x;
		const int iy = blockDim.y * blockIdx.y + threadIdx.y;
		const int LINESIZE = UNROLLX * gridDim.x * blockDim.x;

		int l;
	
		double imageval1 = 0.0;
		double imageval2 = 0.0;
		double imageval3 = 0.0;
		double imageval4 = 0.0;
		double imageval5 = 0.0;
		double imageval6 = 0.0;
		double imageval7 = 0.0;
		double imageval8 = 0.0;

		for(l = 0; l < nAtoms; l++) {
			int atomid = atominfoid[l];
			float dY = fabsf(atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 1] * b - (iy * dy));
			float x = atominfoxy[ATOMS_IN_CONST_MEMORY_MULTIPLICATOR * l + 0] * a;

			dY = ( dY >= b / 2.0 ) ? dY - b : dY;
			dY = dY * dY;

		

	// 		float dX1 = ix * dx - x;
	// 		float dX2 = dX1 + gridspacing_u;
	// 		float dX3 = dX2 + gridspacing_u;
	// 		float dX4 = dX3 + gridspacing_u;
	// 		float dX5 = dX4 + gridspacing_u;
	// 		float dX6 = dX5 + gridspacing_u;
	// 		float dX7 = dX6 + gridspacing_u;
	// 		float dX8 = dX7 + gridspacing_u;

			float dX1 = fabsf(x - (ix + 0 * blockDim.x) * dx);
			float dX2 = fabsf(x - (ix + 1 * blockDim.x) * dx);
 			float dX3 = fabsf(x - (ix + 2 * blockDim.x) * dx);
			float dX4 = fabsf(x - (ix + 3 * blockDim.x) * dx);
 			float dX5 = fabsf(x - (ix + 4 * blockDim.x) * dx);
 			float dX6 = fabsf(x - (ix + 5 * blockDim.x) * dx);
 			float dX7 = fabsf(x - (ix + 6 * blockDim.x) * dx);
 			float dX8 = fabsf(x - (ix + 7 * blockDim.x) * dx);

			dX1 = ( dX1 >= a / 2.0 ) ? dX1 - a : dX1;
			dX2 = ( dX2 >= a / 2.0 ) ? dX2 - a : dX2;
			dX3 = ( dX3 >= a / 2.0 ) ? dX3 - a : dX3;
			dX4 = ( dX4 >= a / 2.0 ) ? dX4 - a : dX4;
			dX5 = ( dX5 >= a / 2.0 ) ? dX5 - a : dX5;
			dX6 = ( dX6 >= a / 2.0 ) ? dX6 - a : dX6;
			dX7 = ( dX7 >= a / 2.0 ) ? dX7 - a : dX7;
			dX8 = ( dX8 >= a / 2.0 ) ? dX8 - a : dX8;
		
			float dR1 = sqrtf(dX1 * dX1 + dY);
			float dR2 = sqrtf(dX2 * dX2 + dY);
			float dR3 = sqrtf(dX3 * dX3 + dY);
			float dR4 = sqrtf(dX4 * dX4 + dY);
			float dR5 = sqrtf(dX5 * dX5 + dY);
			float dR6 = sqrtf(dX6 * dX6 + dY);
			float dR7 = sqrtf(dX7 * dX7 + dY);
			float dR8 = sqrtf(dX8 * dX8 + dY);

			if(dR1 < r) {
				dR1 = (dR1 < 1.0e-10) ? 1.0e-10 : dR1;
				imageval1 += calculateProjectedPotential(atomid, dR1);
			}
			if(dR2 < r) {
				dR2 = (dR2 < 1.0e-10) ? 1.0e-10 : dR2;
				imageval2 += calculateProjectedPotential(atomid, dR2);
			}
			if(dR3 < r) {
				dR3 = (dR3 < 1.0e-10) ? 1.0e-10 : dR3;
				imageval3 += calculateProjectedPotential(atomid, dR3);
			}
			if(dR4 < r) {
				dR4 = (dR4 < 1.0e-10) ? 1.0e-10 : dR4;
				imageval4 += calculateProjectedPotential(atomid, dR4);
			}
			if(dR5 < r) {
				dR5 = (dR5 < 1.0e-10) ? 1.0e-10 : dR5;
				imageval5 += calculateProjectedPotential(atomid, dR5);
			}
			if(dR6 < r) {
				dR6 = (dR6 < 1.0e-10) ? 1.0e-10 : dR6;
				imageval6 += calculateProjectedPotential(atomid, dR6);
			}
			if(dR7 < r) {
				dR7 = (dR7 < 1.0e-10) ? 1.0e-10 : dR7;
				imageval7 += calculateProjectedPotential(atomid, dR7);
			}
			if(dR8 < r) {
				dR8 = (dR8 < 1.0e-10) ? 1.0e-10 : dR8;
				imageval8 += calculateProjectedPotential(atomid, dR8);
			}
		}

		potential[ LINESIZE * iy + ix					] = potential[ LINESIZE * iy + ix					] + imageval1;
  		potential[ LINESIZE * iy + ix + 1 * blockDim.x	] = potential[ LINESIZE * iy + ix + 1 * blockDim.x	] + imageval2; 
  		potential[ LINESIZE * iy + ix + 2 * blockDim.x	] = potential[ LINESIZE * iy + ix + 2 * blockDim.x	] + imageval3;
  		potential[ LINESIZE * iy + ix + 3 * blockDim.x	] = potential[ LINESIZE * iy + ix + 3 * blockDim.x	] + imageval4;
  		potential[ LINESIZE * iy + ix + 4 * blockDim.x	] = potential[ LINESIZE * iy + ix + 4 * blockDim.x	] + imageval5;
  		potential[ LINESIZE * iy + ix + 5 * blockDim.x	] = potential[ LINESIZE * iy + ix + 5 * blockDim.x	] + imageval6;
  		potential[ LINESIZE * iy + ix + 6 * blockDim.x	] = potential[ LINESIZE * iy + ix + 6 * blockDim.x	] + imageval7;
  		potential[ LINESIZE * iy + ix + 7 * blockDim.x	] = potential[ LINESIZE * iy + ix + 7 * blockDim.x	] + imageval8;

	}

	__device__ double	calculateProjectedPotential(int numberAtom, double r) {
		double sumf;
		double sums;
 		double dR1;

		sumf = 0.0;
		sums = 0.0;
		dR1 = 6.2831853071796 * r; // 2 * PI * r

 		for(int k = 0; k < 3; k++) {
 			int Offs = (numberAtom) * 12 + k * 2;
 			sumf += FParamsDevice[Offs + 0] * bessk0(dR1 * sqrt(FParamsDevice[Offs + 1]));  
 		}
		sumf *= 300.73079394295; // 4 * PI * PI *a0 * e
	
 		for(int k = 0; k < 3; k++) {
 			int Offs = (numberAtom) * 12 + k * 2;
 			sums += (FParamsDevice[Offs + 6] / FParamsDevice[Offs + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[Offs + 7]);
 		}
		sums *= 150.36539697148; // 2 * PI * PI * a0 * e

		return (sumf + sums);
	}

	__device__ void		swap2(double& a, double& b) {
		double buffer = a;
		a = b;
		b = buffer;
	}

	__device__ double	bessk0( double ax ) {
		double x2;
		double sum;

		if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
			x2 = __ddiv_rn(ax, 2.0);
			x2 = __dmul_rd(x2, x2);
			sum = __fma_rn(k0a[6], x2, k0a[5]);
			sum = __fma_rn(sum, x2, k0a[4]);
			sum = __fma_rn(sum, x2, k0a[3]);
			sum = __fma_rn(sum, x2, k0a[2]);
			sum = __fma_rn(sum, x2, k0a[1]);
			sum = __fma_rn(sum, x2, k0a[0]);



			sum = -log(ax / 2.0) * bessi0( ax ) + sum;


		} else if( ax > 2.0 ) {
			x2 = __ddiv_rn(2.0, ax);
			sum = __fma_rn(k0b[6], x2, k0b[5]);
			sum = __fma_rn(sum, x2, k0b[4]);
			sum = __fma_rn(sum, x2, k0b[3]);
			sum = __fma_rn(sum, x2, k0b[2]);
			sum = __fma_rn(sum, x2, k0b[1]);
			sum = __fma_rn(sum, x2, k0b[0]);

			sum = exp( -ax ) * sum / __dsqrt_rn(ax);
		} else sum = 1.0e20;
		return ( sum );
	}

	__device__ double	bessi0( double ax ) {
		double sum;
		double t;

		if( ax <= 3.75 ) {
			t = __ddiv_rn(ax, 3.75);
			t = __dmul_rd(t, t);
			sum = __fma_rn(i0a[6], t, i0a[5]);
			sum = __fma_rn(sum, t, i0a[4]);
			sum = __fma_rn(sum, t, i0a[3]);
			sum = __fma_rn(sum, t, i0a[2]);
			sum = __fma_rn(sum, t, i0a[1]);
			sum = __fma_rn(sum, t, i0a[0]);
		} else {
			t = __ddiv_rn(3.75, ax);
			sum = __fma_rn(i0b[8], t, i0a[7]);
			sum = __fma_rn(sum, t, i0a[6]);
			sum = __fma_rn(sum, t, i0a[5]);
			sum = __fma_rn(sum, t, i0a[4]);
			sum = __fma_rn(sum, t, i0a[3]);
			sum = __fma_rn(sum, t, i0a[2]);
			sum = __fma_rn(sum, t, i0a[1]);
			sum = __fma_rn(sum, t, i0a[0]);

			sum = exp( ax ) * sum / __dsqrt_rn( ax );
		}
		return( sum );
	}

	int	ModelPotential::savePotential(const char* filename) {
		Image *image = new Image(nx, ny, nz, sizeof(double), 1);
		char filenamept[256];
		strcpy(filenamept, filename);
		strcat(filenamept, "_pt");
		memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
		image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
		delete image;

		return 0;
	}

	AModel::Model* ModelPotential::getModel() {
		return model;
	}
}