#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"

namespace PotentialBuilder {
	ModelPotential::ModelPotential(void) {

	}

	ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double radius, double bindim, const char* fileNameOutput) {
		this->model = model;
		this->nx = nx;
		this->ny = ny;
		this->nz = nz;
		this->radius = radius;
		this->bindim = bindim;

		strcpy(this->fileNameOutput, fileNameOutput);
	}

	ModelPotential::~ModelPotential(void) {
		if(this->model != nullptr) { model = nullptr; }
	}

	int ModelPotential::calculatePotentialGrid() {
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		struct stat statbuf;
		if(stat(fileNameOutput,&statbuf)) {
			wchar_t wzfileNameOutput[256];
			mbstowcs(wzfileNameOutput, fileNameOutput, 256);
			if (CreateDirectory(wzfileNameOutput,NULL))
				std::cout << "Directory [" << fileNameOutput << "] created." << std::endl;
			else {
				std::cout << "Error create [" << fileNameOutput << "] directory." << std::endl << std::endl;
				return -1;
			}
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		const size_t nAtoms = model->getNumberAtoms();
		const double a_h = model->getA();
		const double b_h = model->getB();
		const double c_h = model->getC();
		const double dx = a_h / this->nx;
		const double dy = b_h / this->ny;
		const double dz = c_h / this->nz;

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(radius_d), &radius, sizeof(double)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dx_d), &dx, sizeof(double)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dy_d), &dy, sizeof(double)) );

		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(a_d), &a_h, sizeof(double)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(b_d), &b_h, sizeof(double)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(c_d), &c_h, sizeof(double)) );

		double bindimx = this->bindim; // angstrem
		double bindimy = this->bindim; // angstrem
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(bindimx_d), &bindimx, sizeof(double)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(bindimy_d), &bindimy, sizeof(double)) );

		int	binx = ceil(a_h / bindimx); // dimensionless
		int	biny = ceil(b_h / bindimy); // dimensionless
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(binx_d), &binx, sizeof(int)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(biny_d), &biny, sizeof(int)) );

		//////////////////////////////////////////////////////////////////////////////////////////////////////

		double *potentialSlice;
		checkCudaErrors( hipMallocManaged(&(potentialSlice), nx * ny * sizeof(double)));
		memset(potentialSlice, 0, nx * ny * sizeof(double));
		CUERR

		//////////////////////////////////////////////////////////////////////////////////////////////////////

		dim3 threads(BLOCKSIZEX, BLOCKSIZEY, 1);		
		dim3 grid(this->nx / BLOCKSIZEX, this->ny / BLOCKSIZEY, 1 );

		AModel::Cortege *pAtoms = model->getTableCell();
		std::sort(pAtoms, pAtoms + nAtoms);

		std::vector<atom> slice;
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		float time_kernel = 0.0f;
		float time_total = 0.0f;
		hipEvent_t start_total,stop_total;
		hipEventCreate(&start_total);
		hipEventCreate(&stop_total);
		hipEventRecord(start_total,0);

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		for(size_t kz = 0; kz < nz; kz++) {
			//////////////////////////////////////////////////////////////////////////////////////////////////////
			//////////////	Divide on slices /////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////////////////////

			for(size_t i = 0; i < nAtoms; i++) {
				if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c_h && pAtoms[i].element.xsCoordinate.z * c_h <= (kz + 1) * dz ) {
					atom buff;
					buff.id = i + 1;
					//buff.num = model->getNumberByName(pAtoms[i].element.Atom) - 1;
					buff.num = pAtoms[i].element.Atom - 1;
					buff.x = pAtoms[i].element.xsCoordinate.x;
					buff.y = pAtoms[i].element.xsCoordinate.y;

					slice.push_back(buff);
				}
			}

			//////////////////////////////////////////////////////////////////////////////////////////////////////
			//////////////	atoms			//////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////////////////////
			
			atom *SliceAtoms;
			checkCudaErrors( hipMallocManaged(&(SliceAtoms), slice.size() * sizeof(atom)));
			for(size_t l = 0; l < slice.size(); l++) {
				SliceAtoms[l] = slice[l];
			}

			//memcpy(SliceAtoms, &slice, slice.size() * sizeof(atom));

			hipEvent_t start,stop;
			float ctime = 0.0f;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start,0);
			
			calculatePotentialGridGPU<<<grid, threads>>>(potentialSlice, SliceAtoms, slice.size());
			CUERR

			checkCudaErrors( hipDeviceSynchronize() );

			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&ctime, start, stop);
			time_kernel += ctime;

			std::cout << "slice: " << kz << std::endl << "calculated atoms: " << slice.size() << std::endl;
			
			char slicename[256];
			sprintf(slicename, "%s/slice%003u.slc", fileNameOutput, kz);
			FILE *pFile;
			pFile = fopen(slicename, "wb");
			fwrite(potentialSlice, sizeof(double), nx * ny, pFile);
			fclose(pFile);

			slice.clear();
			hipFree(SliceAtoms);
		}

		hipEventRecord(stop_total,0);
		hipEventSynchronize(stop_total);
		hipEventElapsedTime(&time_total, start_total, stop_total);

		pAtoms = nullptr;

		std::cout << std::endl;
		std::cout << "Kernel time calculating potential grid: " << time_kernel	<< "ms." << std::endl;
		std::cout << "Total  time calculating potential grid: " << time_total	<< "ms." << std::endl << std::endl;


		return 0;
	}

	__global__ void calculatePotentialGridGPU(double *potential, atom* atoms, unsigned int n) {
		const int ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
		const int iy = __umul24(blockDim.y, blockIdx.y) + threadIdx.y;
		const int is = __umul24(blockDim.x, threadIdx.y) + threadIdx.x;
		const int LINESIZE = __umul24(gridDim.x, blockDim.x);

		double latticex = ix * dx_d; // lattice x
		double latticey = iy * dy_d; // lattice y

		__shared__ double imageval[BLOCKSIZEX*BLOCKSIZEY];
		imageval[is] = 0.0;

		for(unsigned int i = 0; i < n; i++) {
			int numberAtom = atoms[i].num;
			double x = fabs(atoms[i].x * a_d - latticex);
			double y = fabs(atoms[i].y * b_d - latticey);

			x = ( x >= a_d / 2.0 ) ? x - a_d : x;
			y = ( y >= b_d / 2.0 ) ? y - b_d : y;

			double r = __dsqrt_rn(x * x + y * y);
			
			//if(r > radius_d) continue;
			r = (r < 1e-20) ? 1e-20 : r;
			double dR1 = 6.2831853071796 * r; // 2 * PI * r;

			imageval[is] += ( 
						FParamsDevice[(numberAtom) * 12 + 0 * 2 + 0] * bessk0(dR1 * __dsqrt_rn(FParamsDevice[(numberAtom) * 12 + 0 * 2 + 1]))
					+	FParamsDevice[(numberAtom) * 12 + 1 * 2 + 0] * bessk0(dR1 * __dsqrt_rn(FParamsDevice[(numberAtom) * 12 + 1 * 2 + 1]))
					+	FParamsDevice[(numberAtom) * 12 + 2 * 2 + 0] * bessk0(dR1 * __dsqrt_rn(FParamsDevice[(numberAtom) * 12 + 2 * 2 + 1])) 
					) * 300.73079394295
					+ (
					(	FParamsDevice[(numberAtom) * 12 + 0 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 0 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 0 * 2 + 7])
					+	(FParamsDevice[(numberAtom) * 12 + 1 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 1 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 1 * 2 + 7])
					+	(FParamsDevice[(numberAtom) * 12 + 2 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 2 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 2 * 2 + 7])
					) * 150.36539697148;
		
		}

		__syncthreads();

		potential[ LINESIZE * iy + ix ] = imageval[is]; 

	}

	__device__ void		swap2(double& a, double& b) {
		double buffer = a;
		a = b;
		b = buffer;
	}

	__device__ double	bessk0( double ax ) {
		double x2;
		double sum;

		if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
			x2 = __ddiv_rn(ax, 2.0);
			x2 = __dmul_rd(x2, x2);
			sum = __fma_rn(k0a[6], x2, k0a[5]);
			sum = __fma_rn(sum, x2, k0a[4]);
			sum = __fma_rn(sum, x2, k0a[3]);
			sum = __fma_rn(sum, x2, k0a[2]);
			sum = __fma_rn(sum, x2, k0a[1]);
			sum = __fma_rn(sum, x2, k0a[0]);



			sum = -log(ax / 2.0) * bessi0( ax ) + sum;


		} else if( ax > 2.0 ) {
			x2 = __ddiv_rn(2.0, ax);
			sum = __fma_rn(k0b[6], x2, k0b[5]);
			sum = __fma_rn(sum, x2, k0b[4]);
			sum = __fma_rn(sum, x2, k0b[3]);
			sum = __fma_rn(sum, x2, k0b[2]);
			sum = __fma_rn(sum, x2, k0b[1]);
			sum = __fma_rn(sum, x2, k0b[0]);

			sum = exp( -ax ) * sum / __dsqrt_rn(ax);
		} else sum = 1.0e20;
		return ( sum );
	}

	__device__ double	bessi0( double ax ) {
		double sum;
		double t;

		if( ax <= 3.75 ) {
			t = __ddiv_rn(ax, 3.75);
			t = __dmul_rd(t, t);
			sum = __fma_rn(i0a[6], t, i0a[5]);
			sum = __fma_rn(sum, t, i0a[4]);
			sum = __fma_rn(sum, t, i0a[3]);
			sum = __fma_rn(sum, t, i0a[2]);
			sum = __fma_rn(sum, t, i0a[1]);
			sum = __fma_rn(sum, t, i0a[0]);
		} else {
			t = __ddiv_rn(3.75, ax);
			sum = __fma_rn(i0b[8], t, i0a[7]);
			sum = __fma_rn(sum, t, i0a[6]);
			sum = __fma_rn(sum, t, i0a[5]);
			sum = __fma_rn(sum, t, i0a[4]);
			sum = __fma_rn(sum, t, i0a[3]);
			sum = __fma_rn(sum, t, i0a[2]);
			sum = __fma_rn(sum, t, i0a[1]);
			sum = __fma_rn(sum, t, i0a[0]);

			sum = exp( ax ) * sum / __dsqrt_rn( ax );
		}
		return( sum );
	}

	int	ModelPotential::savePotential(const char* filename) {
		Image *image = new Image(nx, ny, nz, sizeof(double), 1);
		char filenamept[256];
		strcpy(filenamept, filename);
		strcat(filenamept, "_pt");
		//memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
		image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
		delete image;

		return 0;
	}

	int	ModelPotential::savePotentialStack(const char* filename, const char* stackDirectory) {
		Image *image = new Image(nx, ny, nz, sizeof(double), 1);
		char filenamept[256];
		strcpy(filenamept, filename);
		strcat(filenamept, "_pt");
		image->saveStackMRC(filenamept, stackDirectory, model, nx, ny, nz, mrc_FLOAT);
		delete image;

		return 0;
	}


	AModel::Model* ModelPotential::getModel() {
		return model;
	}
}