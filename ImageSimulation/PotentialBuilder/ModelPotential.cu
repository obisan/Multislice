#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "ModelPotential.h"

namespace PotentialBuilder {
	ModelPotential::ModelPotential(void) {

	}

	ModelPotential::ModelPotential(AModel::Model *model, size_t nx, size_t ny, size_t nz, double radius, double bindim, const char* fileNameOutput) {
		this->model = model;
		this->nx = nx;
		this->ny = ny;
		this->nz = nz;
		this->radius = radius;
		this->bindim = bindim;

		strcpy(this->fileNameOutput, fileNameOutput);
	}

	ModelPotential::~ModelPotential(void) {
		if(this->model != nullptr) { model = nullptr; }
	}

	int ModelPotential::calculatePotentialGrid() {
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		struct stat statbuf;
		if(stat(fileNameOutput,&statbuf)) {
			wchar_t wzfileNameOutput[256];
			mbstowcs(wzfileNameOutput, fileNameOutput, 256);
			if (CreateDirectory(wzfileNameOutput,NULL))
				std::cout << "Directory [" << fileNameOutput << "] created." << std::endl;
			else {
				std::cout << "Error create [" << fileNameOutput << "] directory." << std::endl << std::endl;
				return -1;
			}
		}

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		const size_t nAtoms = model->getNumberAtoms();
		const double a_h = model->getA();
		const double b_h = model->getB();
		const double c_h = model->getC();
		const double dx = a_h / this->nx;
		const double dy = b_h / this->ny;
		const double dz = c_h / this->nz;

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		
		double *potentialSlice;
		potentialSlice = (double*) malloc(nx * ny * sizeof(double));
		memset(potentialSlice, 0, nx * ny * sizeof(double));
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		AModel::Cortege *pAtoms = model->getTableCell();
		std::sort(pAtoms, pAtoms + nAtoms);

		std::vector<atom> slice;
		
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		float time_kernel = 0.0f;
		float time_total = 0.0f;
		hipEvent_t start_total,stop_total;
		hipEventCreate(&start_total);
		hipEventCreate(&stop_total);
		hipEventRecord(start_total,0);

		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////

		for(size_t kz = 0; kz < nz; kz++) {
			//////////////////////////////////////////////////////////////////////////////////////////////////////
			//////////////	Divide on slices /////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////////////////////

			for(size_t i = 0; i < nAtoms; i++) {
				if( kz * dz <= pAtoms[i].element.xsCoordinate.z * c_h && pAtoms[i].element.xsCoordinate.z * c_h <= (kz + 1) * dz ) {
					atom buff;
					buff.id = i + 1;
					//buff.num = model->getNumberByName(pAtoms[i].element.Atom) - 1;
					buff.num = pAtoms[i].element.Atom - 1;
					buff.x = pAtoms[i].element.xsCoordinate.x;
					buff.y = pAtoms[i].element.xsCoordinate.y;

					slice.push_back(buff);
				}
			}

			//////////////////////////////////////////////////////////////////////////////////////////////////////
			//////////////	atoms			//////////////////////////////////////////////////////////////////////
			//////////////////////////////////////////////////////////////////////////////////////////////////////
			
			std::vector<atom> &atoms = slice;

			hipEvent_t start,stop;
			float ctime = 0.0f;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start,0);
			
			for(size_t iy = 0; iy < ny; iy++) {
				for(size_t ix = 0; ix < nx; ix++) {

					double latticex = ix * dx; // lattice x
					double latticey = iy * dy; // lattice y

					double imageval = 0.0;

					for(unsigned int i = 0; i < slice.size(); i++) {
						int numberAtom = atoms[i].num;
						double x = fabs(atoms[i].x * a_h - latticex);
						double y = fabs(atoms[i].y * b_h - latticey);

						x = ( x >= a_h / 2.0 ) ? x - a_h : x;
						y = ( y >= b_h / 2.0 ) ? y - b_h : y;

						double r = sqrt(x * x + y * y);
			
						if(r > radius) continue;
						r = (r < 1e-20) ? 1e-20 : r;
						double dR1 = 6.2831853071796 * r; // 2 * PI * r;

						imageval += ( 
									FParamsDevice[(numberAtom) * 12 + 0 * 2 + 0] * bessk0(dR1 * sqrt(FParamsDevice[(numberAtom) * 12 + 0 * 2 + 1]))
								+	FParamsDevice[(numberAtom) * 12 + 1 * 2 + 0] * bessk0(dR1 * sqrt(FParamsDevice[(numberAtom) * 12 + 1 * 2 + 1]))
								+	FParamsDevice[(numberAtom) * 12 + 2 * 2 + 0] * bessk0(dR1 * sqrt(FParamsDevice[(numberAtom) * 12 + 2 * 2 + 1])) 
								) * 300.73079394295
								+ (
								(	FParamsDevice[(numberAtom) * 12 + 0 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 0 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 0 * 2 + 7])
								+	(FParamsDevice[(numberAtom) * 12 + 1 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 1 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 1 * 2 + 7])
								+	(FParamsDevice[(numberAtom) * 12 + 2 * 2 + 6] / FParamsDevice[(numberAtom) * 12 + 2 * 2 + 7]) * exp(-(6.2831853071796 * r * r) / FParamsDevice[(numberAtom) * 12 + 2 * 2 + 7])
								) * 150.36539697148;
		
					}
					potentialSlice[ nx * ny * iy + ix ] = imageval; 
				}				
			}

			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&ctime, start, stop);
			time_kernel += ctime;

			std::cout << "slice: " << kz << std::endl << "calculated atoms: " << slice.size() << std::endl;
			
			char slicename[256];
			sprintf(slicename, "%s/slice%003u.slc", fileNameOutput, kz);
			FILE *pFile;
			pFile = fopen(slicename, "wb");
			fwrite(potentialSlice, sizeof(double), nx * ny, pFile);
			fclose(pFile);

			slice.clear();
			free(potentialSlice);
		}

		hipEventRecord(stop_total,0);
		hipEventSynchronize(stop_total);
		hipEventElapsedTime(&time_total, start_total, stop_total);

		pAtoms = nullptr;

		std::cout << std::endl;
		std::cout << "Kernel time calculating potential grid: " << time_kernel	<< "ms." << std::endl;
		std::cout << "Total  time calculating potential grid: " << time_total	<< "ms." << std::endl << std::endl;


		return 0;
	}

	/*-------------------- bessk0() ---------------*/
	/*
		modified Bessel function K0(x)
		see Abramowitz and Stegun page 380
    
		Note: K0(0) is not define and this function
			returns 1E20
 
		x = (double) real arguments
    
		this routine calls bessi0() = Bessel function I0(x)
    
		12-feb-1997 E. Kirkland
	 */
	double bessk0( double x )
	{
		double bessi0(double);
	
		int i;
		double ax, x2, sum;
		double k0a[] = { -0.57721566, 0.42278420, 0.23069756,
			0.03488590, 0.00262698, 0.00010750, 0.00000740};
	
		double k0b[] = { 1.25331414, -0.07832358, 0.02189568,
			-0.01062446, 0.00587872, -0.00251540, 0.00053208};
	
		ax = fabs( x );
		if( (ax > 0.0)  && ( ax <=  2.0 ) ) {
			x2 = ax / 2.0;
			x2 = x2 * x2;
			sum = k0a[6];
			for( i=5; i>=0; i--) sum = sum*x2 + k0a[i];
			sum = -log(ax/2.0) * bessi0(x) + sum;
		} else if( ax > 2.0 ) {
			x2 = 2.0/ax;
			sum = k0b[6];
			for( i=5; i>=0; i--) sum = sum*x2 + k0b[i];
			sum = exp( -ax ) * sum / sqrt( ax );
		} else sum = 1.0e20;
		return ( sum );
	
	}  /* end bessk0() */


	/*-------------------- bessi0() ---------------*/
	/*
		modified Bessel function I0(x)
		see Abramowitz and Stegun page 379

		x = (double) real arguments

		12-feb-1997 E. Kirkland
	 */
	 double bessi0( double x )	 {
 		int i;
 		double ax, sum, t;
 	
 		double i0a[] = { 1.0, 3.5156229, 3.0899424, 1.2067492,
			0.2659732, 0.0360768, 0.0045813 };

 		double i0b[] = { 0.39894228, 0.01328592, 0.00225319,
 			-0.00157565, 0.00916281, -0.02057706, 0.02635537,
 			-0.01647633, 0.00392377};

		ax = fabs( x );
		if( ax <= 3.75 ) {
			t = x / 3.75;
			t = t * t;
			sum = i0a[6];
			for( i=5; i>=0; i--) sum = sum*t + i0a[i]; 
		} else {
			t = 3.75 / ax;
			sum = i0b[8];
			for( i=7; i>=0; i--) sum = sum*t + i0b[i];
			sum = exp( ax ) * sum / sqrt( ax );
		}
		return( sum );

	}  /* end bessi0() */

	int	ModelPotential::savePotential(const char* filename) {
		Image *image = new Image(nx, ny, nz, sizeof(double), 1);
		char filenamept[256];
		strcpy(filenamept, filename);
		strcat(filenamept, "_pt");
		//memcpy(image->imageData, this->potential, nx * ny * nz * sizeof(double));
		image->saveMRC(filenamept, model, nx, ny, nz, mrc_FLOAT);
		delete image;

		return 0;
	}

	int	ModelPotential::savePotentialStack(const char* filename, const char* stackDirectory) {
		Image *image = new Image(nx, ny, nz, sizeof(double), 1);
		char filenamept[256];
		strcpy(filenamept, filename);
		strcat(filenamept, "_pt");
		image->saveStackMRC(filenamept, stackDirectory, model, nx, ny, nz, mrc_FLOAT);
		delete image;

		return 0;
	}


	AModel::Model* ModelPotential::getModel() {
		return model;
	}
}