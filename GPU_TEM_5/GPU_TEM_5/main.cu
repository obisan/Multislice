#include "hip/hip_runtime.h"
﻿///********************************************************************
//*  GPU_TEM
//*  (c) pust@lemoi.phys.dvgu.ru
//*********************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <windows.h> 
#include <winuser.h>
#include <csetjmp>
#include <cstdio>

#include <direct.h>


#include <hip/hip_complex.h>
//#include <cutil_inline.h>
#include <GL/glew.h>
#include <GL/freeglut.h>

//#include <cutil_gl_inline.h>
#include <cuda_gl_interop.h>
// Utilities and timing functions
//#include <hiprand.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include "VersionNo.h"



#include "../../include/mrc.h"
#include "../../include/functions.h" 
#include "tem_gl.h"
#include "resource.h"

  static jmp_buf env;

bool FileNameShow=true;
HANDLE hStdout, hStdin; 

//TCHAR  filename[2048];
char path[2048];
char fname[2048];
char APP_path[2048];

char * pch;
HICON hIcon;

HINSTANCE hinst;            // handle to current instance 
HWND hwnd;                  // main window hand

//#define ClientWidth 800
//#define ClientHeight 600
//#include <test_gl_kernel.cu>

//#define REFRESH_DELAY 100
int REFRESH_DELAY=100;

char filename[4096]="d:\\VisualProfiler\\fe75b25.wav";

GLuint pbo = 0;     // OpenGL pixel buffer object
GLuint texid = 0;   // texture
int vx=0,vy=0;
//unsigned int timer = 0;


int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
unsigned int frameCount = 0;
unsigned int *d_result;

float *SeriaImage=0;
//unsigned char *SeriaImage=0;

double Plank_h = 6.6256E-34;
float c_light = 2.9979E+08f;
float eV=1.602E-19f;
float voltage=300.0f;
float Cc=1.0f;
int ox, oy;
int buttonState = 0;
float OldZoom=1.0f;
float tx_old=0.0f,ty_old=0.0f;
bool changes=false;
bool NewFile2Open=false;
bool SeriaOn=false;
int SeriaCounter=0;
int SeriaEnd=0;
char fps[256];
float aspect=1.0f;
float DfStep=10.0f,DfStepSmall=1.0f;
string ver;
float dFOld=0.0;
float DefocusStart=-50.0f,DefocusStep=1.0f,DefocusSeriaCs=-5000.0;
char ErrorMessage[180];
bool ErrorOn=false;


void  OpenParamFile(char * filenamepath);


void initCudaBuffers(hipfftComplex *h_img,int dimx,int dimy,int nproj);
	void* bitmap_fonts[7] = {
      GLUT_BITMAP_9_BY_15,
      GLUT_BITMAP_8_BY_13,
      GLUT_BITMAP_TIMES_ROMAN_10,
      GLUT_BITMAP_TIMES_ROMAN_24,
      GLUT_BITMAP_HELVETICA_10,
      GLUT_BITMAP_HELVETICA_12,
      GLUT_BITMAP_HELVETICA_18     
   };

   char* bitmap_font_names[7] = {
      "GLUT_BITMAP_9_BY_15",
      "GLUT_BITMAP_8_BY_13",
      "GLUT_BITMAP_TIMES_ROMAN_10",
      "GLUT_BITMAP_TIMES_ROMAN_24",
      "GLUT_BITMAP_HELVETICA_10",
      "GLUT_BITMAP_HELVETICA_12",
      "GLUT_BITMAP_HELVETICA_18"     
   };
  

void
print_bitmap_string(void* font, char* s)
{
   if (s && strlen(s)) {
      while (*s) {
         glutBitmapCharacter(font, *s);
         s++;
      }
   }
}
/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;
	hipError_t error;
	int devID = 0;
	hipDeviceProp_t deviceProp;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no GPU device. Press any key.\n");
		_getch();
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no GPU device supporting CUDA. Press any key.\n");
		_getch();
		return false;
	}
	
	hipSetDevice(devID);

	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
//	int	  devID;
	//devID = cutGetMaxGflopsDeviceId();
	//hipSetDevice(1);
	//hipSetDevice(0);
	//hipGetDevice(&devID);
	//printf("CUDA initialized.%d\n",devID);
	//_getch();
	return true;
}

#endif
HWND GetConsoleHwnd(void)
   {
       #define MY_BUFSIZE 1024 // Buffer size for console window titles.
       HWND hwndFound;         // This is what is returned to the caller.
       char pszNewWindowTitle[MY_BUFSIZE]; // Contains fabricated
                                           // WindowTitle.
       char pszOldWindowTitle[MY_BUFSIZE]; // Contains original
                                           // WindowTitle.

       // Fetch current window title.

       GetConsoleTitle(pszOldWindowTitle, MY_BUFSIZE);

       // Format a "unique" NewWindowTitle.

       wsprintf(pszNewWindowTitle,"%d/%d",
                   GetTickCount(),
                   GetCurrentProcessId());

       // Change current window title.

       SetConsoleTitle(pszNewWindowTitle);

       // Ensure window title has been updated.

       Sleep(40);

       // Look for NewWindowTitle.

       hwndFound=FindWindow(NULL, pszNewWindowTitle);

       // Restore original window title.

       SetConsoleTitle(pszOldWindowTitle);

       return(hwndFound);
   }
	HICON SetIcon(HWND hWnd, HICON hIcon, bool bBigIcon=true)
{
return (HICON)
SendMessage(hWnd, 
(UINT)WM_SETICON,
(WPARAM) bBigIcon ? ICON_BIG : ICON_SMALL,
(LPARAM) hIcon);
}
bool ShowOpenDialogMy(HWND HWndOwner, char *Title, 
    char *InitialDir, char *Filter,int iFlag,char *FileName) 
{ 
    FileName[0] = '\0';

    OPENFILENAME ofn;
    ZeroMemory(&ofn, sizeof(OPENFILENAME));

    //  OPENFILENAME 
    ofn.lStructSize = sizeof(OPENFILENAME); 
    ofn.hwndOwner = HWndOwner; 
    ofn.lpstrFile = FileName; 
    ofn.nMaxFile = MAX_PATH; 
    ofn.lpstrFilter = Filter; 
    ofn.nFilterIndex = 0; 
    ofn.lpstrTitle = Title; 
    ofn.lpstrInitialDir = InitialDir; 
    ofn.Flags = OFN_EXPLORER | OFN_FILEMUSTEXIST;  
	return GetOpenFileName(&ofn);
    //if ( iFlag == FILE_LOAD )return GetOpenFileName(&ofn);
    //else if ( iFlag == FILE_SAVE )return GetSaveFileName(&ofn); 
}
void LoadNewImage(){
	//hipfftComplex * image_c=0;
	//TCHAR  filename[2048];
	HWND test=GetConsoleHwnd();
	ShowOpenDialogMy(test, "Open wave complex function", "", "wav\0*.wav\0All\0*.*\0",0,filename);
	NewFile2Open=true;
	//glutLeaveMainLoop();
	 //longjmp(env, 1);

	// exit(0);
	//	char * pch;
 //   pch=strrchr(filename,'\\');

	//printf("test %s %d\n",path,pch-path+1);


	//strncpy(path,filename,pch-filename+1);
	//	   path[pch-filename+2]='\0';

	//float * image=0;
	//hipfftComplex *image_c=0;
	//int dimx,dimy,nproj;

	////image=ReadMRC2Float(filename,dimx,dimy,nproj);
	///*image_c=ReadMRC2Complex(filename,dimx,dimy,nproj);
	//if (image_c == NULL) {
	//		fprintf(stderr,"End of story\n");
	//			return ;
	//		}
	//int size=dimx*dimy*nproj*sizeof(hipfftComplex);
	//checkCudaErrors( hipMemcpy( d_img, image_c, size, hipMemcpyHostToDevice));
	//for (int i=0;i<nproj;i++)
	//	CreateFFT(d_img+i*dimx*dimy,dimx,dimy);

	//free(image_c);*/

	//hipfftDoubleComplex *image_d_c=0;
	////image=ReadMRC2Float(filename,dimx,dimy,nproj);
	////image_c=ReadMRC2Complex(filename,dimx,dimy,nproj);
	////HT=300;
	////a=70,b=70;
	//nproj=1;
	//image_d_c=ReadWavAndParams(filename,a,b,c,alpha,beta,gamma,HT,dimx,dimy,thickness);
	//if (image_d_c == NULL) {
	//		fprintf(stderr,"End of story\n");
	//			return ;
	//		}
	//float Eng=HT*eV*1000.0f;
	//float E0=511000*eV;
	//Lambda = ( Plank_h * c_light / sqrt( 2 * Eng * E0 + Eng * Eng ) ) * 1E9;
	//ux=1.0f/a*10.0f;
	//uy=1.0f/b*10.0f;
	////Df=-41.0f;
	//H_Cc=Cc*(3.5e-06)*(1-Eng/E0)/(1+Eng/(2*E0));
	//Df=-1.2f*sqrt(Cs*Lambda);
	//tetta=conv/(Lambda*sqrt(log(2.0f))) ;

	//image_c = (hipfftComplex*)malloc(dimx*dimy*sizeof(hipfftComplex));
	//memset(image_c,0,dimx*dimy*sizeof(hipfftComplex));
	//
	//double pix_per_nm;
	//pix_per_nm=1024.0/max(a,b);
	//width=(int)(a*pix_per_nm);
	//height=(int)(b*pix_per_nm);
	//for(int i=0;i<dimx*dimy;i++)
	//	{
	//	image_c[i].x=(float)image_d_c[i].x;
	//	image_c[i].y=(float)image_d_c[i].y;
	//	}
	//int size=dimx*dimy*nproj*sizeof(hipfftComplex);
	//
	////checkCudaErrors( hipMemcpy( d_img, image_c, size, hipMemcpyHostToDevice));
	//checkCudaErrors(hipFree(d_img));
	//checkCudaErrors(hipFree(d_tmp_c));
	//checkCudaErrors(hipFree(d_result_tex));
	////checkCudaErrors(hipFree(cu_array));
 //   initCudaBuffers(image_c,dimx,dimy,nproj);
	//hipfftPlan2d(&plan, dimx,dimy, HIPFFT_C2C);
	//for (int i=0;i<nproj;i++)
	//	CreateFFT(d_img+i*dimx*dimy,dimx,dimy);

	//free(image_d_c);
	//free(image_c);



	}
void CreateConsole(){
	system("cls");
	int Mag=(int)(Zoom*(((float)1024/(float)96)*0.0254f)/(a*1e-09f));
	COORD CursorPosition;
	CursorPosition.X=0;
	CursorPosition.Y=0;
	SetConsoleCursorPosition(hStdout,CursorPosition);
	int devID=0;
	hipGetDevice(&devID);
	for(int i=0;i<25;i++)
	 printf("                                                                              \n");
	SetConsoleCursorPosition(hStdout,CursorPosition);
	//printf("File loaded: %s \n",filename);
	printf("Commands:                                                                     \n");
	printf("d - image or difraction; c - CTF or image; r - real or imaginary part of CTF   \n");
	printf("s - save data; i - save image; 8/2 +- 10 nm defocus; 4/6 +- 1 nm defocus       \n");
	printf("7/1 - increase decrease contrast for difraction; 9/3 - inc/dec apperture radius \n");
	//printf("9/3 - increase decrease apperture radius                                       \n");
	printf("5 - restore defaults; up(+shift) or down(+shift) - change Cs                  \n");
	printf("a - insert/remove apperture; left or right + Ctrl change image size            \n");
	printf("Grey + or - magnification. Up(+ctrl) or down(+ctrl) - change Exp.time         \n");
	printf("Mouse left/middle/right - shift/magnification/menu                            \n\n");
	printf("Esc - exit                                                                    \n\n");
	printf("Current parameters:                                                            \n");
	printf("Image size %.1f nm; DF= %.0f nm Cs=%.1e dZ=%.0f                              \n",ImagesizeInnm,Df,Cs,dZ);
	printf("Mode: %s                                                \n",ImageOrCTF==1?"image":RealCTFOrComplexCTF==0?"Phase CTF":"Real CTF");
	printf("Apperture %s , radius %.3f 1/nm (d= %.2f nm), Ax=%.1f, Ay=%.1f            \n",Apperture==1?"in":"out",AppertureRadius,1.0f/AppertureRadius,Ax,Ay);
	printf("Contrast for difraction or CTF %.3f               \n",Contrast);
	//printf("img_min %.3e, img_max %.3e\n",img_min,img_max);
	printf("Mag %.3f %s                                      \n",(Mag>1.0e6)?(Mag/1.0e6):(Mag>1.0e3)?(Mag/1.0e3):(int)(Mag),(Mag>1.0e6)?"Mx":(Mag>1.0e3)?"kx":"x");
	printf("Zoom %.3e %.3e Exposition %.2fs                                \n\n",Zoom,ZoomImage,(float)REFRESH_DELAY/1000.0f);
	//char * pch;
    //pch=strrchr(path,'\\');
 	//printf("(c) pust@lemoi.phys.dvgu.ru gpuid=%d, dimx %d dimy %d version %s  %s %s      ",devID,dimx,dimy,STRPRODUCTVER,DATESTAMP,TIMESTAMP);
 	printf("(c) pust@lemoi.phys.dvgu.ru, x %d y %d ver. %s %s %s",dimx,dimy,STRPRODUCTVER,DATESTAMP,TIMESTAMP);
		if (FileNameShow) {
		printf("\nfile name=%s",filename);
		}
		
 	//printf("(c) pust@lemoi.phys.dvgu.ru %s",ver);
	 // adasd
	//printf("test %.3e\n",a*1e-09f);
	//printf("test %d\n",(int)(((float)1024/(float)96)*0.0254f*Zoom/(a*1e-09f))/(1000000));
	
	}
void init(){
  glClearColor(1.0, 1.0, 1.0, 0.0);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluOrtho2D(0.0, 800.0, 0.0, 800.0);
};


int xx=400,yy=300;
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
        buttonState |= 1<<button;
    else if (state == GLUT_UP)
        buttonState = 0;

    ox = x; oy = y;


    glutPostRedisplay();
}


//void mouse(int button, int state, int x, int y){
//    xx=x; yy=y;
//    if((button==0)&(state==0))
//        glClearColor((float)(rand()%100)/100.0, (float)(rand()%100)/100.0, (float)(rand()%100)/100.0, 0.0);
//        
//	glutPostRedisplay();
//}   

void motion(int x, int y)
{
if (ImageOrCTF==1){
    float dx, dy;
    dx = (float)x - ox;
    dy = (float)y - oy;

    if (buttonState & 1) {
        // left = translate
        tx -= dx/Zoom;
        ty -= dy/Zoom;
    }
    else if (buttonState & 2) {
        // middle = zoom
		if (ImageOrDifr==1) {
		OldZoom=Zoom;
        Zoom -= (dy / 1000.0f);
		if (Zoom<1.0e-4f)
			Zoom=OldZoom;
		ZoomImage=Zoom;

		changes=true;
			}

    }

    ox = x; oy = y;
	//changes=true;

    glutPostRedisplay();
	}
}
//
void computeFPS()
{
    frameCount++;
    fpsCount++;
    //if (fpsCount == fpsLimit-1) {
    //  //  g_Verify = true;
    //}
    if (fpsCount == fpsLimit) {
        float ifps = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
		sprintf(fps, "%s (HT=%.f kV, w=%.1f %s(%d px), h=%.1f %s(%d px), Cs=%.3fmm, Df=%.1f nm, dZ=%.0f nm %s): %3.1f fps v.%s exp: %.1fs noise %d","GPU TEM simulation",
			HT,ImageOrCTF==1?a/Zoom:Zoom/a,ImageOrCTF==1?"nm":"1/nm",width,ImageOrCTF==1?b/Zoom:Zoom/b,ImageOrCTF==1?"nm":"1/nm",height,Cs/1000000,Df,dZ,ImageOrCTF==1?"image":RealCTFOrComplexCTF==0?"Im CTF":"Re CTF",ifps,STRPRODUCTVER,(float)REFRESH_DELAY/1000.0f,NoiseLevel);  

        glutSetWindowTitle(fps);
		//SetIcon( GetConsoleHwnd(), hIcon);

        fpsCount = 0; 
        sdkResetTimer(&timer);  

      //  AutoQATest();
    }
}
// display results using OpenGL
void display()
{
    sdkStartTimer(&timer);  

    // execute filter, writing results to pbo
    checkCudaErrors(cudaGLMapBufferObject((void**)&d_result, pbo));
    //CreateTexture
	DrawImage(d_img, d_result, dimx, dimy, slice);

    checkCudaErrors(cudaGLUnmapBufferObject(pbo));

    // load texture from pbo
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBindTexture(GL_TEXTURE_2D, texid);
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, dimx, dimy, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    //glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    // display results
    glClear(GL_COLOR_BUFFER_BIT);

    glEnable(GL_TEXTURE_2D);
    glDisable(GL_DEPTH_TEST);

    glBegin(GL_QUADS);
    glTexCoord2f(0, 1); glVertex2f(0, 0);
    glTexCoord2f(1, 1); glVertex2f(1, 0);
    glTexCoord2f(1, 0); glVertex2f(1, 1);
    glTexCoord2f(0, 0); glVertex2f(0, 1);
    glEnd();

    glDisable(GL_TEXTURE_2D);

    //if (g_CheckRender && g_CheckRender->IsQAReadback() && g_Verify) {
    //    // readback for QA testing
    //    printf("> (Frame %d) Readback BackBuffer\n", frameCount);
    //    g_CheckRender->readback( width, height );
    //    g_CheckRender->savePPM(sOriginal[g_Index], true, NULL);
    //    if (!g_CheckRender->PPMvsPPM(sOriginal[g_Index], sReference[g_Index], MAX_EPSILON_ERROR, THRESHOLD)) {
    //        g_TotalErrors++;
    //    }
    //    g_Verify = false;
    //}
	if (FileNameShow) {
		glColor3f(0.0,1.0,0.0);
		glRasterPos2f(  0.01f, (float)(height-15)/height);
		//glRasterPos2f(  0.0, (float)dimy/2);
		print_bitmap_string(bitmap_fonts[6],filename_short);
   //print_bitmap_string(bitmap_fonts[6],filename);
		glColor3f(1.0,1.0,1.0);
		}
	if (SeriaOn) {
		glColor3f(1.0,0.0,0.0);
		glRasterPos2f(  0.01f, (float)(height-40)/height);
		//glRasterPos2f(  0.0, (float)dimy/2);
		print_bitmap_string(bitmap_fonts[6],"Wait for focal series!");
   //print_bitmap_string(bitmap_fonts[6],filename);
		glColor3f(1.0,1.0,1.0);
	   	}
	if (ErrorOn) {
		glColor3f(0.5,0.0,0.0);
		glRasterPos2f(  0.01f,  (float)(height-40)/height);
		//glRasterPos2f(  0.0, (float)dimy/2);
		print_bitmap_string(bitmap_fonts[6],ErrorMessage);
		//print_bitmap_string(bitmap_fonts[6],filename);
		glColor3f(1.0,1.0,1.0);
	}
    glutSwapBuffers();
	if (changes) {
		CreateConsole();
		changes=false;
		}
    sdkStopTimer(&timer);  
	seed=(int)(sdkGetAverageTimerValue(&timer)*1000.0f);

    computeFPS();

}
void idle()
{
    glutPostRedisplay();
}
void keyboard(unsigned char key, int x, int y)
{
    switch(key) {
        case 27:
            exit(0);
			//glutLeaveMainLoop ();
            break;
		case 'v':
			if (!CTFoff){
			 CTFoff=true;
			} else {
				CTFoff=false;

			}
			break;
		case '0':
			glutPositionWindow(0 , 0);
			break;
        case '-':
        //    slice-=1;
          //  if (slice < 0) slice = nproj-1;
			//RecalcuateMinMax=1;
			if (ImageOrDifr==1) {
				Zoom=ZoomImage;
				OldZoom=Zoom;
				Zoom/=1.3f;
				if (Zoom<1.0e-4f)
					Zoom=OldZoom;	
				ZoomImage=Zoom;} else { Zoom=1.0f; }

			//glPixelZoom(Zoom,Zoom);

            break;
            
        case '+':
			if (ImageOrDifr==1) {Zoom=ZoomImage; Zoom*=1.3f;ZoomImage=Zoom;} else { Zoom=1.0f; }
			//glPixelZoom(Zoom,Zoom);
            //slice+=1;
			//RecalcuateMinMax=1;
            //if (slice >= nproj) slice = 0;
			break;
			
		case '6':
			if (ImageOrDifr==1) {Df+=DfStepSmall;} else {Ax+=1.0f;}
			break;

		case '4':
			if (ImageOrDifr==1) {Df-=DfStepSmall;} else {Ax-=1.0f;}
			break; 
		case '8':
			if (ImageOrDifr==1) {Df+=DfStep;} else {Ay-=1.0f;}
			break;

		case '2':
			if (ImageOrDifr==1) {Df-=DfStep;} else {Ay+=1.0f;}
			break;
		case '5':
			if	(Cs>0) {
			Df=-1.2f*sqrt(Cs*Lambda);
				} else {
				 Df=1.2f*sqrt(abs(Cs)*Lambda);
				}
			Contrast=1.0f;
			Zoom=1.0f;
			tx=0;
			ty=0;
			Ax=0.0f;
			Ay=0.0f;
			dZ=0.0;
			//glPixelZoom(Zoom,Zoom);
			glRotated(0.0f,0.0,0.0,0.0);
			//Cs=0.6e+6;
			break;
		case 'c':
			if (ImageOrCTF==1) {
			ImageOrCTF=0;
			OldZoom=Zoom;
			Zoom=2.0f;
			tx_old=tx;
			ty_old=ty;
			tx=0.0f;
			ty=0.0f;
			glutReshapeWindow(dimx,dimy);
				} else  {
			ImageOrCTF=1;
			Zoom=OldZoom;
			tx=tx_old;
			ty=ty_old;
			glutReshapeWindow(width,height);

				}
			//RecalcuateMinMax=1;
			break;
		case 'r':
			if (RealCTFOrComplexCTF==1) {
			RealCTFOrComplexCTF=0;
				} else  {
			RealCTFOrComplexCTF=1;
				}
			//RecalcuateMinMax=1;
			break;
		case 'd':
			if (ImageOrDifr==1) {
			ImageOrDifr=0;
			ZoomImage=Zoom;

			Zoom=1.0f;
	  				} else  {
			ImageOrDifr=1;
			Zoom=ZoomImage;
				}
			//RecalcuateMinMax=1;
			break;
		case '7':
			Contrast*=1.3f;
			break;

		case '1':
			Contrast*=0.7f;
			break;
		case '9':
			AppertureRadius*=1.3f;
			break;

		case '3':
			AppertureRadius*=0.7f;
			break;
		case 'a':
			if (Apperture==1) {
			Apperture=0;
				} else  {
			Apperture=1;
				}
			//RecalcuateMinMax=1;
			break;
		case 'x':
			//glutReshapeWindow(width,995);
			glRotated(10.0f,1.0,1.0,0.0);

			break;
		case 'l':
			LoadNewImage();
			glutLeaveMainLoop();
			break;
		case 's':
			pch=strrchr(filename,'\\');

			//printf("test %s %d\n",path,pch-path+1);


			strncpy(path,filename,pch-filename+1);
			path[pch-filename+2]='\0';
		SaveImage(strcat (path,"0_d_tmp_c_slice.mrc"),d_tmp_c,dimx,dimy);
		//SaveImage("d:\\image\\mrc\\0_d_tmp_c_slice.mrc",d_tmp_c,dimx,dimy);
		//SaveImage("d:\\image\\mrc\\0_d_img_slice.mrc",d_img+slice*dimx*dimy,dimx,dimy);
		printf("Original image saved slice %d\n",slice);
			break;
		case 'i':
					
			pch=strrchr(filename,'\\');

			//printf("test %s %d\n",path,pch-path+1);


		strncpy(path,filename,pch-filename+1);
		path[pch-filename+2]='\0';
		//strncpy(fname,filename+pch+1,filename-pch);
		//fname[filename-pch+2]='\0';
		substr(fname,filename,(int)(pch - filename + 1),(int)strlen(filename)-(int)(pch - filename));
		//SaveImage1("d:\\image\\mrc\\0_buffer_slice.mrc",d_result,dimx,dimy);
		//SaveImage1(strcat (path,"0_buffer_slice.mrc"),d_result,dimx,dimy);
		SaveImage1(strcat (path,fname),d_result,dimx,dimy);
		printf("Buffer image saved\n");
			break;
		case 'f':
			FileNameShow=!FileNameShow;
			break;
		case 'p':
			OpenParamFile(filename);
			//printf("defocus seria cs= %.3e\n",DefocusSeriaCs);
			//_getch();
			break;
		case 'n':

			if (NoiseOn==1) {
			NoiseOn=0;
				} else  {
			NoiseOn=1;
				}
			break;
		case 'w':
		   //SeriaEnd=4;
		   //	PROCESS_MEMORY_COUNTERS info;
			//GetProcessMemoryInfo( GetCurrentProcess( ), &info, sizeof(info) );
			//MessageBox(NULL,(size_t)info.PeakWorkingSetSize, "Memory alert", MB_OK);
		  //long seriaSize,memSize;
		   //seriaSize=(long)dimx*(long)dimy*(long)SeriaEnd;
		   //memSize=(long)1024*(long)1024*(long)2048-(long)info.PeakWorkingSetSize;
			//MessageBox(NULL, "Focal series stack to large!", "Memory alert", MB_OK);
			//printf(" 0 seria memsize= %d programm memory %d\n",dimx*dimy*SeriaEnd, info.PeakWorkingSetSize+dimx*dimy*SeriaEnd);
			//_getch();
		   //if   (seriaSize<memSize) {
			//SeriaEnd=81;

				//SeriaImage=(unsigned char *)malloc(dimx*dimy*SeriaEnd*sizeof(unsigned char));
				SeriaImage=(float *)malloc(dimx*dimy*SeriaEnd*sizeof(float));
				if	(SeriaImage==NULL) {
   						 SeriaOn=false;
						 char mem[2048];
						 sprintf(mem,"Focal series stack (%.2f Mbytes) exeed free memory size! /n dx=%d, dy=%d, dz=%d",sizeof(float)*(float)((float)dimx*(float)dimy*(float)SeriaEnd)/(1024*1024),dimx,dimy,SeriaEnd); 
						 MessageBox(NULL, mem, "Memory alert", MB_OK);
					} else {
						//memset(SeriaImage,0,dimx*dimy*SeriaEnd);
						dFOld=Df;
						Df=DefocusStart;
						DfStepSmall=DefocusStep;
						Cs=DefocusSeriaCs;
						SeriaCounter=0;
						SeriaOn=true;
					}

				 //SeriaImage[0]=0;
				 //memset()
			//printf(" 1 memsize= %d , free memory %d\n",info.PeakWorkingSetSize,(long)(1024*1024)*2048-info.PeakWorkingSetSize);
			//_getch();

			 
			break;
		case 'W':
						 MessageBox(NULL, "W", "Memory alert", MB_OK);

			break;
        default:
            break;
    }
	if(key!=0)
		ErrorOn=false;

   // printf("sigma = %f\n", sigma);
	CreateConsole();
    glutPostRedisplay();
}
void processSpecialKeys(int key, int x, int y) {
int specialKey = glutGetModifiers();
//printf("Skey %d %d\n",key,specialKey);

	switch(key) {
		case GLUT_KEY_F1:
			
			ShellExecute(GetDesktopWindow(), "open", "notepad", NULL, NULL, SW_SHOWNORMAL); 

		case GLUT_KEY_LEFT:

			if (   specialKey==GLUT_ACTIVE_CTRL) {
				height=height-10;
				width=width-(int)(10*aspect);
				}
	 			break;
		case GLUT_KEY_RIGHT:

			if (   specialKey==GLUT_ACTIVE_CTRL) {
				height=height+10;
				width=width+(int)(10*aspect);
				}


			break;

		case GLUT_KEY_UP:

			 switch(specialKey) {
				 case GLUT_ACTIVE_SHIFT:
					 Cs+=10000;
					 break;
				 case GLUT_ACTIVE_ALT:
					 dZ+=1.0;
					 break;
				 case GLUT_ACTIVE_CTRL:
					 REFRESH_DELAY+=200;
					 if (REFRESH_DELAY>10000)
						 REFRESH_DELAY=10000;
					 break;
				 default :
					 Cs+=100000;
				 }
	/*		if (specialKey==GLUT_ACTIVE_SHIFT) {
				  Cs+=10000;
			
				} else {
			if (specialKey==GLUT_ACTIVE_ALT)
				{
					dZ+=1.0;

				} else {
				  Cs+=100000;
				}
				}*/
			//if	   (binaryLevel>= img_max)
			//	   binaryLevel=	img_max;
			//changes=true;
			break;
				 case GLUT_KEY_DOWN:
				 switch(specialKey) {
				 case GLUT_ACTIVE_SHIFT:
					 Cs-=10000;
					 break;
				 case GLUT_ACTIVE_ALT:
					 dZ-=1.0;
					 break;
				 case GLUT_ACTIVE_CTRL:
					 REFRESH_DELAY-=200;
					 if (REFRESH_DELAY<100)
						 REFRESH_DELAY=100;
					 break;
				 default :
					 Cs-=100000;
				 }
			//if (   specialKey==GLUT_ACTIVE_SHIFT) {
			//	  Cs-=10000;
			//
			//	} else {
			//
			//		if (specialKey==GLUT_ACTIVE_ALT)
			//		{
			//		dZ-=1.0;
			//	
			//		} else {
			//	  Cs-=100000;
			//	}
			//
			//	}

			//if	   (binaryLevel<= img_min)
			//	   binaryLevel=	img_min;
			//changes=true;
				 // Cs-=100000;

			break;
	}
		CreateConsole();
   // glutPostRedisplay();

	}
void TimeEvent(int i) {
	if (SeriaOn ) {
		//SaveImage1(strcat (path,fname),d_result,dimx,dimy);
		//CopyByteImageToSeria(SeriaImage,d_result,dimx,dimy,SeriaCounter);
		CopyFloatImageToSeria(SeriaImage,d_result,dimx,dimy,SeriaCounter);
		defocusSeria[SeriaCounter]=Df*1.0e-09f;
  		keyboard('6', 0, 0);
		SeriaCounter++;
		if (SeriaCounter==SeriaEnd) {
		   SeriaOn=false;
		   keyboard('4', 0, 0);
	   		pch=strrchr(filename,'\\');
			strncpy(path,filename,pch-filename);
			path[pch-filename+1]='\0';
  			//substr(fname,filename,(int)(pch - filename + 2),(int)strlen(filename)-(int)(pch - filename)+1);
			char fname1[4096];
			sprintf(fname1,"\\float_dF_%.0fA_Cs_%.0f_um_dZ_%.0f_nm_%s",DfStepSmall*10.0,Cs/1.0e03,dZ,filename_short);
			//sprintf(fname1,"\\byte_dF_%.0fA_Cs_%.0f_um_dZ_%.0f_nm_%s",DfStepSmall*10.0,Cs/1.0e03,dZ,filename_short);
			//SaveByteImageSeria(SeriaImage,strcat (path,fname1),dimx,dimy,SeriaCounter);
			SaveFloatImageSeria(SeriaImage,strcat (path,fname1),dimx,dimy,SeriaCounter);
			free(SeriaImage);
			DfStepSmall=1.0f;
			Df=dFOld;
			Cs=-10000.0;
			}
		}
	if(NoiseOn==1) {
		hipMemcpy(d_tmp_show,d_tmp_a,dimx*dimy*sizeof(hipfftComplex),hipMemcpyDeviceToDevice);
		hipMemset(d_tmp_a,0,dimx*dimy*sizeof(hipfftComplex));
	//	//hipMemset(d_result,0,dimx*dimy*sizeof(uint));
	//	hipMemset(d_tmp_c,0,dimx*dimy*sizeof(hipfftComplex));
					//REFRESH_DELAY=1000;
	} else {
					REFRESH_DELAY=100;

	}
	glutReshapeWindow(width,height);
	 //printf("\t timer %d\n",i);
	glutTimerFunc( REFRESH_DELAY, TimeEvent,1);
    glutPostRedisplay();
	}
void reshape(int x, int y)
{
    glViewport(0, 0, x, y);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0); 
	//glutReshapeWindow(width,height);
	  //gluPerspective(45.0, (float)width / (float)height, 0.1, 100.0);
//	if (y==0)
//y=1; // avoid div by 0
//glViewport(0, 0, x, y);
//glMatrixMode(GL_MODELVIEW);
//glLoadIdentity();
//if (y<=x)
//glOrtho(0, 250, 0, 250*y/x, 1.0, -1.0);
//else
//glOrtho(0, 250*y/x, 0, 250, 1.0, -1.0);
//glMatrixMode(GL_PROJECTION);
//glLoadIdentity();
	vx=x;
	vy=y;

}

void mainMenu(int i)
{
    keyboard(i, 0, 0);
}
void NMenu(int i)
{
	if (i<=0) {
		switch( i) {
		case -1:
			REFRESH_DELAY+=200;
			if (REFRESH_DELAY>10000)
				REFRESH_DELAY=10000;

			break;
		case -2:
			REFRESH_DELAY-=200;
			if (REFRESH_DELAY<100)
				REFRESH_DELAY=100;
			break;
		case -3:
			NoiseLevel+=5;
			if (NoiseLevel>100)
				NoiseLevel=100;
			break;
		case -4:
					NoiseLevel-=5;
					if (NoiseLevel<5)
						NoiseLevel=5;
					break;
		}
	


	} else {
		keyboard(i, 0, 0);
	}
}
void CsMn(int i){
	//int j;
	switch(i){
	case 1:
		//j=	GLUT_KEY_UP;
		//processSpecialKeys(j, 0, 0);
		Cs+=100000;
		break;
	case 2:
		//j=GLUT_KEY_DOWN;
		//processSpecialKeys(j, 0,0);
		Cs-=100000;

		break;
	case 3:
		Cs+=10000;
		break;
	case 4:
		Cs-=10000;
		break;
	}

}
void SubMenudZ(int i){
	switch(i){
	case 1:
		dZ+=1.0;
		break;
	case 2:
		dZ-=1.0;
		break;
	case 3:
		dZ+=10.0;
		break;
	case 4:
		dZ-=10.0;
		break;}
}
void initMenus()
{
	GLint NoiseMenu,FocusMenu,CsMenu,dZMenu;
	NoiseMenu= glutCreateMenu(NMenu);
	glutAddMenuEntry("Noise on/off   [n]", 'n');
	glutAddMenuEntry("Exposition + [Ctrl+Up]", -1);
	glutAddMenuEntry("Exposition - [Ctrl+Down]", -2);
	glutAddMenuEntry("Noise + []", -3);
	glutAddMenuEntry("Noise - []", -4);
	FocusMenu= glutCreateMenu(mainMenu);
	glutAddMenuEntry("Focus +10 nm   [8]", '8');
	glutAddMenuEntry("Focus -10 nm   [2]", '2');
	glutAddMenuEntry("Focus +1 nm    [6]", '6');
	glutAddMenuEntry("Focus -1 nm    [4]", '4');
	glutAddMenuEntry("Zoom in        [+]", '+');
	glutAddMenuEntry("Zoom out       [-]", '-');
	CsMenu=glutCreateMenu(CsMn);
	glutAddMenuEntry("Cs +0.1 mm   [Up]", 1);
	glutAddMenuEntry("Cs -0.1 mm   [Down]", 2);
	glutAddMenuEntry("Cs +0.01 mm  [Shft+Up]", 3);
	glutAddMenuEntry("Cs -0.01 mm  [Shft+Down]", 4);
	//glutAddMenuEntry("Noise and expo [n]", 'n');
	dZMenu=glutCreateMenu(SubMenudZ);
	glutAddMenuEntry("Z +1 nm   [Alt+Up]", 1);
	glutAddMenuEntry("Z -1 nm   [Alt+Down]", 2);
	glutAddMenuEntry("Z +10 nm  []", 3);
	glutAddMenuEntry("Z -10 nm  []", 4);

	glutCreateMenu(mainMenu);
    glutAddMenuEntry("Reset image    [5]", '5');
	glutAddMenuEntry("CTF            [c]", 'c');
    glutAddMenuEntry("CTF off        [v]", 'v');
	glutAddMenuEntry("File name show [f]", 'f');
	glutAddMenuEntry("Defocus seria  [w]", 'w');
	glutAddMenuEntry("Df params read [p]", 'p');
	glutAddSubMenu("Focus & Zoom",FocusMenu);
	glutAddSubMenu("Z",dZMenu);
	glutAddSubMenu("Cs",CsMenu);
	glutAddSubMenu("Noise",NoiseMenu);
    glutAddMenuEntry("Quit           [esc]", 27);
    glutAttachMenu(GLUT_RIGHT_BUTTON);
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
	 vx=glutGet (GLUT_SCREEN_WIDTH);
	 vy=glutGet (GLUT_SCREEN_HEIGHT);
	//width=width/2;
	//height=height/2;
	glutInitWindowSize(width, height);
	 //window_x = (glutGet (GLUT_SCREEN_WIDTH) - window_width)/2;
    //window_y = (glutGet (GLUT_SCREEN_HEIGHT) - window_height)/2;
	//glutInitWindowPosition(1850, 0);  

    glutCreateWindow("TEM GPU");
	//glutSetIconTitle("microsope.ico");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
	glutSpecialFunc(processSpecialKeys);
	glutTimerFunc( REFRESH_DELAY, TimeEvent,1);
	initMenus();
   // printf("Press '+' and '-' to change filter width\n");
   // printf("0, 1, 2 - change filter order\n");

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Required OpenGL extensions missing.");
        hipDeviceReset();
        exit(-1);
    }
}
void initCudaBuffers(hipfftComplex *h_img,int dimx,int dimy,int nproj)
{
    unsigned int size = dimx * dimy * nproj*sizeof(hipfftComplex);
    unsigned int sizef = dimx * dimy * nproj*sizeof(float);

    // allocate device memory
    checkCudaErrors( hipMalloc( (void**) &d_img, size));
    checkCudaErrors( hipMalloc( (void**) &d_tmp_c, dimx * dimy * sizeof(hipfftComplex)));
    checkCudaErrors( hipMalloc( (void**) &d_tmp_show, dimx * dimy * sizeof(hipfftComplex)));
    checkCudaErrors( hipMalloc( (void**) &d_tmp_a, dimx * dimy * sizeof(hipfftComplex)));
    checkCudaErrors( hipMalloc( (void**) &d_result_tex, sizef));
	checkCudaErrors( hipMemset(d_tmp_a,0,dimx*dimy*sizeof(hipfftComplex)));
	checkCudaErrors( hipMemset(d_tmp_show,0,dimx*dimy*sizeof(hipfftComplex)));

    checkCudaErrors( hipMemcpy( d_img, h_img, size, hipMemcpyHostToDevice));

     sdkCreateTimer( &timer);
	 ///////!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	//free(h_img);

	tex_image_2d.filterMode = hipFilterModeLinear;
	//tex_image_2d.filterMode = hipFilterModePoint;
	//tex_image_2d.addressMode[0] = hipAddressModeClamp; 
	//tex_image_2d.addressMode[1] = hipAddressModeClamp; 
	tex_image_2d.addressMode[0] = hipAddressModeWrap; 
	tex_image_2d.addressMode[1] = hipAddressModeWrap; 
	//tex_image_2d.addressMode[1] = hipAddressModeWrap; 

	//tex_image_2d.normalized = true;

	checkCudaErrors( hipMallocArray( &cu_array, &channelDesc, dimx, dimy )); 
	checkCudaErrors( hipBindTextureToArray( tex_image_2d, cu_array, channelDesc));
	/*int BLOCK_DIM=16;
		dim3 grid(dimx / BLOCK_DIM, dimy / BLOCK_DIM, 1);
		dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
			setup_kernel<<<grid,threads>>>(devStates,(unsigned long)time(NULL),dimx,dimy);
*/
}
void cleanup()
{
     sdkDeleteTimer( &timer);
    /*if (!h_img) {
	    free(h_img);
	}*/

    checkCudaErrors(hipFree(d_img));
	
    //checkCudaErrors(hipFree(devStates));

    //if (!runBenchmark) {
        if (pbo) {
            checkCudaErrors(cudaGLUnregisterBufferObject(pbo));    
            glDeleteBuffersARB(1, &pbo);
        }
        if (texid) {
            glDeleteTextures(1, &texid);
        }
    //}
	hipfftDestroy(  plan );
	//printf("cleanup\n");
	//_getch();
    /*if (g_CheckRender) {
        delete g_CheckRender; g_CheckRender = NULL;
    }*/
}
void initGLBuffers()
{
    // create pixel buffer object to store final image
    glGenBuffersARB(1, &pbo);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, dimx*dimy*sizeof(GLubyte)*4, h_img, GL_STREAM_DRAW_ARB);

    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    checkCudaErrors(cudaGLRegisterBufferObject(pbo));

    // create texture for display
    glGenTextures(1, &texid);
    glBindTexture(GL_TEXTURE_2D, texid);
    //glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, dimx, dimy, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);
}



//bool getinfo()
//{
//    string temp;
//    bool bResult = false;
//    DWORD size;
//    DWORD dummy;
//    char filename[ 2048 ];
//    unsigned int len;
//    GetModuleFileName( NULL, filename, 2047 );
//    size = GetFileVersionInfoSize( filename, &dummy );
//	printf("size %d\n",size);
//	printf("file name %s\n", filename);
//    if( size == 0 )
//    {
//        ver = "No Version Information!";
//        return true;
//    }
//    char* buffer = new char[ size ];
//    VS_FIXEDFILEINFO* data = NULL;
//    if( buffer == NULL ){ return true; }
//    bResult = GetFileVersionInfo( filename, 0, size, (void*)buffer );
//    if( !bResult )
//    {
//        ver = "error"; // STRLASTERROR is a custom macro
//        return true;
//    }
//    bResult = VerQueryValue( buffer, "\\", (void**)&data, &len );
//    if( !bResult || data == NULL || len != sizeof(VS_FIXEDFILEINFO) )
//    {
//        ver = "Could Not Retrieve Values!";
//        return true;
//    }
//    // here I would extract the needed values
//	ver="OK";
//    delete[] buffer;
//    return true;
//	}
void  OpenParamFile(char * filenamepath){
		
	char *pch1=strrchr(filenamepath,'\\');
	char path1[4096];
	//size_t found;
	strncpy(path1,filenamepath,pch1-filenamepath);

	path1[pch1-filenamepath]='\0';
	      string textLine;
		 // printf("path='%s'\n",path1);
		 // _getch();
      // try to open a file
	  //ifstream ifs("D:\\model\\amoph_complex\\step\\new_fe1000_30x30x1_step_00_Angstr.ace", ifstream::in); 
	/*  ifstream ifs(strcat (path1,"\\GPU_TEM.ini"), ifstream::in); 
	  //ifstream ifs("D:\\model\\Crystall\\Si_20x20x5_.ACE", ifstream::in); 
      if (ifs.good())   { // if opening is successful
            // while file has lines
            while (!ifs.eof()) {
                  // read line of text
                  getline(ifs, textLine);
                  // print it to the console
                 // cout << textLine << endl;

				  //if (textLine.find("Atom "))
//cout << textLine.find("Atom ") << endl;
				   // different member versions of find in the same order as above:
				 // found=textLine.find("defocusseriasize");
				//	if (found!=textLine.npos ) {

				//		}
				  char ip[256], mac[256];
				  if(sscanf(textLine, "%s=%s", ip, mac) != 2)
        {
                fprintf(stderr, "Syntax error, line \n");
                continue;
        }	   
			  printf("Line :  param %s val %s\n",  ip, mac);
				}
		  }	 else	  {
            // otherwise print a message
			  cout << "ERROR: can't open GPU_TEM.ini file: " << path1 <<endl;
			  cout << "Press any key to continue."  <<endl;
			_getch();
			
		  }
	  //_getch();
	  */
	 		 FILE *fp;
		fopen_s(&fp,strcat (path1,"\\gpu_tem.ini"), "r");
    if (!fp) {
       sprintf(ErrorMessage, "Error load '%s' %s", path1,strerror( errno ) );
       // fprintf(stderr, "Press any key to continue.");
		//_getch();
		ErrorOn=true;
        return;
    } else {
		sprintf(ErrorMessage, "Loaded '%s'", path1);
		// fprintf(stderr, "Press any key to continue.");
		//_getch();
		ErrorOn=true;
	}

	 char line[256];
   int linenum=0;
while(fgets(line, 256, fp) != NULL)
{
        char parameter_name[256], parameter_value[256];

        linenum++;
        if(line[0] == '#') continue;

        if(sscanf(line, "%s %s", parameter_name, parameter_value) != 2)
        {
                fprintf(stderr, "Syntax error, line %d\n", linenum);
                continue;
        }
	     if (strcmp(parameter_name,"defocusseriasize")==0)
		   SeriaEnd=atoi(parameter_value);
	     if (strcmp(parameter_name,"defocusstart")==0)
		   DefocusStart=(float)atof(parameter_value);
		 if (strcmp(parameter_name,"defocusstep")==0)
		   DefocusStep=(float)atof(parameter_value);
		 if (strcmp(parameter_name,"defocusseriacs")==0)
		   DefocusSeriaCs=(float)atof(parameter_value);
		 
        //printf("Line %d:  IP %s MAC %s\n", linenum, ip, mac);
}
	//_getch();
	fclose(fp);
	}
int main(int argc,  char** argv)
{
	printf("argc=%d\n",argc);
	//	_getch();
	if	 (argc >= 2   ) {

	if (strcmp(argv[1],"/d") || strcmp(argv[1],"/D"))
		debugON=true;
	}

	if(!InitCUDA()) {
		return 0;
	}
	if (debugON)
		printf("InitCUDA OK\n");

    hStdin = GetStdHandle(STD_INPUT_HANDLE); 
    hStdout = GetStdHandle(STD_OUTPUT_HANDLE); 
    if (hStdin == INVALID_HANDLE_VALUE || hStdout == INVALID_HANDLE_VALUE) 
    {
        MessageBox(NULL, TEXT("GetStdHandle"),TEXT("Console Error"), MB_OK);
        return -1;
    }
	if (debugON)
		printf("hStdin &  hStdout OK\n");

	//strcpy(APP_path,argv[0]);
	//printf("path = %s\n",APP_path);
//	wchar_t fullPath[MAX_PATH];
//GetModuleFileName(NULL, fullPath, MAX_PATH);
//
//// cut the string at the last slash, because we just want the directory
//// not the actual executeable file name. 
//TCHAR *lastSlash = wcsrchr(fullPath, L'\\');
//lastSlash = 0;
//	printf("path = %s\n",fullPath);
	/*char myPath[_MAX_PATH+1];
GetModuleFileName(NULL,myPath,_MAX_PATH);
	printf("path = %s\n",myPath);
	wgetcwd*/
	//wchar_t* buffer;

   // Get the current working directory: 
  // if( (buffer = _wgetcwd( NULL, 0 )) == NULL )
   //   perror( "_wgetcwd error" );
  // else
   //{
   //   wprintf( L"%s \n", buffer );

  //    free(buffer);
  // }
	//_getch();

	//getinfo();
	//printf("%s \n",ver);
	//_getch();
	//return 0;
//	HICON hIcon1 = (HICON) LoadImage( NULL, MAKEINTRESOURCE(IDI_ICON1), IMAGE_ICON,16,16,LR_DEFAULTSIZE); 
	//SetClassLong(hStdout, -14, (LONG) LoadIcon(hinst, MAKEINTRESOURCE(102))   ); 
//	HANDLE hIcon1 = LoadImage(NULL, "microsope.ico", IMAGE_ICON, 0, 0, LR_LOADFROMFILE);
	//SendMessage( hStdout, STM_SETIMAGE, IMAGE_ICON, (LPARAM)(UINT)hIcon1);

	//hIcon = LoadIcon(NULL,MAKEINTRESOURCE(1));
	/*HICON hIcon = LoadIcon (NULL, IDI_APPLICATION) ;

	SetIcon( FindWindow(GlutClassName, Title), hIcon);*/

	
	 /*glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowPosition(100, 100);
    glutInitWindowSize(ClientWidth, ClientHeight);
    glutCreateWindow("Point");
    
    srand( (unsigned)time( NULL ) );
    glClearColor((float)(rand()%100)/100.0, (float)(rand()%100)/100.0, (float)(rand()%100)/100.0, 0.0);
    
    glutMouseFunc(mouse);
    glutKeyboardFunc(KeyDown);
    
    glutIdleFunc(Display);
    glutDisplayFunc(Display);
    glutReshapeFunc(Reshape);
    
    glutMainLoop();*/
	//float dE=
	//printf("Plank = %.3e c= %.3e E= %.3e\n",Plank_h,c_light,Eng);
		//printf("Lambda %.6e\n",Lambda);
	
	if	 (argc < 2   ) {
		HWND test=GetConsoleHwnd();
		bool res=ShowOpenDialogMy(test, "Open wave complex function", "", "wav\0*.wav\0img\0*.img\0All\0*.*\0",0,filename);
	 //printf("res ='%d'\n",res);
	   if (res==NULL)
		   exit(-2);
			 NewFile2Open=true;
		} else {
			if (!debugON){
 
		  strcpy(filename,argv[1]);
			 NewFile2Open=true;
			} else {
				HWND test=GetConsoleHwnd();
				bool res=ShowOpenDialogMy(test, "Open wave complex function", "", "wav\0*.wav\0img\0*.img\0All\0*.*\0",0,filename);
				//printf("res ='%d'\n",res);
				if (res==NULL)
					exit(-2);
				NewFile2Open=true;

			}

		}
	 //OpenParamFile(argv[0]);
		//printf(" argc %d, %s %s\n",argc,argv[0],argv[1]);
		//_getch();
//char path[2048];
	 //printf("filename ='%s'\n",filename);
	 //printf("filename ='%s'\n",filename);
	 //seed=(int)time(NULL);
	//checkCudaErrors( hipMalloc( (void**) &seed, sizeof( int ) ));
	SeriaEnd=81;
	OpenParamFile(filename)	;
	//float * image=0;
	while (NewFile2Open) {
	char * pch;
    pch=strrchr(filename,'\\');
	int result = (int)(pch-	filename+1);
	//printf(" position found at %d\n",result);
	//_getch();
	//printf("test %s %d\n",path,pch-path+1);
	//strstr
	//&filename_short=&filename;
	 if ( pch != NULL )
		 {
	//strncpy(filename_short,filename,4096);
    //filename_short[result]='\0';
	memcpy(filename_short,filename+result,4096-result);
	 printf("filename short ='%s'\n",filename_short);

	//memcpy(filename_short,filename+result,*filename-result);
	//filename_short[*filename-result]='\0';
		 }
	 hipfftComplex *image_c=0;
	hipfftDoubleComplex *image_d_c=0;
	//image=ReadMRC2Float(filename,dimx,dimy,nproj);
	//image_c=ReadMRC2Complex(filename,dimx,dimy,nproj);
	//HT=300;
	//a=70,b=70;
	char *pdest;
   //int  result;
//   printf( "String to be searched:\n   %s\n", string );
  // printf( "   %s\n   %s\n\n", fmt1, fmt2 );
   pdest = strstr( filename_short, ".img" );
   result = (int)(pdest - filename_short + 1);
   if ( pdest != NULL )
	   ImgNotWav=true;
      //printf( "%s found at position %d\n", ".img", result );
  

	nproj=1;
	//_getch();
	if (ImgNotWav) {
		float pxa,pxb;
			image_d_c=ReadImgAndParams(filename,pxa,pxb,dimx,dimy,thickness);
			a=pxa*dimx;
			b=pxb*dimy;
			HT=300;
		} else {
			image_d_c=ReadWavAndParams2(filename,a,b,c,alpha,beta,gamma,HT,dimx,dimy,thickness);
		}

	if (image_d_c == NULL) {
			fprintf(stderr,"End of story\n");
			_getch();
				return -1;
			}
	  NewFile2Open=false;
	float Eng=HT*eV*1000.0f;
	float E0=511000*eV;
	Lambda = (float)( Plank_h * c_light / sqrt( 2 * Eng * E0 + Eng * Eng ) ) * 1E9f;
	a/=10.0f;
	b/=10.0f;
	ux=1.0f/a;
	uy=1.0f/b;
	//Df=-41.0f;	
	if (debugON)
	printf("dimx=%d dimy=%d a=%.3f b=%.3f ux=%.3e uy=%.3e\n",dimx,dimy,a,b,ux,uy);
	//_getch();

	H_Cc=Cc*(3.5e-06f)*(1.0f-Eng/E0)/(1.0f+Eng/(2.0f*E0));
	Df=-1.2f*sqrt(Cs*Lambda);
	tetta=conv/(Lambda*sqrt(log(2.0f)));

	image_c = (hipfftComplex*)malloc(dimx*dimy*sizeof(hipfftComplex));
	memset(image_c,0,dimx*dimy*sizeof(hipfftComplex));
	cx=dimx*0.5f;
	cy=dimy*0.5f;
	tx=0.0f;
	ty=0.0f;
	ImagesizeInnm=a;
	//float imax,imin;
	//	imax=numeric_limits<float>::min();
	//	imin=numeric_limits<float>::max();
	/*for (int i=0;i<nproj;i++){
		for(int j=0;j<dimy;j++){
			for(int k=0;k<dimx;k++){
					int	idx	=	i*dimx*dimy	+	j*dimx	+	k;
					if (image[idx]<imin)
					imin=image[idx];
					if (image[idx]>imax)
					imax=image[idx];

				}
			}

		}
	for (int i=0;i<nproj;i++){
	for(int j=0;j<dimy;j++){
			for(int k=0;k<dimx;k++){
					int	idx	=	i*dimx*dimy	+	j*dimx	+	k;
					image[idx]=(image[idx]-imin)/(imax-imin);
					image_c[idx].y=(image[idx]-imin)/(imax-imin);
					image_c[idx].x=0.0f;
				}
			}
			}*/
	//printf("min max %.3f %.3f\n",imin,imax);
	//CalcGPUMinMax(image,dimx,dimy);
	//width=dimx,height=dimy;
    //GLint m_viewport[4];
 
	//glGetIntegerv( GL_VIEWPORT, m_viewport );
	//printf("x= %d y=%d\n",m_viewport[0],m_viewport[1]);
	//_getch();
	// int w=glutGet (GLUT_SCREEN_WIDTH);
	// int h=glutGet (GLUT_SCREEN_HEIGHT);
	//printf("x= %d y=%d\n",w,h);
	//_getch();
	double pix_per_nm,k;
	int w,h;
	w=min(dimx,1024);
	k=dimx/w;
	h=dimy/k;
	pix_per_nm=1024.0/max(a,b);
	width=w;//(int)(a*pix_per_nm);
	height=h;//(int)(b*pix_per_nm);
	aspect=(float)width/(float)height;
	if (debugON)
	printf("pix_per_nm %.3f w=%.3f h=%.3f a=%.5f  b=%.5f aspect=%.3f \n",pix_per_nm,a*pix_per_nm,b*pix_per_nm,a,b,aspect );
	if (debugON)
	printf("w=%d h=%d u=%.3f v=%.5f\n",w,h,w/a,h/b);
	//_getch();

	int device;
    struct hipDeviceProp_t prop;
    hipGetDevice( &device );
    hipGetDeviceProperties( &prop, device );
    if( !strncmp( "Tesla", prop.name, 5 ) ) {
        printf("Tesla card detected, running the test in benchmark mode (no OpenGL display)\n");
//        runBenchmark = CUTTrue;
       // g_bQAReadback = true;
		return -2;
    }      
	if (debugON)
 	 printf("device=%d!!!!!!!!!!!!!!!\n",device);
	for(int i=0;i<dimx*dimy;i++)
		{
		image_c[i].x=(float)image_d_c[i].x;
		image_c[i].y=(float)image_d_c[i].y;
		}

    hipMalloc ( (void**) &d_seed, BLOCK_DIM*BLOCK_DIM*sizeof( int ) );
	h_seed=(int*)malloc(BLOCK_DIM*BLOCK_DIM*sizeof( int ));
    //checkCudaErrors( hipMalloc( (void**) &devStates, dimx*dimy*sizeof( hiprandState ) ));

	//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
/*	union mrc_h header_out;
		memset(&header_out.buff,0,1024); 
		header_out.header.mode=4;
		header_out.header.next=0;
		header_out.header.nx=dimx;
		header_out.header.ny=dimy;
		header_out.header.nz=1;
		header_out.header.alpha=90.0f;
		header_out.header.beta=90.0f;
		header_out.header.gamma=90.0f;
		header_out.header.mx=dimx;
		header_out.header.my=dimy;
		header_out.header.mz=1;
		header_out.header.xlen=(float)dimx;
		header_out.header.ylen=(float)dimy;
		header_out.header.zlen=(float)1;
		//header_out.header.amax=imax;
		//header_out.header.amin=imin;
		//header_out.header.amean=average;
		union all_mrc_ext_h ext_header_out;
		memset(&ext_header_out.buff,0,1032*128);
		WriteMRC<hipfftComplex>("d:\\image_complex_test.mrc", image_c, header_out.header , ext_header_out);

	 */
	//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!111111



	//free(image_d_c);
    initGL(argc, argv);
    //cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
   // cudaGLSetGLDevice( 1 );

    initCudaBuffers(image_c,dimx,dimy,nproj);
	hipfftPlan2d(&plan, dimy, dimx, HIPFFT_C2C);
	for (int i=0;i<nproj;i++)
		CreateFFT(d_img+i*dimx*dimy,dimx,dimy);
	if (debugON)
 	printf("dimx1=%d, dimy1=%d\n",dimx,dimy);
	//checkCudaErrors( hipMemcpy( image_c, d_img, dimx*dimy*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	//printf("OK");
	//WriteRawFileT<hipfftComplex>(image_c,"d:\\test_cmplx_TEM.dat",dimx,dimy,1);
	//_getch();
		CreateConsole();

	initGLBuffers();
 
    atexit(cleanup);
    if (!setjmp(env))
    glutMainLoop();
	//printf("After glutMainLoop\n"); 
	//_getch();
//    glutMainLoop();
//display();
//reshape(0,0);
//DrawGLScene();
//_getch();
//	cleanup();
	//printf("finish\n");
	//_getch();
	}
    hipDeviceReset();
	//free(image);
	return 0;
}


